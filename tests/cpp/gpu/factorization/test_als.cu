#include "hip/hip_runtime.h"
#include "gtest/gtest.h"

#include "../gpu/factorization/als.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

TEST(Factorization, get_hermitianT10_eye) {
  // most simple test case, all sizes 20
  constexpr float lambda = 0.0;
  constexpr int f = 20;
  constexpr int m = 20;
  constexpr int n = 20;
  constexpr int batch_size = m;
  int block_dim = f / ALS_T10 * (f / ALS_T10 + 1) / 2;
  if (block_dim < f / 2)
    block_dim = f / 2;

  thrust::device_vector<float> tt_device(f * f * batch_size, 0.0);

  // sparse eye matrix
  int csrRowIndex_host[m + 1] = {0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10,
                                 11, 12, 13, 14, 15, 16, 17, 18, 19, 20};

  int csrColIndex_host[20] = {0,  1,  2,  3,  4,  5,  6,  7,  8,  9,
                              10, 11, 12, 13, 14, 15, 16, 17, 18, 19};

  thrust::device_vector<int> csrRowIndex_device(csrRowIndex_host,
                                                csrRowIndex_host + m + 1);
  thrust::device_vector<int> csrColIndex_device(
      csrColIndex_host, csrColIndex_host + sizeof(csrColIndex_host) /
                                               sizeof(csrColIndex_host[0]));

  thrust::device_vector<float> thetaT_device(f * m);
  for (int i = 0; i < f * m; ++i) {
    thetaT_device[i] = float(i * 0.1);
  }

  get_hermitianT10<<<batch_size, block_dim,
                     SCAN_BATCH * f / 2 * sizeof(float2)>>>(
      0, thrust::raw_pointer_cast(tt_device.data()),
      thrust::raw_pointer_cast(csrRowIndex_device.data()),
      thrust::raw_pointer_cast(csrColIndex_device.data()), lambda, m, f,
      thrust::raw_pointer_cast(thetaT_device.data()));

  OK(hipDeviceSynchronize());

  for (int v = 0; v < n; ++v) {
    thrust::host_vector<float> thetaT_theta =
        thrust::host_vector<float>(f * f, 0.0);
    for (int row = 0; row < f; ++row) {
      for (int col = 0; col < f; ++col) {
        thetaT_theta[row * f + col] =
            thetaT_device[row + v * f] * thetaT_device[col + v * f];
        ASSERT_FLOAT_EQ(tt_device[v * f * f + row * f + col],
                        thetaT_theta[row * f + col])
            << v << ", " << row << ", " << col;
      }
    }
  }
}

TEST(Factorization, get_hermitianT10_2columns) {
  // most simple test case, all sizes 20
  constexpr float lambda = 0.0;
  constexpr int f = 20;
  constexpr int m = 20;
  constexpr int n = 20;
  constexpr int batch_size = m;
  int block_dim = f / ALS_T10 * (f / ALS_T10 + 1) / 2;
  if (block_dim < f / 2)
    block_dim = f / 2;

  thrust::device_vector<float> tt_device(f * f * batch_size, 0.0);

  int csrRowIndex_host[m + 1] = {0,  2,  4,  6,  8,  10, 12, 14, 16, 18, 20,
                                 22, 24, 26, 28, 30, 32, 34, 36, 38, 40};

  // v , (v + 1) % m
  int csrColIndex_host[40] = {0,  1,  1,  2,  2,  3,  3,  4,  4,  5,
                              5,  6,  6,  7,  7,  8,  8,  9,  9,  10,
                              10, 11, 11, 12, 12, 13, 13, 14, 14, 15,
                              15, 16, 16, 17, 17, 18, 18, 19, 0,  19};

  thrust::device_vector<int> csrRowIndex_device(csrRowIndex_host,
                                                csrRowIndex_host + m + 1);
  thrust::device_vector<int> csrColIndex_device(
      csrColIndex_host, csrColIndex_host + sizeof(csrColIndex_host) /
                                               sizeof(csrColIndex_host[0]));

  thrust::device_vector<float> thetaT_device(f * m);
  for (int i = 0; i < f * m; ++i) {
    thetaT_device[i] = float(i * 0.1);
  }

  get_hermitianT10<<<batch_size, block_dim,
                     SCAN_BATCH * f / 2 * sizeof(float2)>>>(
      0, thrust::raw_pointer_cast(tt_device.data()),
      thrust::raw_pointer_cast(csrRowIndex_device.data()),
      thrust::raw_pointer_cast(csrColIndex_device.data()), lambda, m, f,
      thrust::raw_pointer_cast(thetaT_device.data()));

  OK(hipDeviceSynchronize());

  for (int v = 0; v < n; ++v) {
    thrust::host_vector<float> thetaT_theta =
        thrust::host_vector<float>(f * f, 0.0);
    for (int row = 0; row < f; ++row) {
      for (int col = 0; col < f; ++col) {
        thetaT_theta[row * f + col] +=
            thetaT_device[row + v * f] * thetaT_device[col + v * f];
        int v1 = (v + 1) % n;
        thetaT_theta[row * f + col] +=
            thetaT_device[row + v1 * f] * thetaT_device[col + v1 * f];

        ASSERT_FLOAT_EQ(tt_device[v * f * f + row * f + col],
                        thetaT_theta[row * f + col])
            << v << ", " << row << ", " << col;
      }
    }
  }
}

TEST(Factorization, get_hermitianT10_eye_2_batches) {
  constexpr int X_BATCH = 2;
  // most simple test case, all sizes 20
  constexpr float lambda = 0.0;
  constexpr int f = 20;
  constexpr int m = 20;
  constexpr int n = 20;
  int block_dim = f / ALS_T10 * (f / ALS_T10 + 1) / 2;
  if (block_dim < f / 2)
    block_dim = f / 2;

  // sparse eye matrix
  int csrRowIndex_host[m + 1] = {0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10,
                                 11, 12, 13, 14, 15, 16, 17, 18, 19, 20};

  int csrColIndex_host[20] = {0,  1,  2,  3,  4,  5,  6,  7,  8,  9,
                              10, 11, 12, 13, 14, 15, 16, 17, 18, 19};

  thrust::device_vector<int> csrRowIndex_device(csrRowIndex_host,
                                                csrRowIndex_host + m + 1);
  thrust::device_vector<int> csrColIndex_device(
      csrColIndex_host, csrColIndex_host + sizeof(csrColIndex_host) /
                                               sizeof(csrColIndex_host[0]));

  thrust::device_vector<float> thetaT_device(f * m);
  for (int i = 0; i < f * m; ++i) {
    thetaT_device[i] = float(i * 0.1);
  }

  for (int batch_id = 0; batch_id < X_BATCH; batch_id++) {
    int batch_size = 0;
    if (batch_id != X_BATCH - 1)
      batch_size = m / X_BATCH;
    else
      batch_size = m - batch_id * (m / X_BATCH);

    int batch_offset = batch_id * (m / X_BATCH);

    thrust::device_vector<float> tt_device(f * f * batch_size, 0.0);

    get_hermitianT10<<<batch_size, block_dim,
                       SCAN_BATCH * f / 2 * sizeof(float2)>>>(
        batch_offset, thrust::raw_pointer_cast(tt_device.data()),
        thrust::raw_pointer_cast(csrRowIndex_device.data()),
        thrust::raw_pointer_cast(csrColIndex_device.data()), lambda, m, f,
        thrust::raw_pointer_cast(thetaT_device.data()));

    OK(hipDeviceSynchronize());

    for (int v = 0; v < batch_size; ++v) {
      thrust::host_vector<float> thetaT_theta =
          thrust::host_vector<float>(f * f, 0.0);
      for (int row = 0; row < f; ++row) {
        for (int col = 0; col < f; ++col) {
          thetaT_theta[row * f + col] =
              thetaT_device[row + (v + batch_offset) * f] *
              thetaT_device[col + (v + batch_offset) * f];

          ASSERT_FLOAT_EQ(tt_device[v * f * f + row * f + col],
                          thetaT_theta[row * f + col])
              << v << ", " << row << ", " << col;
        }
      }
    }
  }
}