#include "hip/hip_runtime.h"
/*!
 * Modifications Copyright 2017-2018 H2O.ai, Inc.
 */
#include <hipblas.h>

#include <algorithm>
#include <limits>

#include "cgls.cuh"
#include "cml/cml_blas.cuh"
#include "cml/cml_vector.cuh"
#include "matrix/matrix_dense.h"
#include "matrix/matrix_sparse.h"
#include "projector/projector_cgls.h"
#include "projector_helper.cuh"
#include "util.h"

#include "equil_helper.cuh"

namespace h2o4gpu {

namespace {

bool kCglsQuiet = true;

template<typename T>
struct GpuData {
  hipblasHandle_t handle;
  GpuData() {
    hipblasCreate(&handle);
    CUDA_CHECK_ERR();
  }
  ~GpuData() {
    hipblasDestroy(handle);
    CUDA_CHECK_ERR();
  }
};

// CGLS Gemv struct for matrix multiplication.
template <typename T, typename M>
struct Gemv : cgls::Gemv<T> {
  const M& A;
  Gemv(const M& A) : A(A) { }
  int operator()(char op, const T alpha, const T *x, const T beta, T *y)
      const {
    return A.Mul(op, alpha, x, beta, y);
  }
};

}  // namespace

template <typename T, typename M>
ProjectorCgls<T, M>::ProjectorCgls(int wDev, const M& A)
    : _A(A) {
  // Set GPU specific this->_info.
  GpuData<T> *info = new GpuData<T>();
  this->_info = reinterpret_cast<void*>(info);
}

template <typename T, typename M>
ProjectorCgls<T, M>::~ProjectorCgls() {
  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  delete info;
  this->_info = 0;
}

template <typename T, typename M>
int ProjectorCgls<T, M>::Init() {
  if (this->_done_init)
    return 1;
  this->_done_init = true;

  ASSERT(_A.IsInit());

  return 0;
}

template <typename T, typename M>
int ProjectorCgls<T, M>::Project(const T *x0, const T *y0, T s, T *x, T *y,
                                 T tol) {
  DEBUG_EXPECT(this->_done_init);
  DEBUG_EXPECT(s >= static_cast<T>(0.));
  if (!this->_done_init || s < static_cast<T>(0.))
    return 1;

  // Get Cublas handle
  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  hipblasHandle_t hdl = info->handle;

  // Set initial x and y.
  hipMemset(x, 0, _A.Cols() * sizeof(T));
  hipMemcpy(y, y0, _A.Rows() * sizeof(T), hipMemcpyDeviceToDevice);

  // y := y0 - Ax0;
  _A.Mul('n', static_cast<T>(-1.), x0, static_cast<T>(1.), y);

  int kMaxIter = 100;
  // Minimize ||Ax - b||_2^2 + s||x||_2^2
  cgls::Solve(hdl, Gemv<T, M>(_A), static_cast<cgls::INT>(_A.Rows()),
      static_cast<cgls::INT>(_A.Cols()), y, x, s, tol, kMaxIter, kCglsQuiet);
  hipDeviceSynchronize();
 
  // x := x + x0
  cml::vector<T> x_vec = cml::vector_view_array(x, _A.Cols());
  const cml::vector<T> x0_vec = cml::vector_view_array(x0, _A.Cols());
  cml::blas_axpy(hdl, static_cast<T>(1.), &x0_vec, &x_vec);
  hipDeviceSynchronize();

  // y := Ax
  _A.Mul('n', static_cast<T>(1.), x, static_cast<T>(0.), y);
  hipDeviceSynchronize();

#ifdef DEBUG
  // Verify that projection was successful.
  T kTol = static_cast<T>(kNormEstTol);
  CheckProjection(&_A, x0, y0, x, y, s, static_cast<T>(1e1 * kTol));
#endif

  return 0;
}

#if !defined(H2O4GPU_DOUBLE) || H2O4GPU_DOUBLE==1
template class ProjectorCgls<double, MatrixDense<double> >;
template class ProjectorCgls<double, MatrixSparse<double> >;
#endif

#if !defined(H2O4GPU_SINGLE) || H2O4GPU_SINGLE==1
template class ProjectorCgls<float, MatrixDense<float> >;
template class ProjectorCgls<float, MatrixSparse<float> >;
#endif

}  // namespace h2o4gpu

