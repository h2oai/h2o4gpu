#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <math.h>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "arima.h"
#include "cuda_utils2.h"
#include "utils.cuh"

namespace h2o4gpu {

#define BLOCK_SIZE 32

inline hipsolverStatus_t cusolverDnTgeqrf_bufferSize(hipsolverHandle_t handle,
                                                    int m, int n, float *A,
                                                    int lda, int *lwork) {
  return hipsolverDnSgeqrf_bufferSize(handle, m, n, A, lda, lwork);
}

inline hipsolverStatus_t cusolverDnTgeqrf_bufferSize(hipsolverHandle_t handle,
                                                    int m, int n, double *A,
                                                    int lda, int *lwork) {
  return hipsolverDnDgeqrf_bufferSize(handle, m, n, A, lda, lwork);
}

inline hipsolverStatus_t cusolverDnTgeqrf(hipsolverHandle_t handle, int m,
                                         int n, float *A, int lda, float *TAU,
                                         float *Workspace, int Lwork,
                                         int *devInfo) {
  return hipsolverDnSgeqrf(handle, m, n, A, lda, TAU, Workspace, Lwork, devInfo);
}

inline hipsolverStatus_t cusolverDnTgeqrf(hipsolverHandle_t handle, int m,
                                         int n, double *A, int lda, double *TAU,
                                         double *Workspace, int Lwork,
                                         int *devInfo) {
  return hipsolverDnDgeqrf(handle, m, n, A, lda, TAU, Workspace, Lwork, devInfo);
}

inline hipsolverStatus_t cusolverDnTormqr(hipsolverHandle_t handle,
                                         hipblasSideMode_t side,
                                         hipblasOperation_t trans, int m, int n,
                                         int k, const float *A, int lda,
                                         const float *tau, float *C, int ldc,
                                         float *work, int lwork, int *devInfo) {
  return hipsolverDnSormqr(handle, side, trans, m, n, k, A, lda, tau, C, ldc,
                          work, lwork, devInfo);
}

inline hipsolverStatus_t cusolverDnTormqr(
    hipsolverHandle_t handle, hipblasSideMode_t side, hipblasOperation_t trans,
    int m, int n, int k, const double *A, int lda, const double *tau, double *C,
    int ldc, double *work, int lwork, int *devInfo) {
  return hipsolverDnDormqr(handle, side, trans, m, n, k, A, lda, tau, C, ldc,
                          work, lwork, devInfo);
}

inline hipblasStatus_t cublasTtrsm(hipblasHandle_t handle, hipblasSideMode_t side,
                                  hipblasFillMode_t uplo,
                                  hipblasOperation_t trans,
                                  hipblasDiagType_t diag, int m, int n,
                                  const float *alpha, const float *A, int lda,
                                  float *B, int ldb) {
  return hipblasStrsm(handle, side, uplo, trans, diag, m, n, alpha, A, lda, B,
                     ldb);
}

inline hipblasStatus_t cublasTtrsm(hipblasHandle_t handle, hipblasSideMode_t side,
                                  hipblasFillMode_t uplo,
                                  hipblasOperation_t trans,
                                  hipblasDiagType_t diag, int m, int n,
                                  const double *alpha, const double *A, int lda,
                                  double *B, int ldb) {
  return hipblasDtrsm(handle, side, uplo, trans, diag, m, n, alpha, A, lda, B,
                     ldb);
}

template <class T>
__global__ void copy_kernel(const T *__restrict d_in, T *__restrict d_out,
                            const int M, const int N) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < N) && (j < N)) d_out[j * N + i] = d_in[j * M + i];
}

template <class T>
__global__ void ts_data_to_matrix_kernel(const T *__restrict data, T *X,
                                         const int ldx, const int n) {
  // row
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  // col, time axis
  const int j = blockIdx.y * blockDim.y + threadIdx.y;

  // TODO: optimize with shared memory(read data only once)
  if (i < n && i < ldx) {
    X[j * ldx + i] = data[j + i];
  }
}

template <class T>
__global__ void update_residual_kernel(T *residual, const T *data, const T *phi,
                                       const int p, const T *last_residual,
                                       const T *theta, const int q,
                                       const int n) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  // TODO: optimize with shared memory(read data only once)
  // at least cache phi and theta
  if (i < n) {
    T AR_prediction = 0;
    for (int j = 0; j < p; ++j) {
      AR_prediction += data[i + j + 1] * phi[j];
    }

    T MA_prediction = 0;
    for (int j = 0; j < q; ++j) {
      MA_prediction += last_residual[i + j + 1] * theta[j];
    }

    residual[i] = data[i] - AR_prediction - MA_prediction;
  }
}

LeastSquaresSolver::LeastSquaresSolver(int rows, int cols)
    : rows(rows), cols(cols) {
  safe_cusolver(hipsolverDnCreate(&this->solver_handle));
  safe_cublas(hipblasCreate(&this->cublas_handle));
}

LeastSquaresSolver::~LeastSquaresSolver() {
  safe_cusolver(hipsolverDnDestroy(this->solver_handle));
  safe_cublas(hipblasDestroy(this->cublas_handle));
}

template <typename T>
void LeastSquaresSolver::Solve(T *A, T *B) {
  int work_size = 0;
  int *devInfo;

  OK(hipMalloc(&devInfo, sizeof(int)));

  /**********************************/
  /* COMPUTING THE QR DECOMPOSITION */
  /**********************************/

  // --- CUDA QR GEQRF preliminary operations
  T *d_TAU;
  OK(hipMalloc(&d_TAU, min(rows, cols) * sizeof(T)));
  safe_cusolver(cusolverDnTgeqrf_bufferSize(solver_handle, rows, cols, A, rows,
                                            &work_size));
  T *work;
  OK(hipMalloc(&work, work_size * sizeof(T)));

  // CUDA GEQRF execution: The matrix R is overwritten in upper triangular
  // part of A, including diagonal
  // elements. The matrix Q is not formed explicitly, instead, a sequence of
  // householder vectors are stored in lower triangular part of A.

  safe_cusolver(cusolverDnTgeqrf(solver_handle, rows, cols, A, rows, d_TAU,
                                 work, work_size, devInfo));
  int devInfo_h = 0;
  OK(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
  assert(devInfo_h == 0);

  /*****************************/
  /* SOLVING THE LINEAR SYSTEM */
  /*****************************/

  // --- CUDA ORMQR execution: Computes the multiplication Q^T * B and stores it
  // in B
  safe_cusolver(cusolverDnTormqr(solver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                                 rows, 1, min(rows, cols), A, rows, d_TAU, B,
                                 rows, work, work_size, devInfo));

  OK(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
  OK(hipFree(d_TAU));
  OK(hipFree(devInfo));
  OK(hipFree(work));
  assert(devInfo_h == 0);

  // --- Solving an upper triangular linear system R * x = Q^T * B

  const T alpha = 1.;
  safe_cublas(cublasTtrsm(
      cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N,
      HIPBLAS_DIAG_NON_UNIT, cols, 1, &alpha, A, rows, B, cols));

  OK(hipDeviceSynchronize());
}

template <class T>
__global__ void differencing(T *out, const T *in, const int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    if (i < n - 1)
      out[i] = in[i] - in[i + 1];
    else
      out[n - 1] = NAN;
  }
}

template <class T>
__global__ void undifferencing(T *out, const T *in, const int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    if (i > 0)
      out[i] = in[i] - in[i - 1];
    else
      out[0] = NAN;
  }
}

template <class T>
ARIMAModel<T>::ARIMAModel(int p, int d, int q, int length)
    : p(p), d(d), q(q), length(length) {
  assert(q >= 0);
  assert(d >= 0);
  assert(p >= 0);
  assert(length > 0);
  assert(p > 0 || q > 0);
  if (q > 0) {
    OK(hipHostMalloc(&this->theta, sizeof(T) * q));
    memset(this->theta, 0, sizeof(T) * q);
    OK(hipMalloc(&this->d_theta, sizeof(T) * q));
    OK(hipMemset(this->d_theta, 0, sizeof(T) * q));
  }

  if (p > 0) {
    OK(hipHostMalloc(&this->phi, sizeof(T) * p));
    memset(this->phi, 0, sizeof(T) * p);
    OK(hipMalloc(&this->d_phi, sizeof(T) * p));
    OK(hipMemset(this->d_phi, 0, sizeof(T) * p));
  }

  OK(hipMalloc(&this->d_buffer, sizeof(T) * this->DifferencedLength()));
};

template <class T>
ARIMAModel<T>::~ARIMAModel() {
  if (q > 0) {
    OK(hipHostFree(this->theta));
    OK(hipFree(this->d_theta));
  }
  if (p > 0) {
    OK(hipHostFree(this->phi));
    OK(hipFree(this->d_phi));
  }
  OK(hipFree(this->d_buffer));
}

template <class T>
void ARIMAModel<T>::Difference(T *out, const T *in, int length) {
  int block_size, grid_size;
  compute1DInvokeConfig(length, &grid_size, &block_size, differencing<T>);
  differencing<T><<<grid_size, block_size>>>(out, in, length);
}

template <class T>
void ARIMAModel<T>::AsMatrix(const T *ts_data, T *A, int depth, int lda,
                             int length) {
  if (depth > 0) {
    int n = length - depth + 1;
    dim3 grid_size(DIVUP(n, BLOCK_SIZE), depth);
    dim3 block_size(BLOCK_SIZE, 1);
    ts_data_to_matrix_kernel<T><<<grid_size, block_size>>>(ts_data, A, lda, n);
  }
}

template <class T>
void ARIMAModel<T>::AsMatrix(const T *ts_a, const T *ts_b, T *A, int a_depth,
                             int b_depth, int lda, int length) {
  ARIMAModel<T>::AsMatrix(ts_a, A, a_depth, lda, length);
  ARIMAModel<T>::AsMatrix(ts_b, A + a_depth * lda, b_depth, lda, length);
}

template <class T>
void ARIMAModel<T>::Apply(T *residual, const T *ts_data, const T *phi,
                          const int p, const T *last_residual, const T *theta,
                          const int q, int length) {
  int block_size, grid_size;
  compute1DInvokeConfig(length - max(p, q), &grid_size, &block_size,
                        update_residual_kernel<T>);

  update_residual_kernel<T><<<grid_size, block_size>>>(
      residual, ts_data, phi, p, last_residual, theta, q, length - max(p, q));
}

template <class T>
void ARIMAModel<T>::Fit(const T *data, const int maxiter) {
  OK(hipMalloc(&this->d_data_src, sizeof(T) * this->length));
  OK(hipMalloc(&this->d_data_differenced, sizeof(T) * this->length));
  OK(hipMalloc(&this->d_last_residual, sizeof(T) * this->length));
  OK(hipMemset(this->d_last_residual, 0, sizeof(T) * this->length));

  OK(hipMemcpy(this->d_data_src, data, sizeof(T) * this->length,
                hipMemcpyHostToDevice));

  for (auto i = 0; i < this->d; ++i) {
    this->Difference(this->d_data_differenced, this->d_data_src, this->length);
    OK(hipGetLastError());
    OK(hipDeviceSynchronize());
    std::swap(this->d_data_src, this->d_data_differenced);
  }

  OK(hipMemcpy(this->d_data_differenced, this->d_data_src,
                this->length * sizeof(T), hipMemcpyDeviceToDevice));

  OK(hipMemcpy(this->d_buffer, this->d_data_src,
                sizeof(T) * this->DifferencedLength(),
                hipMemcpyDeviceToDevice));

  if (this->p > 0) {
    T *X;
    OK(hipMalloc(&X, sizeof(T) * this->ARLength() * this->p));
    this->AsMatrix(this->d_data_src + 1, X, this->p, this->ARLength(),
                   this->DifferencedLength());

    OK(hipGetLastError());
    OK(hipDeviceSynchronize());

    LeastSquaresSolver solver(this->ARLength(), this->p);
    solver.Solve(X, this->d_data_differenced);

    OK(hipMemcpy(this->d_phi, this->d_data_differenced, sizeof(T) * this->p,
                  hipMemcpyDeviceToDevice));

    OK(hipMemcpy(this->phi, this->d_data_differenced, sizeof(T) * this->p,
                  hipMemcpyDeviceToHost));
    OK(hipFree(X));
  }

  if (this->q > 0) {
    T *X;
    int rows = min(this->MALength() == 0 ? this->ARLength() : this->MALength(),
                   this->ARLength() == 0 ? this->MALength() : this->ARLength());

    OK(hipMalloc(&X, sizeof(T) * rows * (this->q + this->p)));

    for (int i = 0; i < maxiter; ++i) {
      OK(hipMemcpy(this->d_data_src, this->d_buffer,
                    sizeof(T) * this->DifferencedLength(),
                    hipMemcpyDeviceToDevice));

      this->Apply(this->d_data_differenced, this->d_data_src, this->d_phi,
                  this->p, this->d_last_residual, this->d_theta, this->q,
                  this->DifferencedLength());
      OK(hipGetLastError());
      OK(hipDeviceSynchronize());

      this->AsMatrix(this->d_data_src + 1, this->d_data_differenced + 1, X,
                     this->p, this->q, rows, this->DifferencedLength());

      OK(hipGetLastError());
      OK(hipDeviceSynchronize());
      LeastSquaresSolver solver(rows, this->q + this->p);

      solver.Solve(X, this->d_data_src);

      if (this->p > 0) {
        OK(hipMemcpy(this->d_phi, this->d_data_src, sizeof(T) * this->p,
                      hipMemcpyDeviceToDevice));
      }
      OK(hipDeviceSynchronize());
      OK(hipGetLastError());

      OK(hipMemcpy(this->d_theta, this->d_data_src + this->p,
                    sizeof(T) * this->q, hipMemcpyDeviceToDevice));

      std::swap(this->d_last_residual, this->d_data_differenced);
    }

    OK(hipMemcpy(this->phi, this->d_data_src, sizeof(T) * this->p,
                  hipMemcpyDeviceToHost));

    OK(hipMemcpy(this->theta, this->d_data_src + this->p, sizeof(T) * this->q,
                  hipMemcpyDeviceToHost));

    OK(hipFree(X));
  }
  OK(hipFree(this->d_data_src));
  OK(hipFree(this->d_data_differenced));
  OK(hipFree(this->d_last_residual));
}

template class ARIMAModel<float>;
template class ARIMAModel<double>;
}  // namespace h2o4gpu

template <typename T>
void arima_fit(const int p, const int d, const int q, const T *ts_data,
               const int length, T *theta, T *phi, const int maxiter) {
  h2o4gpu::ARIMAModel<T> model(p, d, q, length);
  model.Fit(ts_data, maxiter);
  if (p > 0) std::memcpy(phi, model.Phi(), sizeof(T) * p);
  if (q > 0) std::memcpy(theta, model.Theta(), sizeof(T) * q);
}

void arima_fit_float(const int p, const int d, const int q,
                     const float *ts_data, const int length, float *theta,
                     float *phi, const int maxiter) {
  arima_fit<float>(p, d, q, ts_data, length, theta, phi, maxiter);
}

void arima_fit_double(const int p, const int d, const int q,
                      const double *ts_data, const int length, double *theta,
                      double *phi, const int maxiter) {
  arima_fit<double>(p, d, q, ts_data, length, theta, phi, maxiter);
}