#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <algorithm>
#include <thrust/inner_product.h>
#include <thrust/extrema.h>

namespace tsvd
{
	void max_index_per_column(Matrix<tsvd_float>& A, std::vector<int>& result_array, DeviceContext& context){

		int result;
		for (int i=0; i<A.columns(); i++) {
			safe_cublas(hipblasIsamax(context.cublas_handle, A.rows(), A.data() + i*A.rows(), 1, &result));
			if (i == 0) {
				result_array[i] = result - 1;
			} else  {
				result_array[i] = result - 1 + A.rows();
			}
		}
	}

	void max_index_per_column(Matrix<tsvd_double>& A, std::vector<int>& result_array, DeviceContext& context){

		int result;
		for (int i=0; i<A.columns(); i++) {
			safe_cublas(hipblasIdamax(context.cublas_handle, A.rows(), A.data() + i*A.rows(), 1, &result));
			if (i == 0) {
				result_array[i] = result - 1;
			} else  {
				result_array[i] = result - 1 + A.rows();
			}
		}
	}

	template<typename T, typename U>
	void multiply(Matrix<T>& A, const U a, DeviceContext& context)
	{
		thrust::transform(A.dptr(), A.dptr() + A.size(), A.dptr(), [=]__device__ (U val)
						  {
							  return val * a;
						  }
		);
	}

	template<typename T>
	void subtract(const Matrix<T>& A, const Matrix<T>& B, Matrix<T>& C, DeviceContext& context)
	{
		auto counting = thrust::make_counting_iterator(0);
		const T* d_A = A.data();
		const T* d_B = B.data();
		T* d_C = C.data();
		thrust::for_each(counting, counting + A.rows() * A.columns(), [=]__device__(int idx)
						 {
							 d_C[idx] = d_A[idx] - d_B[idx];
						 });
	}

	template<typename T>
	void add(const Matrix<T>& A, const Matrix<T>& B, Matrix<T>& C, DeviceContext& context)
	{
		auto counting = thrust::make_counting_iterator(0);
		const T* d_A = A.data();
		const T* d_B = B.data();
		T* d_C = C.data();
		thrust::for_each(counting, counting + A.rows() * A.columns(), [=]__device__(int idx)
						 {
							 d_C[idx] = d_A[idx] + d_B[idx];
						 });
	}


	template<typename T>
	void normalize_vector_thrust(Matrix<T>& M, DeviceContext& context){
		float M_inner = thrust::inner_product(M.dptr(), M.dptr() + M.size(), M.dptr(), 0.0f); //Will allocate memory for every call to fxn.
		M.transform([=]__device__ (float val){return val / std::sqrt(M_inner);});
	}

	void multiply_diag(const Matrix<tsvd_float>& A, const Matrix<tsvd_float>& B, Matrix<tsvd_float>& C, DeviceContext& context, bool left_diag)
	{
		hipblasSideMode_t mode = left_diag ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT;

		int m = C.rows();
		int n = C.columns();
		int lda = m;
		int incx = 1; //Review what this should be...
		int ldc = m;

		safe_cublas(hipblasSdgmm(context.cublas_handle, mode, m, n, A.data(), lda, B.data(), incx, C.data(), ldc));
	}

	void multiply_diag(const Matrix<tsvd_double>& A, const Matrix<tsvd_double>& B, Matrix<tsvd_double>& C, DeviceContext& context, bool left_diag)
	{
		hipblasSideMode_t mode = left_diag ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT;

		int m = C.rows();
		int n = C.columns();
		int lda = m;
		int incx = 1; //Review what this should be...
		int ldc = m;

		safe_cublas(hipblasDdgmm(context.cublas_handle, mode, m, n, A.data(), lda, B.data(), incx, C.data(), ldc));
	}

	void multiply(const Matrix<tsvd_float>& A, const Matrix<tsvd_float>& B, Matrix<tsvd_float>& C, DeviceContext& context, bool transpose_a, bool transpose_b, tsvd_float alpha)
	{
		hipblasOperation_t op_a = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		hipblasOperation_t op_b = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

		const tsvd_float beta = 0;

		int m = C.rows();
		int n = C.columns();
		int k = transpose_a ? A.rows() : A.columns();
		int lda = transpose_a ? k : m;
		int ldb = transpose_b ? n : k;
		int ldc = m;

		safe_cublas(hipblasSgemm(context.cublas_handle, op_a, op_b, m, n, k, &alpha, A.data(), lda, B.data(), ldb, &beta, C.data(), ldc));
	}

	void multiply(const Matrix<tsvd_double>& A, const Matrix<tsvd_double>& B, Matrix<tsvd_double>& C, DeviceContext& context, bool transpose_a, bool transpose_b, tsvd_double alpha)
	{
		hipblasOperation_t op_a = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		hipblasOperation_t op_b = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

		const tsvd_double beta = 0;

		int m = C.rows();
		int n = C.columns();
		int k = transpose_a ? A.rows() : A.columns();
		int lda = transpose_a ? k : m;
		int ldb = transpose_b ? n : k;
		int ldc = m;

		safe_cublas(hipblasDgemm(context.cublas_handle, op_a, op_b, m, n, k, &alpha, A.data(), lda, B.data(), ldb, &beta, C.data(), ldc));
	}

	void transpose(const Matrix<tsvd_float>& A, Matrix<tsvd_float>& B, DeviceContext& context)
	{
		tsvd_check(A.rows() == B.columns()&&A.columns() == B.rows(), "Transpose dimensions incorrect");
		const tsvd_float alpha = 1.0f;
		const tsvd_float beta = 0.0f;
		safe_cublas(hipblasSgeam(context.cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, B.rows(), B.columns(), &alpha, A.data(), A.rows(), &beta, NULL, B.rows(), B.data(), B.rows()));
	}

	void transpose(const Matrix<tsvd_double>& A, Matrix<tsvd_double>& B, DeviceContext& context)
	{
		tsvd_check(A.rows() == B.columns()&&A.columns() == B.rows(), "Transpose dimensions incorrect");
		const tsvd_double alpha = 1.0f;
		const tsvd_double beta = 0.0f;
		safe_cublas(hipblasDgeam(context.cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, B.rows(), B.columns(), &alpha, A.data(), A.rows(), &beta, NULL, B.rows(), B.data(), B.rows()));
	}

	void normalize_columns(Matrix<tsvd_float>& M, Matrix<tsvd_float>& M_temp, Matrix<tsvd_float>& column_length, const Matrix<tsvd_float>& ones, DeviceContext& context)
	{
		thrust::transform(M.dptr(), M.dptr() + M.size(), M_temp.dptr(), sqr_op());
		auto d_column_length = column_length.data();
		auto d_ones = ones.data();
		const tsvd_float alpha = 1.0f;
		const tsvd_float beta = 0.0f;
		safe_cublas(hipblasSgemv(context.cublas_handle, HIPBLAS_OP_T, M.rows(), M.columns(), &alpha, M_temp.data(), M.rows(), d_ones, 1, &beta, d_column_length, 1));

		thrust::transform(column_length.dptr(), column_length.dptr() + column_length.size(), column_length.dptr(), [=]__device__(tsvd_float val)
		                  {
							  if (val == 0.0)
							  {
								  return 0.0;
							  }

			                  return 1.0/ sqrt(val);
		                  });

		safe_cublas(hipblasSdgmm(context.cublas_handle, HIPBLAS_SIDE_RIGHT, M.rows(), M.columns(), M.data(), M.rows(), d_column_length, 1, M.data(), M.rows()));
	}

	void normalize_columns(Matrix<tsvd_double>& M, Matrix<tsvd_double>& M_temp, Matrix<tsvd_double>& column_length, const Matrix<tsvd_double>& ones, DeviceContext& context)
	{
		thrust::transform(M.dptr(), M.dptr() + M.size(), M_temp.dptr(), sqr_op());
		auto d_column_length = column_length.data();
		auto d_ones = ones.data();
		const tsvd_double alpha = 1.0f;
		const tsvd_double beta = 0.0f;
		safe_cublas(hipblasDgemv(context.cublas_handle, HIPBLAS_OP_T, M.rows(), M.columns(), &alpha, M_temp.data(), M.rows(), d_ones, 1, &beta, d_column_length, 1));

		thrust::transform(column_length.dptr(), column_length.dptr() + column_length.size(), column_length.dptr(), [=]__device__(tsvd_double val)
		                  {
							  if (val == 0.0)
							  {
								  return 0.0;
							  }

			                  return 1.0/ sqrt(val);
		                  });

		safe_cublas(hipblasDdgmm(context.cublas_handle, HIPBLAS_SIDE_RIGHT, M.rows(), M.columns(), M.data(), M.rows(), d_column_length, 1, M.data(), M.rows()));
	}

	void normalize_columns(Matrix<tsvd_float>& M, DeviceContext& context)
	{
		Matrix<float> M_temp(M.rows(), M.columns());
		Matrix<float> columns_length(1, M.columns());
		Matrix<float> ones(1, M.columns());
		ones.fill(1.0f);
		normalize_columns(M, M_temp, columns_length, ones, context);
	}

	void normalize_columns(Matrix<tsvd_double>& M, DeviceContext& context)
	{
		Matrix<double> M_temp(M.rows(), M.columns());
		Matrix<double> columns_length(1, M.columns());
		Matrix<double> ones(1, M.columns());
		ones.fill(1.0f);
		normalize_columns(M, M_temp, columns_length, ones, context);
	}

	void normalize_vector_cublas(Matrix<tsvd_float>& M, DeviceContext& context){
        float norm2 = 0.0;
        safe_cublas(hipblasSnrm2(context.cublas_handle, M.rows(), M.data(), 1.0, &norm2));
        M.transform([=]__device__ (float val){return val * (1/norm2);});
    }

	void normalize_vector_cublas(Matrix<tsvd_double>& M, DeviceContext& context){
        double norm2 = 0.0;
        safe_cublas(hipblasDnrm2(context.cublas_handle, M.rows(), M.data(), 1.0, &norm2));
        M.transform([=]__device__ (float val){return val * (1/norm2);});
    }

	void residual(const Matrix<tsvd_float>& X, const Matrix<tsvd_float>& D, const Matrix<tsvd_float>& S, Matrix<tsvd_float>& R, DeviceContext& context)
	{
		multiply(D, S, R, context);
		subtract(X, R, R, context);
	}

	void residual(const Matrix<tsvd_double>& X, const Matrix<tsvd_double>& D, const Matrix<tsvd_double>& S, Matrix<tsvd_double>& R, DeviceContext& context)
	{
		multiply(D, S, R, context);
		subtract(X, R, R, context);
	}

	void calculate_eigen_pairs_exact(const Matrix<tsvd_float>& X, Matrix<tsvd_float>& Q, Matrix<tsvd_float>& w, DeviceContext& context)
	{
		tsvd_check(X.rows() == X.columns(), "X must be a symmetric matrix");
		tsvd_check(X.rows() == Q.rows() && X.columns() == Q.columns(), "X and Q must have the same dimension");
		tsvd_check(w.rows() == Q.columns(), "Q and w should have the same number of columns");

		int lwork;
		safe_cusolver(hipsolverDnSsyevd_bufferSize(context.cusolver_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, X.rows(), X.data(), X.columns(), w.data(), &lwork));

		float *d_work;
		safe_cuda(hipMalloc(&d_work, sizeof(float) * lwork));

		int *dev_info = NULL;
		safe_cuda(hipMalloc ((void**)&dev_info, sizeof(int)));
		Q.copy(X);
		safe_cusolver(hipsolverDnSsyevd(context.cusolver_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, Q.rows(), Q.data(), Q.columns(), w.data(), d_work, lwork, dev_info));
		safe_cuda(hipDeviceSynchronize());
		safe_cuda(hipFree(d_work));
		safe_cuda(hipFree(dev_info));
		safe_cuda(hipGetLastError());
	}

	void calculate_eigen_pairs_exact(const Matrix<tsvd_double>& X, Matrix<tsvd_double>& Q, Matrix<tsvd_double>& w, DeviceContext& context)
	{
		tsvd_check(X.rows() == X.columns(), "X must be a symmetric matrix");
		tsvd_check(X.rows() == Q.rows() && X.columns() == Q.columns(), "X and Q must have the same dimension");
		tsvd_check(w.rows() == Q.columns(), "Q and w should have the same number of columns");

		int lwork;
		safe_cusolver(hipsolverDnDsyevd_bufferSize(context.cusolver_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, X.rows(), X.data(), X.columns(), w.data(), &lwork));

		double *d_work;
		safe_cuda(hipMalloc(&d_work, sizeof(double) * lwork));

		int *dev_info = NULL;
		safe_cuda(hipMalloc ((void**)&dev_info, sizeof(int)));
		Q.copy(X);
		safe_cusolver(hipsolverDnDsyevd(context.cusolver_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, Q.rows(), Q.data(), Q.columns(), w.data(), d_work, lwork, dev_info));
		safe_cuda(hipDeviceSynchronize());
		safe_cuda(hipFree(d_work));
		safe_cuda(hipFree(dev_info));
		safe_cuda(hipGetLastError());
	}

	void dot_product(Matrix<tsvd_float>& b_k1, Matrix<tsvd_float>& b_k, float* eigen_value_estimate, DeviceContext& context)
	{
		safe_cublas(hipblasSdot(context.cublas_handle, b_k1.rows(), b_k1.data(), 1.0, b_k.data(), 1.0, eigen_value_estimate));
	}

	void dot_product(Matrix<tsvd_double>& b_k1, Matrix<tsvd_double>& b_k, double* eigen_value_estimate, DeviceContext& context)
	{
		safe_cublas(hipblasDdot(context.cublas_handle, b_k1.rows(), b_k1.data(), 1.0, b_k.data(), 1.0, eigen_value_estimate));
	}

	//----------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//Stricly floating point operations that are not used
	void linear_solve(const Matrix<tsvd_float>& A, Matrix<tsvd_float>& X, const Matrix<tsvd_float>& B, DeviceContext& context)
	{
		tsvd_check(A.rows()>= A.columns(),"Linear solve requires m >= n");
		tsvd_check(X.rows()>= X.columns(),"Linear solve requires n >= k"); //TODO: is this restriction necessary?

		Matrix<tsvd_float> A_copy(A);
		Matrix<tsvd_float> B_copy(A.rows(), A.columns());
		thrust::copy(B.dptr(), B.dptr() + B.size(), B_copy.dptr());
		thrust::fill(B_copy.dptr() + B.size(), B_copy.dptr() + B_copy.size(), 0.0f);

		int work_size = 0;
		safe_cusolver(hipsolverDnSgeqrf_bufferSize(context.cusolver_handle, A_copy.rows(), A_copy.columns(), A_copy.data(), A_copy.rows(), &work_size));

		thrust::device_vector<tsvd_float> work(work_size);
		tsvd_float* d_work = thrust::raw_pointer_cast(work.data());

		thrust::device_vector<tsvd_float> tau((std::min)(A.rows(), A.columns()));
		tsvd_float* d_tau = thrust::raw_pointer_cast(tau.data());

		thrust::device_vector<int> dev_info(1);
		int* d_dev_info = thrust::raw_pointer_cast(dev_info.data());

		safe_cusolver(hipsolverDnSgeqrf(context.cusolver_handle, A_copy.rows(), A_copy.columns(), A_copy.data(), A_copy.rows(), d_tau, d_work, work_size, d_dev_info));

		tsvd_check(dev_info[0] == 0, "geqrf unsuccessful");

		safe_cusolver(hipsolverDnSormqr(context.cusolver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, A.rows(), A.columns(), (std::min)(A.rows(), A.columns()), A_copy.data(), A.rows(), d_tau, B_copy.data(), A.rows(), d_work, work_size, d_dev_info));
		tsvd_check(dev_info[0] == 0, "ormqr unsuccessful");

		Matrix<tsvd_float> R(A.columns(), A.columns());
		Matrix<tsvd_float> QTB(A.columns(), B.columns());
		auto counting = thrust::make_counting_iterator(0);
		int n = R.columns();
		int m = A.rows();
		auto d_R = R.data();
		auto d_A_copy = A_copy.data();
		auto d_QTB = QTB.data();
		auto d_B_copy = B_copy.data();
		int qtb_columns = QTB.columns();
		thrust::for_each(counting, counting + R.size(), [=]__device__ (int idx)
		                 {
			                 int row = idx % n;
			                 int column = idx / n;
			                 d_R[idx] = d_A_copy[column * m + row];

			                 if (column < qtb_columns)
			                 {
				                 d_QTB[idx] = d_B_copy[column * m + row];
			                 }
		                 });

		const tsvd_float alpha = 1.0f;
		safe_cublas(hipblasStrsm(context.cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, QTB.rows(), QTB.columns(), &alpha, R.data(), R.rows(), QTB.data(), QTB.rows()));

		thrust::copy(QTB.dptr(), QTB.dptr() + QTB.size(), X.data());
	}

	void pseudoinverse(const Matrix<tsvd_float>& A, Matrix<tsvd_float>& pinvA, DeviceContext& context)
	{
		tsvd_check(A.rows() == pinvA.columns() && A.columns() == pinvA.rows(), "pseudoinverse dimensions incorrect");

		//Add zero rows if m < n such that m >= n
		Matrix<tsvd_float> A_extended((std::max)(A.columns(), A.rows()), A.columns());
		auto counting = thrust::make_counting_iterator(0);
		int A_column_size = A.rows();
		int A_extended_column_size = A_extended.rows();
		auto d_A = A.data();
		auto d_A_extended = A_extended.data();

		thrust::for_each(counting, counting + A_extended.size(), [=]__device__(int idx)
		                 {
			                 int row = idx % A_extended_column_size;

			                 if (row < A_column_size)
			                 {
				                 int column = idx / A_extended_column_size;
				                 d_A_extended[idx] = d_A[A_column_size * column + row];
			                 }
			                 else
			                 {
				                 d_A_extended[idx] = 0;
			                 }
		                 });

		int work_size = 0;
		safe_cusolver(hipsolverDnSgesvd_bufferSize(context.cusolver_handle, A_extended.rows(), A_extended.columns(), &work_size));

		Matrix<tsvd_float> work(work_size, 1);

		Matrix<tsvd_float> S((std::min)(A_extended.rows(), A_extended.columns()), 1);
		Matrix<tsvd_float> U(A_extended.rows(), A_extended.rows());
		Matrix<tsvd_float> VT(A_extended.columns(), A_extended.columns());
		Matrix<int> dev_info(1, 1);

		safe_cusolver (hipsolverDnSgesvd(context.cusolver_handle, 'A', 'A', A_extended.rows(), A_extended.columns(), d_A_extended, A_extended.rows(), S.data(), U.data(), U.rows(), VT.data(), VT.rows(), work.data(), work_size, NULL, dev_info.data()));

		tsvd_float eps = 1e-5;
		thrust::transform(S.dptr(), S.dptr() + S.size(), S.dptr(), [=]__device__(tsvd_float val)
		                  {
			                  if (abs(val) < eps)
			                  {
				                  return 0.0;
			                  }
			                  else
			                  {
				                  return 1.0 / val;
			                  }
		                  });

		Matrix<tsvd_float> UT(A_extended.rows(), A_extended.rows());

		//Calculate transpose of U
		const tsvd_float alpha = 1.0;
		const tsvd_float beta = 0.0;
		safe_cublas(hipblasSgeam(context.cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, UT.rows(), UT.columns(), &alpha, U.data(), UT.rows(), &beta,NULL, UT.rows(), UT.data(), UT.rows()));

		safe_cublas(hipblasSdgmm(context.cublas_handle, HIPBLAS_SIDE_LEFT, UT.rows(), UT.columns(), UT.data(), UT.rows(), S.data(), 1, U.data(), U.rows()));

		Matrix<tsvd_float> pinvA_extended(A_extended.columns(), A_extended.rows());
		multiply(VT, U, pinvA_extended, context, true);

		thrust::copy(pinvA_extended.dptr(), pinvA_extended.dptr() + pinvA.size(), pinvA.dptr());
	}

	void f_normalize(Matrix<tsvd_float>& M, DeviceContext& context)
	{
		Matrix<tsvd_float> temp(M.rows(), M.columns());
		thrust::transform(M.dptr(), M.dptr() + M.size(), temp.dptr(), sqr_op());
		tsvd_float sum = thrust::reduce(temp.dptr(), temp.dptr() + temp.size());
		multiply(M, 1.0 / std::sqrt(sum), context);
		thrust::transform(M.dptr(), M.dptr() + M.size(), temp.dptr(), sqr_op());
		tsvd_float final_sum = thrust::reduce(temp.dptr(), temp.dptr() + temp.size());
		printf("f norm sum squares: %1.4f\n", final_sum);
	}

	void normalize_columns_cub(Matrix<tsvd_float>& M, DeviceContext& context)
	{
		//Create alias so device Lamba does not dereference this pointer
		int m = M.rows();

		thrust::device_vector<tsvd_float> temp(M.size());
		thrust::device_vector<tsvd_float> length_squared(M.columns());

		thrust::transform(M.dptr(), M.dptr() + M.size(), temp.begin(), [=]__device__(tsvd_float val)
		                  {
			                  return val * val;
		                  });


		thrust::device_vector<int> column_segments(M.columns() + 1);
		auto counting = thrust::make_counting_iterator(0);
		thrust::transform(counting, counting + column_segments.size(), column_segments.begin(), [=]__device__(int idx)
		                  {
			                  return idx * m;
		                  });

		// Determine temporary device storage requirements
		void* d_temp_storage = NULL;
		size_t temp_storage_bytes = 0;
		auto segments = thrust::raw_pointer_cast(column_segments.data());
		auto sum_in = thrust::raw_pointer_cast(temp.data());
		auto sum_out = thrust::raw_pointer_cast(length_squared.data());
		hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, sum_in, sum_out,
		                                M.columns(), segments, segments + 1);
		// Allocate temporary storage
		hipMalloc(&d_temp_storage, temp_storage_bytes);
		hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, sum_in, sum_out,
		                                M.columns(), segments, segments + 1);

		//Scale
		auto d_length_squared = thrust::raw_pointer_cast(length_squared.data());
		auto d_data = M.data();
		thrust::transform(counting, counting + M.size(), M.dptr(), [=]__device__(int idx)
		                  {
			                  int col = idx / m;

			                  tsvd_float length_squared = d_length_squared[col];

			                  if (length_squared > 0.0)
			                  {
				                  return d_data[idx] / std::sqrt(d_length_squared[col]);
			                  }
			                  else
			                  {
				                  return 0.0f;
			                  }
		                  });

		hipFree(d_temp_storage);
	}

}

//Orignal Impl
template void tsvd::multiply<double>(Matrix<double>& A, const float a, DeviceContext& context);

//Impl for floats and doubles
template void tsvd::multiply<float>(Matrix<float>& A, const float a, DeviceContext& context);
template void tsvd::multiply<double>(Matrix<double>& A, const double a, DeviceContext& context);
template void tsvd::subtract<float>(const Matrix<float>& A, const Matrix<float>& B, Matrix<float>& C, DeviceContext& context);
template void tsvd::subtract<double>(const Matrix<double>& A, const Matrix<double>& B, Matrix<double>& C, DeviceContext& context);
template void tsvd::add<float>(const Matrix<float>& A, const Matrix<float>& B, Matrix<float>& C, DeviceContext& context);
template void tsvd::add<double>(const Matrix<double>& A, const Matrix<double>& B, Matrix<double>& C, DeviceContext& context);
template void tsvd::normalize_vector_thrust<float>(Matrix<float>& M, DeviceContext& context);
template void tsvd::normalize_vector_thrust<double>(Matrix<double>& M, DeviceContext& context);

