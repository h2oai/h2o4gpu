#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <algorithm>
#include <thrust/inner_product.h>
#include <thrust/extrema.h>

namespace matrix
{
	using namespace h2o4gpu;

	void max_index_per_column(Matrix<float>& A, std::vector<int>& result_array, device::DeviceContext& context){

		int result;
		for (int i=0; i<A.columns(); i++) {
			safe_cublas(hipblasIsamax(context.cublas_handle, A.rows(), A.data() + i*A.rows(), 1, &result));
			result_array[i] = result - 1 + i * A.rows();
		}
	}

	void max_index_per_column(Matrix<double>& A, std::vector<int>& result_array, device::DeviceContext& context){

		int result;
		for (int i=0; i<A.columns(); i++) {
			safe_cublas(hipblasIdamax(context.cublas_handle, A.rows(), A.data() + i*A.rows(), 1, &result));
			result_array[i] = result - 1 + i * A.rows();
		}
	}

	template<typename T, typename U>
	void multiply(Matrix<T>& A, const U a, device::DeviceContext& context)
	{
		thrust::transform(A.dptr(), A.dptr() + A.size(), A.dptr(), [=]__device__ (U val)
						  {
							  return val * a;
						  }
		);
	}

	template<typename T>
	void subtract(const Matrix<T>& A, const Matrix<T>& B, Matrix<T>& C, device::DeviceContext& context)
	{
		auto counting = thrust::make_counting_iterator(0);
		const T* d_A = A.data();
		const T* d_B = B.data();
		T* d_C = C.data();
		thrust::for_each(counting, counting + A.rows() * A.columns(), [=]__device__(int idx)
						 {
							 d_C[idx] = d_A[idx] - d_B[idx];
						 });
	}

	template<typename T>
	void add(const Matrix<T>& A, const Matrix<T>& B, Matrix<T>& C, device::DeviceContext& context)
	{
		auto counting = thrust::make_counting_iterator(0);
		const T* d_A = A.data();
		const T* d_B = B.data();
		T* d_C = C.data();
		thrust::for_each(counting, counting + A.rows() * A.columns(), [=]__device__(int idx)
						 {
							 d_C[idx] = d_A[idx] + d_B[idx];
						 });
	}


	template<typename T>
	void normalize_vector_thrust(Matrix<T>& M, device::DeviceContext& context){
		float M_inner = thrust::inner_product(M.dptr(), M.dptr() + M.size(), M.dptr(), 0.0f); //Will allocate memory for every call to fxn.
		M.transform([=]__device__ (float val){return val / std::sqrt(M_inner);});
	}

	void multiply_diag(const Matrix<float>& A, const Matrix<float>& B, Matrix<float>& C, device::DeviceContext& context, bool left_diag)
	{
		hipblasSideMode_t mode = left_diag ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT;

		int m = C.rows();
		int n = C.columns();
		int lda = m;
		int incx = 1; //Review what this should be...
		int ldc = m;

		safe_cublas(hipblasSdgmm(context.cublas_handle, mode, m, n, A.data(), lda, B.data(), incx, C.data(), ldc));
	}

	void multiply_diag(const Matrix<double>& A, const Matrix<double>& B, Matrix<double>& C, device::DeviceContext& context, bool left_diag)
	{
		hipblasSideMode_t mode = left_diag ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT;

		int m = C.rows();
		int n = C.columns();
		int lda = m;
		int incx = 1; //Review what this should be...
		int ldc = m;

		safe_cublas(hipblasDdgmm(context.cublas_handle, mode, m, n, A.data(), lda, B.data(), incx, C.data(), ldc));
	}

	void multiply(const Matrix<float>& A, const Matrix<float>& B, Matrix<float>& C, device::DeviceContext& context, bool transpose_a, bool transpose_b, float alpha)
	{
		hipblasOperation_t op_a = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		hipblasOperation_t op_b = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

		const float beta = 0;

		int m = C.rows();
		int n = C.columns();
		int k = transpose_a ? A.rows() : A.columns();
		int lda = transpose_a ? k : m;
		int ldb = transpose_b ? n : k;
		int ldc = m;

		safe_cublas(hipblasSgemm(context.cublas_handle, op_a, op_b, m, n, k, &alpha, A.data(), lda, B.data(), ldb, &beta, C.data(), ldc));
	}

	void multiply(const Matrix<double>& A, const Matrix<double>& B, Matrix<double>& C, device::DeviceContext& context, bool transpose_a, bool transpose_b, double alpha)
	{
		hipblasOperation_t op_a = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		hipblasOperation_t op_b = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

		const double beta = 0;

		int m = C.rows();
		int n = C.columns();
		int k = transpose_a ? A.rows() : A.columns();
		int lda = transpose_a ? k : m;
		int ldb = transpose_b ? n : k;
		int ldc = m;

		safe_cublas(hipblasDgemm(context.cublas_handle, op_a, op_b, m, n, k, &alpha, A.data(), lda, B.data(), ldb, &beta, C.data(), ldc));
	}

	void transpose(const Matrix<float>& A, Matrix<float>& B, device::DeviceContext& context)
	{
		h2o4gpu_check(A.rows() == B.columns()&&A.columns() == B.rows(), "Transpose dimensions incorrect");
		const float alpha = 1.0f;
		const float beta = 0.0f;
		safe_cublas(hipblasSgeam(context.cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, B.rows(), B.columns(), &alpha, A.data(), A.rows(), &beta, NULL, B.rows(), B.data(), B.rows()));
	}

	void transpose(const Matrix<double>& A, Matrix<double>& B, device::DeviceContext& context)
	{
		h2o4gpu_check(A.rows() == B.columns()&&A.columns() == B.rows(), "Transpose dimensions incorrect");
		const double alpha = 1.0f;
		const double beta = 0.0f;
		safe_cublas(hipblasDgeam(context.cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, B.rows(), B.columns(), &alpha, A.data(), A.rows(), &beta, NULL, B.rows(), B.data(), B.rows()));
	}

	void normalize_columns(Matrix<float>& M, Matrix<float>& M_temp, Matrix<float>& column_length, const Matrix<float>& ones, device::DeviceContext& context)
	{
		thrust::transform(M.dptr(), M.dptr() + M.size(), M_temp.dptr(), sqr_op());
		auto d_column_length = column_length.data();
		auto d_ones = ones.data();
		const float alpha = 1.0f;
		const float beta = 0.0f;
		safe_cublas(hipblasSgemv(context.cublas_handle, HIPBLAS_OP_T, M.rows(), M.columns(), &alpha, M_temp.data(), M.rows(), d_ones, 1, &beta, d_column_length, 1));

		thrust::transform(column_length.dptr(), column_length.dptr() + column_length.size(), column_length.dptr(), [=]__device__(float val)
		                  {
							  if (val == 0.0)
							  {
								  return 0.0;
							  }

			                  return 1.0/ sqrt(val);
		                  });

		safe_cublas(hipblasSdgmm(context.cublas_handle, HIPBLAS_SIDE_RIGHT, M.rows(), M.columns(), M.data(), M.rows(), d_column_length, 1, M.data(), M.rows()));
	}

	void normalize_columns(Matrix<double>& M, Matrix<double>& M_temp, Matrix<double>& column_length, const Matrix<double>& ones, device::DeviceContext& context)
	{
		thrust::transform(M.dptr(), M.dptr() + M.size(), M_temp.dptr(), sqr_op());
		auto d_column_length = column_length.data();
		auto d_ones = ones.data();
		const double alpha = 1.0f;
		const double beta = 0.0f;
		safe_cublas(hipblasDgemv(context.cublas_handle, HIPBLAS_OP_T, M.rows(), M.columns(), &alpha, M_temp.data(), M.rows(), d_ones, 1, &beta, d_column_length, 1));

		thrust::transform(column_length.dptr(), column_length.dptr() + column_length.size(), column_length.dptr(), [=]__device__(double val)
		                  {
							  if (val == 0.0)
							  {
								  return 0.0;
							  }

			                  return 1.0/ sqrt(val);
		                  });

		safe_cublas(hipblasDdgmm(context.cublas_handle, HIPBLAS_SIDE_RIGHT, M.rows(), M.columns(), M.data(), M.rows(), d_column_length, 1, M.data(), M.rows()));
	}

	void normalize_columns(Matrix<float>& M, device::DeviceContext& context)
	{
		Matrix<float> M_temp(M.rows(), M.columns());
		Matrix<float> columns_length(1, M.columns());
		Matrix<float> ones(1, M.columns());
		ones.fill(1.0f);
		normalize_columns(M, M_temp, columns_length, ones, context);
	}

	void normalize_columns(Matrix<double>& M, device::DeviceContext& context)
	{
		Matrix<double> M_temp(M.rows(), M.columns());
		Matrix<double> columns_length(1, M.columns());
		Matrix<double> ones(1, M.columns());
		ones.fill(1.0f);
		normalize_columns(M, M_temp, columns_length, ones, context);
	}

	void normalize_vector_cublas(Matrix<float>& M, device::DeviceContext& context){
        float norm2 = 0.0;
        safe_cublas(hipblasSnrm2(context.cublas_handle, M.rows(), M.data(), 1.0, &norm2));
        M.transform([=]__device__ (float val){return val * (1/norm2);});
    }

	void normalize_vector_cublas(Matrix<double>& M, device::DeviceContext& context){
        double norm2 = 0.0;
        safe_cublas(hipblasDnrm2(context.cublas_handle, M.rows(), M.data(), 1.0, &norm2));
        M.transform([=]__device__ (float val){return val * (1/norm2);});
    }

	void residual(const Matrix<float>& X, const Matrix<float>& D, const Matrix<float>& S, Matrix<float>& R, device::DeviceContext& context)
	{
		multiply(D, S, R, context);
		subtract(X, R, R, context);
	}

	void residual(const Matrix<double>& X, const Matrix<double>& D, const Matrix<double>& S, Matrix<double>& R, device::DeviceContext& context)
	{
		multiply(D, S, R, context);
		subtract(X, R, R, context);
	}

	void calculate_eigen_pairs_exact(const Matrix<float>& X, Matrix<float>& Q, Matrix<float>& w, device::DeviceContext& context)
	{
		h2o4gpu_check(X.rows() == X.columns(), "X must be a symmetric matrix");
		h2o4gpu_check(X.rows() == Q.rows() && X.columns() == Q.columns(), "X and Q must have the same dimension");
		h2o4gpu_check(w.rows() == Q.columns(), "Q and w should have the same number of columns");

		int lwork;
		safe_cusolver(hipsolverDnSsyevd_bufferSize(context.cusolver_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, X.rows(), X.data(), X.columns(), w.data(), &lwork));

		float *d_work;
		safe_cuda(hipMalloc(&d_work, sizeof(float) * lwork));

		int *dev_info = NULL;
		safe_cuda(hipMalloc ((void**)&dev_info, sizeof(int)));
		Q.copy(X);
		safe_cusolver(hipsolverDnSsyevd(context.cusolver_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, Q.rows(), Q.data(), Q.columns(), w.data(), d_work, lwork, dev_info));
		safe_cuda(hipDeviceSynchronize());
		safe_cuda(hipFree(d_work));
		safe_cuda(hipFree(dev_info));
		safe_cuda(hipGetLastError());
	}

	void calculate_eigen_pairs_exact(const Matrix<double>& X, Matrix<double>& Q, Matrix<double>& w, device::DeviceContext& context)
	{
		h2o4gpu_check(X.rows() == X.columns(), "X must be a symmetric matrix");
		h2o4gpu_check(X.rows() == Q.rows() && X.columns() == Q.columns(), "X and Q must have the same dimension");
		h2o4gpu_check(w.rows() == Q.columns(), "Q and w should have the same number of columns");

		int lwork;
		safe_cusolver(hipsolverDnDsyevd_bufferSize(context.cusolver_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, X.rows(), X.data(), X.columns(), w.data(), &lwork));

		double *d_work;
		safe_cuda(hipMalloc(&d_work, sizeof(double) * lwork));

		int *dev_info = NULL;
		safe_cuda(hipMalloc ((void**)&dev_info, sizeof(int)));
		Q.copy(X);
		safe_cusolver(hipsolverDnDsyevd(context.cusolver_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, Q.rows(), Q.data(), Q.columns(), w.data(), d_work, lwork, dev_info));
		safe_cuda(hipDeviceSynchronize());
		safe_cuda(hipFree(d_work));
		safe_cuda(hipFree(dev_info));
		safe_cuda(hipGetLastError());
	}

	void dot_product(Matrix<float>& b_k1, Matrix<float>& b_k, float* eigen_value_estimate, device::DeviceContext& context)
	{
		safe_cublas(hipblasSdot(context.cublas_handle, b_k1.rows(), b_k1.data(), 1.0, b_k.data(), 1.0, eigen_value_estimate));
	}

	void dot_product(Matrix<double>& b_k1, Matrix<double>& b_k, double* eigen_value_estimate, device::DeviceContext& context)
	{
		safe_cublas(hipblasDdot(context.cublas_handle, b_k1.rows(), b_k1.data(), 1.0, b_k.data(), 1.0, eigen_value_estimate));
	}

	//----------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//Stricly floating point operations that are not used
	void linear_solve(const Matrix<float>& A, Matrix<float>& X, const Matrix<float>& B, device::DeviceContext& context)
	{
		h2o4gpu_check(A.rows()>= A.columns(),"Linear solve requires m >= n");
		h2o4gpu_check(X.rows()>= X.columns(),"Linear solve requires n >= k"); //TODO: is this restriction necessary?

		Matrix<float> A_copy(A);
		Matrix<float> B_copy(A.rows(), A.columns());
		thrust::copy(B.dptr(), B.dptr() + B.size(), B_copy.dptr());
		thrust::fill(B_copy.dptr() + B.size(), B_copy.dptr() + B_copy.size(), 0.0f);

		int work_size = 0;
		safe_cusolver(hipsolverDnSgeqrf_bufferSize(context.cusolver_handle, A_copy.rows(), A_copy.columns(), A_copy.data(), A_copy.rows(), &work_size));

		thrust::device_vector<float> work(work_size);
		float* d_work = thrust::raw_pointer_cast(work.data());

		thrust::device_vector<float> tau((std::min)(A.rows(), A.columns()));
		float* d_tau = thrust::raw_pointer_cast(tau.data());

		thrust::device_vector<int> dev_info(1);
		int* d_dev_info = thrust::raw_pointer_cast(dev_info.data());

		safe_cusolver(hipsolverDnSgeqrf(context.cusolver_handle, A_copy.rows(), A_copy.columns(), A_copy.data(), A_copy.rows(), d_tau, d_work, work_size, d_dev_info));

		h2o4gpu_check(dev_info[0] == 0, "geqrf unsuccessful");

		safe_cusolver(hipsolverDnSormqr(context.cusolver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, A.rows(), A.columns(), (std::min)(A.rows(), A.columns()), A_copy.data(), A.rows(), d_tau, B_copy.data(), A.rows(), d_work, work_size, d_dev_info));
		h2o4gpu_check(dev_info[0] == 0, "ormqr unsuccessful");

		Matrix<float> R(A.columns(), A.columns());
		Matrix<float> QTB(A.columns(), B.columns());
		auto counting = thrust::make_counting_iterator(0);
		int n = R.columns();
		int m = A.rows();
		auto d_R = R.data();
		auto d_A_copy = A_copy.data();
		auto d_QTB = QTB.data();
		auto d_B_copy = B_copy.data();
		int qtb_columns = QTB.columns();
		thrust::for_each(counting, counting + R.size(), [=]__device__ (int idx)
		                 {
			                 int row = idx % n;
			                 int column = idx / n;
			                 d_R[idx] = d_A_copy[column * m + row];

			                 if (column < qtb_columns)
			                 {
				                 d_QTB[idx] = d_B_copy[column * m + row];
			                 }
		                 });

		const float alpha = 1.0f;
		safe_cublas(hipblasStrsm(context.cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, QTB.rows(), QTB.columns(), &alpha, R.data(), R.rows(), QTB.data(), QTB.rows()));

		thrust::copy(QTB.dptr(), QTB.dptr() + QTB.size(), X.data());
	}

	void pseudoinverse(const Matrix<float>& A, Matrix<float>& pinvA, device::DeviceContext& context)
	{
		h2o4gpu_check(A.rows() == pinvA.columns() && A.columns() == pinvA.rows(), "pseudoinverse dimensions incorrect");

		//Add zero rows if m < n such that m >= n
		Matrix<float> A_extended((std::max)(A.columns(), A.rows()), A.columns());
		auto counting = thrust::make_counting_iterator(0);
		int A_column_size = A.rows();
		int A_extended_column_size = A_extended.rows();
		auto d_A = A.data();
		auto d_A_extended = A_extended.data();

		thrust::for_each(counting, counting + A_extended.size(), [=]__device__(int idx)
		                 {
			                 int row = idx % A_extended_column_size;

			                 if (row < A_column_size)
			                 {
				                 int column = idx / A_extended_column_size;
				                 d_A_extended[idx] = d_A[A_column_size * column + row];
			                 }
			                 else
			                 {
				                 d_A_extended[idx] = 0;
			                 }
		                 });

		int work_size = 0;
		safe_cusolver(hipsolverDnSgesvd_bufferSize(context.cusolver_handle, A_extended.rows(), A_extended.columns(), &work_size));

		Matrix<float> work(work_size, 1);

		Matrix<float> S((std::min)(A_extended.rows(), A_extended.columns()), 1);
		Matrix<float> U(A_extended.rows(), A_extended.rows());
		Matrix<float> VT(A_extended.columns(), A_extended.columns());
		Matrix<int> dev_info(1, 1);

		safe_cusolver (hipsolverDnSgesvd(context.cusolver_handle, 'A', 'A', A_extended.rows(), A_extended.columns(), d_A_extended, A_extended.rows(), S.data(), U.data(), U.rows(), VT.data(), VT.rows(), work.data(), work_size, NULL, dev_info.data()));

		float eps = 1e-5;
		thrust::transform(S.dptr(), S.dptr() + S.size(), S.dptr(), [=]__device__(float val)
		                  {
			                  if (abs(val) < eps)
			                  {
				                  return 0.0;
			                  }
			                  else
			                  {
				                  return 1.0 / val;
			                  }
		                  });

		Matrix<float> UT(A_extended.rows(), A_extended.rows());

		//Calculate transpose of U
		const float alpha = 1.0;
		const float beta = 0.0;
		safe_cublas(hipblasSgeam(context.cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, UT.rows(), UT.columns(), &alpha, U.data(), UT.rows(), &beta,NULL, UT.rows(), UT.data(), UT.rows()));

		safe_cublas(hipblasSdgmm(context.cublas_handle, HIPBLAS_SIDE_LEFT, UT.rows(), UT.columns(), UT.data(), UT.rows(), S.data(), 1, U.data(), U.rows()));

		Matrix<float> pinvA_extended(A_extended.columns(), A_extended.rows());
		multiply(VT, U, pinvA_extended, context, true);

		thrust::copy(pinvA_extended.dptr(), pinvA_extended.dptr() + pinvA.size(), pinvA.dptr());
	}

	void f_normalize(Matrix<float>& M, device::DeviceContext& context)
	{
		Matrix<float> temp(M.rows(), M.columns());
		thrust::transform(M.dptr(), M.dptr() + M.size(), temp.dptr(), sqr_op());
		float sum = thrust::reduce(temp.dptr(), temp.dptr() + temp.size());
		multiply(M, 1.0 / std::sqrt(sum), context);
		thrust::transform(M.dptr(), M.dptr() + M.size(), temp.dptr(), sqr_op());
		float final_sum = thrust::reduce(temp.dptr(), temp.dptr() + temp.size());
		printf("f norm sum squares: %1.4f\n", final_sum);
	}

	void normalize_columns_cub(Matrix<float>& M, device::DeviceContext& context)
	{
		//Create alias so device Lamba does not dereference this pointer
		int m = M.rows();

		thrust::device_vector<float> temp(M.size());
		thrust::device_vector<float> length_squared(M.columns());

		thrust::transform(M.dptr(), M.dptr() + M.size(), temp.begin(), [=]__device__(float val)
		                  {
			                  return val * val;
		                  });


		thrust::device_vector<int> column_segments(M.columns() + 1);
		auto counting = thrust::make_counting_iterator(0);
		thrust::transform(counting, counting + column_segments.size(), column_segments.begin(), [=]__device__(int idx)
		                  {
			                  return idx * m;
		                  });

		// Determine temporary device storage requirements
		void* d_temp_storage = NULL;
		size_t temp_storage_bytes = 0;
		auto segments = thrust::raw_pointer_cast(column_segments.data());
		auto sum_in = thrust::raw_pointer_cast(temp.data());
		auto sum_out = thrust::raw_pointer_cast(length_squared.data());
		hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, sum_in, sum_out,
		                                M.columns(), segments, segments + 1);
		// Allocate temporary storage
		hipMalloc(&d_temp_storage, temp_storage_bytes);
		hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, sum_in, sum_out,
		                                M.columns(), segments, segments + 1);

		//Scale
		auto d_length_squared = thrust::raw_pointer_cast(length_squared.data());
		auto d_data = M.data();
		thrust::transform(counting, counting + M.size(), M.dptr(), [=]__device__(int idx)
		                  {
			                  int col = idx / m;

			                  float length_squared = d_length_squared[col];

			                  if (length_squared > 0.0)
			                  {
				                  return d_data[idx] / std::sqrt(d_length_squared[col]);
			                  }
			                  else
			                  {
				                  return 0.0f;
			                  }
		                  });

		hipFree(d_temp_storage);
	}

}

//Orignal Impl
template void matrix::multiply<double>(Matrix<double>& A, const float a, device::DeviceContext& context);

//Impl for floats and doubles
template void matrix::multiply<float>(Matrix<float>& A, const float a, device::DeviceContext& context);
template void matrix::multiply<double>(Matrix<double>& A, const double a, device::DeviceContext& context);
template void matrix::subtract<float>(const Matrix<float>& A, const Matrix<float>& B, Matrix<float>& C, device::DeviceContext& context);
template void matrix::subtract<double>(const Matrix<double>& A, const Matrix<double>& B, Matrix<double>& C, device::DeviceContext& context);
template void matrix::add<float>(const Matrix<float>& A, const Matrix<float>& B, Matrix<float>& C, device::DeviceContext& context);
template void matrix::add<double>(const Matrix<double>& A, const Matrix<double>& B, Matrix<double>& C, device::DeviceContext& context);
template void matrix::normalize_vector_thrust<float>(Matrix<float>& M, device::DeviceContext& context);
template void matrix::normalize_vector_thrust<double>(Matrix<double>& M, device::DeviceContext& context);

