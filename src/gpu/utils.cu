/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Modifications Copyright 2017 H2O.ai, Inc.
 ************************************************************************/
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>
#include <iostream>
#include <assert.h>

#include <hip/hip_runtime_api.h>
#include "nvml.h"

#include "include/cuda_utils2.h"

int cudaresetdevice(int wDev, int nDev) {
  if (nDev > 0) {
    int nVis = 0;
    CUDACHECK(hipGetDeviceCount(&nVis));

    std::vector<int> dList(nDev);
    for (int i = 0; i < nDev; ++i) {
      dList[i] = i % nVis;
      CUDACHECK(hipSetDevice(dList[i]));
      CUDACHECK(hipDeviceReset()); // reset device dList[i]
    }
  }
  return (0);
}

int cudaresetdevice_bare(void) {
  hipDeviceReset();
    return(0);
}

int get_compute_capability(int d_idx, int *major, int *minor, int *ratioperf) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, d_idx);
  *major = prop.major;
  *minor = prop.minor;
  *ratioperf = prop.singleToDoublePrecisionPerfRatio;
  return (0);
}


void get_gpu_info_c(unsigned int *n_gpus, int *gpu_percent_usage, unsigned long long *gpu_total_memory, unsigned long long *gpu_free_memory, char **gpu_name, int *majors, int *minors) {

  nvmlReturn_t rv;
  rv = nvmlInit();
  assert(rv == NVML_SUCCESS);
  //unsigned int n_gpus;
  rv = nvmlDeviceGetCount(n_gpus);

  assert(rv == NVML_SUCCESS);

  //  int gpu_percent_usage[n_gpus];

  for (int i = 0; i < *n_gpus; ++i) {
    nvmlDevice_t device;
    nvmlReturn_t rv;
    rv = nvmlDeviceGetHandleByIndex(i, &device);
    assert(rv == NVML_SUCCESS);
    nvmlUtilization_t utilization;
    rv = nvmlDeviceGetUtilizationRates(device, &utilization);
    assert(rv == NVML_SUCCESS);
    gpu_percent_usage[i] = utilization.gpu;
    nvmlMemory_t memory;
    rv = nvmlDeviceGetMemoryInfo(device, &memory);
    assert(rv == NVML_SUCCESS);
    gpu_total_memory[i] = memory.total;
    gpu_free_memory[i] = memory.free;
    rv = nvmlDeviceGetName(device, gpu_name[i], 30);
    assert(rv == NVML_SUCCESS);
#if (CUDART_VERSION >= 9000)
    rv = nvmlDeviceGetCudaComputeCapability(device, &majors[i], &minors[i]);
    assert(rv == NVML_SUCCESS);
#else
    int ratioperf;
    get_compute_capability(i, &majors[i], &minors[i], &ratioperf);
#endif
  }

}
