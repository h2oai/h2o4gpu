#include "hip/hip_runtime.h"
/*!
 * Modifications Copyright 2017-2018 H2O.ai, Inc.
 */
#include <hipblas.h>
#include <hipsparse.h>

#include "cml/cml_spblas.cuh"
#include "cml/cml_spmat.cuh"
#include "cml/cml_vector.cuh"
#include "equil_helper.cuh"
#include "matrix/matrix.h"
#include "matrix/matrix_sparse.h"
#include "util.h"

namespace h2o4gpu {

////////////////////////////////////////////////////////////////////////////////
////////////////////////////// Helper Functions ////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
namespace {

// File scoped constants.
const NormTypes kNormEquilibrate = kNorm2; 
const NormTypes kNormNormalize   = kNormFro; 

template <typename T>
struct GpuData {
  const T *orig_data;
  const H2O4GPU_INT *orig_ptr, *orig_ind;
  hipblasHandle_t d_hdl;
  hipsparseHandle_t s_hdl;
  hipsparseMatDescr_t descr;
  GpuData(const T *data, const H2O4GPU_INT *ptr, const H2O4GPU_INT *ind)
      : orig_data(data), orig_ptr(ptr), orig_ind(ind) {
    hipblasCreate(&d_hdl);
    hipsparseCreate(&s_hdl);
    hipsparseCreateMatDescr(&descr);
    DEBUG_CUDA_CHECK_ERR();
  }
  ~GpuData() {
    hipblasDestroy(d_hdl);
    hipsparseDestroy(s_hdl);
    hipsparseDestroyMatDescr(descr);
    DEBUG_CUDA_CHECK_ERR();
  }
};

hipsparseOperation_t OpToCusparseOp(char trans) {
  ASSERT(trans == 'n' || trans == 'N' || trans == 't' || trans == 'T');
  return (trans == 'n' || trans == 'N')
      ? HIPSPARSE_OPERATION_NON_TRANSPOSE : HIPSPARSE_OPERATION_TRANSPOSE;
}

template <typename T>
void MultDiag(const T *d, const T *e, H2O4GPU_INT m, H2O4GPU_INT n, H2O4GPU_INT nnz,
              typename MatrixSparse<T>::Ord ord, T *data, const H2O4GPU_INT *ind,
              const H2O4GPU_INT *ptr);

template <typename T>
T NormEst(hipblasHandle_t hdl, NormTypes norm_type, const MatrixSparse<T>& A);

}  // namespace

////////////////////////////////////////////////////////////////////////////////
/////////////////////// MatrixDense Implementation /////////////////////////////
////////////////////////////////////////////////////////////////////////////////
template <typename T>
MatrixSparse<T>::MatrixSparse(int sharedA, int me, int wDev, char ord, H2O4GPU_INT m, H2O4GPU_INT n, H2O4GPU_INT nnz,
                              const T *data, const H2O4GPU_INT *ptr,
                              const H2O4GPU_INT *ind)
  : Matrix<T>(m, n), _wDev(wDev), _data(0), _de(0), _ptr(0), _ind(0), _nnz(nnz) {
  ASSERT(ord == 'r' || ord == 'R' || ord == 'c' || ord == 'C');
  _ord = (ord == 'r' || ord == 'R') ? ROW : COL;

  // It should work up to 2^31 == 2B, but let's be sure.
  DEBUG_EXPECT(nnz < static_cast<H2O4GPU_INT>(1 << 29));

  // Set GPU specific data.
  GpuData<T> *info = new GpuData<T>(data, ptr, ind);
  this->_info = reinterpret_cast<void*>(info);
}

template <typename T>
  MatrixSparse<T>::MatrixSparse(int wDev, char ord, H2O4GPU_INT m, H2O4GPU_INT n, H2O4GPU_INT nnz,
                              const T *data, const H2O4GPU_INT *ptr,
                              const H2O4GPU_INT *ind)
    : MatrixSparse<T>(0,0,wDev,ord,m,n,nnz,data,ptr,ind){}

template <typename T>
  MatrixSparse<T>::MatrixSparse(char ord, H2O4GPU_INT m, H2O4GPU_INT n, H2O4GPU_INT nnz,
                              const T *data, const H2O4GPU_INT *ptr,
                              const H2O4GPU_INT *ind)
    : MatrixSparse<T>(0,0,0,ord,m,n,nnz,data,ptr,ind){}



template <typename T>
MatrixSparse<T>::MatrixSparse(int sharedA, int me, int wDev, const MatrixSparse<T>& A)
  : Matrix<T>(A._m, A._n), _wDev(wDev), _data(0), _de(0), _ptr(0), _ind(0), _nnz(A._nnz), 
      _ord(A._ord) {

  GpuData<T> *info_A = reinterpret_cast<GpuData<T>*>(A._info);
  GpuData<T> *info = new GpuData<T>(info_A->orig_data, info_A->orig_ptr,
      info_A->orig_ind);
  this->_info = reinterpret_cast<void*>(info);
}

template <typename T>
  MatrixSparse<T>::MatrixSparse(int wDev, const MatrixSparse<T>& A)
    : MatrixSparse<T>(A._sharedA,A._me,wDev,A){}

template <typename T>
MatrixSparse<T>::MatrixSparse(const MatrixSparse<T>& A)
  : MatrixSparse<T>(A._wDev,A){}

template <typename T>
MatrixSparse<T>::~MatrixSparse() {
  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  delete info;
  this->_info = 0;

  if (this->_done_init) {
    if (_data) {
      hipFree(_data);
      _data = 0;
      DEBUG_CUDA_CHECK_ERR();
    }

    if (_de && !_sharedA) {
      hipFree(_de);
      _de = 0;
      DEBUG_CUDA_CHECK_ERR();
    }

    if (_ptr) {
      hipFree(_ptr);
      _ptr = 0;
      DEBUG_CUDA_CHECK_ERR();
    }

    if (_ind) {
      hipFree(_ind);
      _ind = 0;
      DEBUG_CUDA_CHECK_ERR();
    }
  }
}

template <typename T>
int MatrixSparse<T>::Init() {
  DEBUG_ASSERT(!this->_done_init);
  if (this->_done_init)
    return 1;
  this->_done_init = true;

  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  const T *orig_data = info->orig_data;
  const H2O4GPU_INT *orig_ptr = info->orig_ptr;
  const H2O4GPU_INT *orig_ind = info->orig_ind;

  // Allocate sparse matrix on gpu.
  hipMalloc(&_data, static_cast<size_t>(2) * _nnz * sizeof(T));
  hipMalloc(&_de, (this->_m + this->_n) * sizeof(T)); hipMemset(_de, 0, (this->_m + this->_n) * sizeof(T));
  // Equil(1); // JONTODO: If make sparse like dense for memory.
  hipMalloc(&_ind, static_cast<size_t>(2) * _nnz * sizeof(H2O4GPU_INT));
  hipMalloc(&_ptr, (this->_m + this->_n + 2) * sizeof(H2O4GPU_INT));
  DEBUG_CUDA_CHECK_ERR();

  if (_ord == ROW) {
    cml::spmat<T, H2O4GPU_INT, CblasRowMajor> A(_data, _ind, _ptr, this->_m,
        this->_n, _nnz);
    cml::spmat_memcpy(info->s_hdl, &A, orig_data, orig_ind, orig_ptr);
  } else {
    cml::spmat<T, H2O4GPU_INT, CblasColMajor> A(_data, _ind, _ptr, this->_m,
        this->_n, _nnz);
    cml::spmat_memcpy(info->s_hdl, &A, orig_data, orig_ind, orig_ptr);
  }
  DEBUG_CUDA_CHECK_ERR();

  return 0;
}

template <typename T>
int MatrixSparse<T>::Mul(char trans, T alpha, const T *x, T beta, T *y) const {
  DEBUG_ASSERT(this->_done_init);
  if (!this->_done_init)
    return 1;

  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);

  cml::vector<T> x_vec, y_vec;
  if (trans == 'n' || trans == 'N') {
    x_vec = cml::vector_view_array<T>(x, this->_n);
    y_vec = cml::vector_view_array<T>(y, this->_m);
  } else {
    x_vec = cml::vector_view_array<T>(x, this->_m);
    y_vec = cml::vector_view_array<T>(y, this->_n);
  }

  if (_ord == ROW) {
    cml::spmat<T, H2O4GPU_INT, CblasRowMajor> A(_data, _ind, _ptr, this->_m,
        this->_n, _nnz);
    cml::spblas_gemv(info->s_hdl, OpToCusparseOp(trans), info->descr, alpha,
        &A, &x_vec, beta, &y_vec);
  } else {
    cml::spmat<T, H2O4GPU_INT, CblasColMajor> A(_data, _ind, _ptr, this->_m,
        this->_n, _nnz);
    cml::spblas_gemv(info->s_hdl, OpToCusparseOp(trans), info->descr, alpha,
        &A, &x_vec, beta, &y_vec);
  }
  DEBUG_CUDA_CHECK_ERR();

  return 0;
}

template <typename T>
int MatrixSparse<T>::Mulvalid(char trans, T alpha, const T *x, T beta, T *y) const {
  DEBUG_ASSERT(this->_done_init);
  if (!this->_done_init)
    return 1;

  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);

  cml::vector<T> x_vec, y_vec;
  if (trans == 'n' || trans == 'N') {
    x_vec = cml::vector_view_array<T>(x, this->_n);
    y_vec = cml::vector_view_array<T>(y, this->_mvalid);
  } else {
    x_vec = cml::vector_view_array<T>(x, this->_mvalid);
    y_vec = cml::vector_view_array<T>(y, this->_n);
  }

  if (_ord == ROW) {
    cml::spmat<T, H2O4GPU_INT, CblasRowMajor> A(_vdata, _ind, _ptr, this->_mvalid,
        this->_n, _nnz);
    cml::spblas_gemv(info->s_hdl, OpToCusparseOp(trans), info->descr, alpha,
        &A, &x_vec, beta, &y_vec);
  } else {
    cml::spmat<T, H2O4GPU_INT, CblasColMajor> A(_vdata, _ind, _ptr, this->_mvalid,
        this->_n, _nnz);
    cml::spblas_gemv(info->s_hdl, OpToCusparseOp(trans), info->descr, alpha,
        &A, &x_vec, beta, &y_vec);
  }
  DEBUG_CUDA_CHECK_ERR();

  return 0;
}

template <typename T>
int MatrixSparse<T>::Equil(bool equillocal) {
  DEBUG_ASSERT(this->_done_init);
  if (!this->_done_init)
    return 1;

  if (this->_done_equil) return 0;
  else this->_done_equil=1;

  // Extract cublas handle from _info.
  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  hipblasHandle_t hdl = info->d_hdl;


  T *d = _de;
  T *e = d + this->_m;
  

  // Number of elements in matrix.
  size_t num_el = static_cast<size_t>(2) * _nnz;

  // Create bit-vector with signs of entries in A and then let A = f(A),
  // where f = |A| or f = |A|.^2.
  unsigned char *sign;
  size_t num_sign_bytes = (num_el + 7) / 8;
  hipMalloc(&sign, num_sign_bytes);
  CUDA_CHECK_ERR();

  size_t num_chars = num_el / 8;
  size_t grid_size = cml::calc_grid_dim(num_chars, cml::kBlockSize);
  if(equillocal){
    // Fill sign bits, assigning each thread a multiple of 8 elements.
    if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
      __SetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
                                                SquareF<T>());
    } else {
      __SetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
                                                AbsF<T>());
    }
    hipDeviceSynchronize();
    CUDA_CHECK_ERR();

    // If numel(A) is not a multiple of 8, then we need to set the last couple
    // of sign bits too.
    if (num_el > num_chars * 8) {
      if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
        __SetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
                                  num_el - num_chars * 8, SquareF<T>());
      } else {
        __SetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
                                  num_el - num_chars * 8, AbsF<T>());
      }
      hipDeviceSynchronize();
      CUDA_CHECK_ERR();
    }
  }
  // Perform Sinkhorn-Knopp equilibration.
  SinkhornKnopp(this, d, e, equillocal);
  hipDeviceSynchronize();

  if(equillocal){
    // Transform A = sign(A) .* sqrt(A) if 2-norm equilibration was performed,
    // or A = sign(A) .* A if the 1-norm was equilibrated.
    if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
      __UnSetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
                                                  SqrtF<T>());
    } else {
      __UnSetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
                                                  IdentityF<T>());
    }
    hipDeviceSynchronize();
    CUDA_CHECK_ERR();

    // Deal with last few entries if num_el is not a multiple of 8.
    if (num_el > num_chars * 8) {
      if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
        __UnSetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
                                    num_el - num_chars * 8, SqrtF<T>());
      } else {
        __UnSetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
                                    num_el - num_chars * 8, IdentityF<T>());
      }
      hipDeviceSynchronize();
      CUDA_CHECK_ERR();
    }
  }
  
  // Compute D := sqrt(D), E := sqrt(E), if 2-norm was equilibrated.
  if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
    thrust::transform(thrust::device_pointer_cast(d),
        thrust::device_pointer_cast(d + this->_m),
        thrust::device_pointer_cast(d), SqrtF<T>());
    thrust::transform(thrust::device_pointer_cast(e),
        thrust::device_pointer_cast(e + this->_n),
        thrust::device_pointer_cast(e), SqrtF<T>());
    hipDeviceSynchronize();
    CUDA_CHECK_ERR();
  }

  // Compute A := D * A * E.
  MultDiag(d, e, this->_m, this->_n, _nnz, _ord, _data, _ind, _ptr);
  hipDeviceSynchronize();
  CUDA_CHECK_ERR();

  // Scale A to have norm of 1 (in the kNormNormalize norm).
  T normA = NormEst(hdl, kNormNormalize, *this);
  CUDA_CHECK_ERR();
  hipDeviceSynchronize();
  cml::vector<T> a_vec = cml::vector_view_array(_data, num_el);
  cml::vector_scale(&a_vec, 1 / normA);
  hipDeviceSynchronize();

  // Scale d and e to account for normalization of A.
  cml::vector<T> d_vec = cml::vector_view_array<T>(d, this->_m);
  cml::vector<T> e_vec = cml::vector_view_array<T>(e, this->_n);
  T normd = cml::blas_nrm2(hdl, &d_vec);
  T norme = cml::blas_nrm2(hdl, &e_vec);
//  T scale = sqrt(normd * sqrt(this->_n) / (norme * sqrt(this->_m)));
  T scale = static_cast<T>(1.);
  cml::vector_scale(&d_vec, 1 / (scale * sqrt(normA)));
  cml::vector_scale(&e_vec, scale / sqrt(normA));
  hipDeviceSynchronize();

  hipFree(sign);
  CUDA_CHECK_ERR();

  DEBUG_PRINTF("norm A = %e, normd = %e, norme = %e\n", normA, normd, norme);

  return 0;
}

////////////////////////////////////////////////////////////////////////////////
/////////////////////// Equilibration Helpers //////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
namespace {

// Estimates norm of A. norm_type should either be kNorm2 or kNormFro.
template <typename T>
T NormEst(hipblasHandle_t hdl, NormTypes norm_type, const MatrixSparse<T>& A) {
  switch (norm_type) {
    case kNorm2: {
      return Norm2Est(hdl, &A);
    }
    case kNormFro: {
      const cml::vector<T> a = cml::vector_view_array(A.Data(), A.Nnz());
      return cml::blas_nrm2(hdl, &a) / std::sqrt(std::min(A.Rows(), A.Cols()));
    }
    case kNorm1:
      // 1-norm normalization doens't make make sense since it treats rows and
      // columns differently.
    default:
      ASSERT(false);
      return static_cast<T>(0.);
  }
}

// Performs D * A * E for A in row major
template <typename T>
void __global__ __MultRow(const T *d, const T *e, T *data,
                          const H2O4GPU_INT *row_ptr, const H2O4GPU_INT *col_ind,
                          H2O4GPU_INT size) {
  H2O4GPU_INT tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (H2O4GPU_INT t = tid; t < size; t += gridDim.x * blockDim.x)
    for (H2O4GPU_INT i = row_ptr[t]; i < row_ptr[t + 1]; ++i)
      data[i] *= d[t] * e[col_ind[i]];
}

// Performs D * A * E for A in col major
template <typename T>
void __global__ __MultCol(const T *d, const T *e, T *data,
                          const H2O4GPU_INT *col_ptr, const H2O4GPU_INT *row_ind,
                          H2O4GPU_INT size) {
  H2O4GPU_INT tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (H2O4GPU_INT t = tid; t < size; t += gridDim.x * blockDim.x)
    for (H2O4GPU_INT i = col_ptr[t]; i < col_ptr[t + 1]; ++i)
      data[i] *= d[row_ind[i]] * e[t];
}

template <typename T>
void MultDiag(const T *d, const T *e, H2O4GPU_INT m, H2O4GPU_INT n, H2O4GPU_INT nnz,
              typename MatrixSparse<T>::Ord ord, T *data, const H2O4GPU_INT *ind,
              const H2O4GPU_INT *ptr) {
  if (ord == MatrixSparse<T>::ROW) {
    size_t grid_dim_row = cml::calc_grid_dim(m, cml::kBlockSize);
    __MultRow<<<grid_dim_row, cml::kBlockSize>>>(d, e, data, ptr, ind, m);
    size_t grid_dim_col = cml::calc_grid_dim(n, cml::kBlockSize);
    __MultCol<<<grid_dim_col, cml::kBlockSize>>>(d, e, data + nnz, ptr + m + 1,
        ind + nnz, n);
  } else {
    size_t grid_dim_col = cml::calc_grid_dim(n, cml::kBlockSize);
    __MultCol<<<grid_dim_col, cml::kBlockSize>>>(d, e, data, ptr, ind, n);
    size_t grid_dim_row = cml::calc_grid_dim(m, cml::kBlockSize);
    __MultRow<<<grid_dim_row, cml::kBlockSize>>>(d, e, data + nnz, ptr + n + 1,
        ind + nnz, m);
  }
}

}  // namespace

#if !defined(H2O4GPU_DOUBLE) || H2O4GPU_DOUBLE==1
template class MatrixSparse<double>;
#endif

#if !defined(H2O4GPU_SINGLE) || H2O4GPU_SINGLE==1
template class MatrixSparse<float>;
#endif

}  // namespace h2o4gpu

