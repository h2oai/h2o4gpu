#include "hip/hip_runtime.h"
#include <cub/device/device_segmented_reduce.cuh>
#include "Arith.hpp"
#include "../../utils/GpuInfo.cuh"

namespace h2o4gpu {
namespace Matrix {

namespace kernel {

// Compute segment offsets for cub segment funtion.
template <typename T>
__global__ void segment_offsets(kParam<int> _res, kParam<T> _val) {
  size_t idx = global_thread_idx();
  if (idx < _res.size()) {
    _res.ptr[idx] = _val.cols * idx;
  }
}

/*
 * Compute min value for each row.
 * @tparam T Numeric type of the data
 * @param _res The output matrix with shape m x 1
 * @param _val The input matrix with shape m x n
 */
template <typename T>
__global__ void row_argmin_sequential(kParam<int> _res, kParam<T> _val) {

  size_t idx = global_thread_idx();
  if (idx < _val.rows) {
    T min = std::numeric_limits<T>::max();
    int min_idx = -1;
    for (size_t i = 0; i < _val.cols; ++i) {
      T value = _val.ptr[idx * _val.cols + i];
      if (value < min) {
        min = value;
        min_idx = i;
      }
    }
    _res.ptr[idx] = min_idx;
  }
}

}  // namespace kernel

// FIXME: The dot function deals with vector, not matrix.
template <typename T>
void DotOp<T>::dot(KmMatrix<T>& _res, KmMatrix<T>& _val) {
  this->dot(_res, _val, _val);
}
template <typename T>
void DotOp<T>::dot(KmMatrix<T>& _res, KmMatrix<T>& _lhs,
                   KmMatrix<T>& _rhs) {
  constexpr T alpha = 1.0;
  constexpr T beta = 1.0;
  hipblasHandle_t handle = GpuInfo::ins().cublas_handle();
  Blas::gemm(handle,
             HIPBLAS_OP_N, HIPBLAS_OP_N,  // FIXME
             _lhs.rows(), _rhs.cols(), _lhs.cols(),
             &alpha,
             _lhs.dev_ptr(), _lhs.cols(),
             _rhs.dev_ptr(), _rhs.cols(),
             &beta,
             _res.dev_ptr(), _res.cols());
}

template <typename T>
void VecBatchDotOp<T>::dot(KmMatrix<T>& _res, KmMatrix<T>& _val) {
  this->dot(_res, _val, _val);
}
template <typename T>
void VecBatchDotOp<T>::dot(KmMatrix<T>& _res,
                           KmMatrix<T>& _lhs, KmMatrix<T>& _rhs) {
  constexpr T alpha = 1.0;
  constexpr T beta = 1.0;
  hipblasHandle_t handle = GpuInfo::ins().cublas_handle();
  Blas::gemm_strided_batched(
      handle,
      HIPBLAS_OP_N, HIPBLAS_OP_T,
      1, 1, _rhs.cols(),  // m, n, k
      &alpha,
      _lhs.dev_ptr(), 1, _lhs.cols(),
      _rhs.dev_ptr(), 1, _rhs.cols(),
      &beta,
      _res.dev_ptr(), _res.cols(), 1,  // c should be columun vector
      _lhs.rows());
}

template <typename T>
T SumOp<T>::sum(KmMatrix<T>& _val) {
  T* raw_ptr = _val.dev_ptr();
  thrust::device_ptr<T> ptr (raw_ptr);
  T res = thrust::reduce(ptr, ptr + _val.size(), (T)0, thrust::plus<T>());
  return res;
}

template <typename T>
void MulOp<T>::mul(KmMatrix<T>& _res, KmMatrix<T>& _lhs, T _rhs) {
  hipblasHandle_t handle = GpuInfo::ins().cublas_handle();
  Blas::axpy(
      handle, _lhs.size(),  // handle, n
      &_rhs,                // alpha
      _lhs.dev_ptr(), 1,
      _res.dev_ptr(), 1);
}

template <typename T>
T MeanOp<T>::mean(KmMatrix<T>& _val) {
  T res = SumOp<T>().sum(_val);
  res = res / _val.size();
  return res;
}

template <typename T>
KmMatrix<int> ArgMinOp<T>::argmin(KmMatrix<T>& _val, KmMatrixDim _dim) {
  if (_dim == KmMatrixDim::ROW) {
    // FIXME: Didn't use cub function, offsets occupies n * sizeof(T) memory,
    // <index, value> occupies 2 * n * sizeof(T) memory considering memory
    // alignment. That would be 3 * n * sizeof(T) in total.
    KmMatrix<int> _res(_val.rows(), 1);
    kernel::row_argmin_sequential<<<div_roundup(_val.rows(), 256), 256>>>(
        _res.k_param(), _val.k_param());
    return _res;
  } else {
    // FIXME
    h2o4gpu_error("Not implemented");
    KmMatrix<int> res;
    return res;
  }
}

template <typename T>
KmMatrix<T> MinOp<T>::min(KmMatrix<T>& _val, KmMatrixDim _dim) {
  if (_dim == KmMatrixDim::ROW) {
    KmMatrix<T> res (_val.rows(), 1);
    KmMatrix<int> offsets (_val.rows() + 1, 1);

    kernel::segment_offsets<<<div_roundup(offsets.rows(), 256), 256>>>(
        offsets.k_param(), _val.k_param());

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    safe_cuda(hipcub::DeviceSegmentedReduce::Min(
        d_temp_storage,
        temp_storage_bytes,
        _val.dev_ptr(),
        res.dev_ptr(),
        _val.rows(),
        offsets.dev_ptr(),
        offsets.dev_ptr() + 1));

    safe_cuda(hipMalloc((void**)&d_temp_storage, temp_storage_bytes));
    safe_cuda(hipcub::DeviceSegmentedReduce::Min(
        d_temp_storage,
        temp_storage_bytes,
        _val.dev_ptr(),
        res.dev_ptr(),
        _val.rows(),
        offsets.dev_ptr(),
        offsets.dev_ptr() + 1));
    safe_cuda(hipFree(d_temp_storage));

    return res;
  } else {
    // FIXME
    h2o4gpu_error("Not implemented");
    KmMatrix<T> res;
    return res;
  }
}

#define INSTANTIATE(T)                                                  \
  template void DotOp<T>::dot(KmMatrix<T>& _res, KmMatrix<T>& _val);    \
  template void DotOp<T>::dot(KmMatrix<T>& _res, KmMatrix<T>& _lhs,     \
                              KmMatrix<T>& _rhs);                       \
  template void VecBatchDotOp<T>::dot(                                  \
      KmMatrix<T>& _res, KmMatrix<T>& _val);                            \
  template void VecBatchDotOp<T>::dot(                                  \
      KmMatrix<T>& _res, KmMatrix<T>& _lhs, KmMatrix<T>& _rhs);         \
  template T SumOp<T>::sum(KmMatrix<T>& _val);                          \
  template void MulOp<T>::mul(KmMatrix<T>& _res, KmMatrix<T>& _lhs, T _rhs); \
  template T MeanOp<T>::mean(KmMatrix<T>& _val);                        \
  template KmMatrix<int> ArgMinOp<T>::argmin(                           \
      KmMatrix<T>& _val, KmMatrixDim _dim);                             \
  template KmMatrix<T> MinOp<T>::min(KmMatrix<T>& _val, KmMatrixDim _dim); \


INSTANTIATE(double)
INSTANTIATE(float)
INSTANTIATE(int)

}      // namespace Matrix
}      // namespace h2o4gpu