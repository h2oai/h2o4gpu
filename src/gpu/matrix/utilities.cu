/*!
 * Modifications Copyright 2017-2018 H2O.ai, Inc.
 */
#include <stdio.h>
#include <assert.h>

#include "hip/hip_runtime.h"


#include <hipsolver.h>

/*******************/
/* iDivUp FUNCTION */
/*******************/
extern "C" int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpu_assert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) { exit(code); }
   }
}

extern "C" void gpuErrchk(hipError_t ans) { gpu_assert((ans), __FILE__, __LINE__); }

/**************************/
/* CUSOLVE ERROR CHECKING */
/**************************/
static const char *_cudaGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
        case HIPSOLVER_STATUS_SUCCESS:
            return "CUSOLVER_SUCCESS";

        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            return "CUSOLVER_STATUS_NOT_INITIALIZED";

        case HIPSOLVER_STATUS_ALLOC_FAILED:
            return "CUSOLVER_STATUS_ALLOC_FAILED";

        case HIPSOLVER_STATUS_INVALID_VALUE:
            return "CUSOLVER_STATUS_INVALID_VALUE";

        case HIPSOLVER_STATUS_ARCH_MISMATCH:
            return "CUSOLVER_STATUS_ARCH_MISMATCH";

        case HIPSOLVER_STATUS_EXECUTION_FAILED:
            return "CUSOLVER_STATUS_EXECUTION_FAILED";

        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            return "CUSOLVER_STATUS_INTERNAL_ERROR";

        case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "CUSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

    }

    return "<unknown>";
}

inline void __cusolveSafeCall(hipsolverStatus_t err, const char *file, const int line)
{
    if(HIPSOLVER_STATUS_SUCCESS != err) {
        fprintf(stderr, "CUSOLVE error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n",__FILE__, __LINE__,err, \
                                _cudaGetErrorEnum(err)); \
        hipDeviceReset(); assert(0); \
    }
}

extern "C" void cusolveSafeCall(hipsolverStatus_t err) { __cusolveSafeCall(err, __FILE__, __LINE__); }
