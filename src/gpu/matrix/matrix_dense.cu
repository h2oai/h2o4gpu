#include "hip/hip_runtime.h"
/*!
 * Modifications Copyright 2017 H2O.ai, Inc.
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include "cml/cml_blas.cuh"
#include "cml/cml_matrix.cuh"
#include "cml/cml_vector.cuh"
#include "equil_helper.cuh"
#include "matrix/matrix.h"
#include "matrix/matrix_dense.h"
#include "util.h"
#include "timer.h"

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <thrust/advance.h>
#include <cmath>
#include <limits>
#include <thrust/fill.h>
#include "../include/cuda_utils.h"

namespace h2o4gpu {

////////////////////////////////////////////////////////////////////////////////
////////////////////////////// Helper Functions ////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
namespace {

// File scoped constants.
const NormTypes kNormEquilibrate = kNorm2; 
const NormTypes kNormNormalize   = kNormFro;

template<typename T>
struct GpuData {
  const T *orig_data; // pointer to data on CPU
  hipblasHandle_t handle; // handle for data on GPU
  GpuData(const T *orig_data) : orig_data(orig_data) {
    hipblasCreate(&handle);
    //    fprintf(stderr,"HEREstart: %ld\n",handle); fflush(stderr);
    DEBUG_CUDA_CHECK_ERR();
  }
  ~GpuData() {
    //    fprintf(stderr,"HEREend: %ld\n",handle); fflush(stderr);

    if(handle!=NULL) hipblasDestroy(handle);
    DEBUG_CUDA_CHECK_ERR();
  }
};

hipblasOperation_t OpToCublasOp(char trans) {
  ASSERT(trans == 'n' || trans == 'N' || trans == 't' || trans == 'T');
  return trans == 'n' || trans == 'N' ? HIPBLAS_OP_N : HIPBLAS_OP_T;
}

  
template <typename T>
T NormEst(hipblasHandle_t hdl, NormTypes norm_type, const MatrixDense<T>& A);

template <typename T>
void MultDiag(const T *d, const T *e, size_t m, size_t n,
              typename MatrixDense<T>::Ord ord, T *data);

}  // namespace

////////////////////////////////////////////////////////////////////////////////
/////////////////////// MatrixDense Implementation /////////////////////////////
////////////////////////////////////////////////////////////////////////////////

  // original MatrixDense where only trainX and no trainY or validX or validY
  // Used by elastic_net.cpp to pass CPU data and put on GPU
template <typename T>
MatrixDense<T>::MatrixDense(int sharedA, int wDev, char ord, size_t m, size_t n, const T *data)
  : Matrix<T>(m, n, 0), _sharedA(sharedA), _wDev(wDev), _datatype(0), _dopredict(0), _data(0), _de(0) {
  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));
  _me=_wDev; // assume thread same as wDev if not given
  _datay=NULL;
  _vdata=NULL;
  _vdatay=NULL;
  _weight=NULL;

  ASSERT(ord == 'r' || ord == 'R' || ord == 'c' || ord == 'C');
  _ord = (ord == 'r' || ord == 'R') ? ROW : COL;

  
  DEBUG_FPRINTF(stderr,"MatrixDense1: ord=%c m=%d n=%d\n",ord,(int)m,(int)n);fflush(stderr);
  
#ifdef DEBUG
  //    CUDACHECK(hipSetDeviceFlags(hipDeviceMapHost)); // TODO: MapHostMemory
  hipDeviceProp_t props;
  CUDACHECK(hipGetDeviceProperties(&props, _wDev));
  fprintf(stderr,"Using: Compute %d.%d CUDA device: [%s] with id=%2d\n", props.major, props.minor, props.name,wDev); fflush(stderr);
#endif
  
  // Set GPU specific _info.


  PUSH_RANGE("MDnew",MDnew,1);
  GpuData<T> *info = new GpuData<T>(data); // new structure (holds pointer to data and GPU handle)
  this->_info = reinterpret_cast<void*>(info);
  GpuData<T> *infoy = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
  this->_infoy = reinterpret_cast<void*>(infoy);
  GpuData<T> *vinfo = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
  this->_vinfo = reinterpret_cast<void*>(vinfo);
  GpuData<T> *vinfoy = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
  this->_vinfoy = reinterpret_cast<void*>(vinfoy);
  GpuData<T> *weightinfo = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
  this->_weightinfo = reinterpret_cast<void*>(weightinfo);
  POP_RANGE("MDnew",MDnew,1);

  if(!this->_done_alloc){
    this->_done_alloc = true;
    // unlike CPU case, input pointer is always CPU so have to always allocate on GPU when calling this function.  So no use of sharedA related to pointer copy like in CPU case.

    // Copy Matrix to GPU.
    PUSH_RANGE("MDsend",MDsend,1);
    //  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info); // cast void -> GpuData
    double t0 = timer<double>();
    hipMalloc(&_data, this->_m * this->_n * sizeof(T)); // allocate on GPU
    double t1 = timer<double>();
    hipMemcpy(_data, info->orig_data, this->_m * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
    double t2 = timer<double>();
#ifdef DEBUG    
    printf("Time to allocate the data matrix on the GPU: %f\n", t1-t0);
    printf("Time to copy the data matrix to the GPU    : %f\n", t2-t1);
#endif

    hipMalloc(&_de, (m + n) * sizeof(T));
    thrust::device_ptr<T> dev_ptr = thrust::device_pointer_cast(static_cast<T*>(&_de[0]));
    T fill_value=0.0;
    thrust::fill(dev_ptr, dev_ptr + (m + n), fill_value);
    if(sharedA>0){
      Init(); // does nothing right now
      Equil(1); // JONTODO: Hack for now.  Need to pass equil
    }
    POP_RANGE("MDsend",MDsend,1);
  }
}
template <typename T>
MatrixDense<T>::MatrixDense(char ord, size_t m, size_t n, const T *data)
  : MatrixDense<T>(0, 0, ord, m, n, data){} // assume sharedA=0 and thread=wDev=0 if not given

template <typename T>
MatrixDense<T>::MatrixDense(int sharedA, int wDev, int datatype, char ord, size_t m, size_t n, T *data)
  : Matrix<T>(m, n, 0), _sharedA(sharedA), _wDev(wDev), _datatype(datatype), _dopredict(0), _data(0),_de(0) {
  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));
  _me=_wDev; // assume thread=wDev if not given
  _datay=NULL;
  _vdata=NULL;
  _vdatay=NULL;
  _weight=NULL;

  ASSERT(ord == 'r' || ord == 'R' || ord == 'c' || ord == 'C');
  _ord = (ord == 'r' || ord == 'R') ? ROW : COL;

  
  DEBUG_FPRINTF(stderr,"MatrixDense2: ord=%c m=%d n=%d\n",ord,(int)m,(int)n);fflush(stderr);
  
#ifdef DEBUG
  //    CUDACHECK(hipSetDeviceFlags(hipDeviceMapHost)); // TODO: MapHostMemory
  hipDeviceProp_t props;
  CUDACHECK(hipGetDeviceProperties(&props, _wDev));
  fprintf(stderr,"Using: Compute %d.%d CUDA device: [%s] with id=%2d\n", props.major, props.minor, props.name,wDev); fflush(stderr);
#endif
  

  if(datatype==1){
    // input data pointer is already on GPU on this wDev, so just copy pointer
    // no info->orig_data, so send 0 to GpuData
    PUSH_RANGE("MDnew",MDnew,1);
    GpuData<T> *info = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_info = reinterpret_cast<void*>(info);
    POP_RANGE("MDnew",MDnew,1);
    GpuData<T> *infoy = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_infoy = reinterpret_cast<void*>(infoy);
    GpuData<T> *vinfo = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_vinfo = reinterpret_cast<void*>(vinfo);
    GpuData<T> *vinfoy = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_vinfoy = reinterpret_cast<void*>(vinfoy);
    GpuData<T> *weightinfo = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_weightinfo = reinterpret_cast<void*>(weightinfo);
    
    // source pointer is on this GPU

    // just copy GPU pointer
    _data = data;
    if(!this->_done_alloc){
      this->_done_alloc = true;
      hipMalloc(&_de, (m + n) * sizeof(T));
      thrust::device_ptr<T> dev_ptr = thrust::device_pointer_cast(static_cast<T*>(&_de[0]));
      T fill_value=0.0;
      thrust::fill(dev_ptr, dev_ptr + (m + n), fill_value);

      Init(); // does nothing right now
      Equil(1); // JONTODO: Hack for now.  Need to pass equil
    }

  }
  else{
    PUSH_RANGE("MDnew",MDnew,1);
    GpuData<T> *info = new GpuData<T>(data); // new structure (holds pointer to data and GPU handle)
    this->_info = reinterpret_cast<void*>(info);
    GpuData<T> *infoy = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_infoy = reinterpret_cast<void*>(infoy);
    GpuData<T> *vinfo = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_vinfo = reinterpret_cast<void*>(vinfo);
    GpuData<T> *vinfoy = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_vinfoy = reinterpret_cast<void*>(vinfoy);
    GpuData<T> *weightinfo = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_weightinfo = reinterpret_cast<void*>(weightinfo);
    POP_RANGE("MDnew",MDnew,1);

    if(!this->_done_alloc){
      this->_done_alloc = true;

      // Unlike CPU case, can't pointer copy as going from CPU to GPU
      
      // Copy CPU Matrix to GPU.
      PUSH_RANGE("MDsend",MDsend,1);
      //  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info); // cast void -> GpuData
      double t0 = timer<double>();
      hipMalloc(&_data, this->_m * this->_n * sizeof(T)); // allocate on GPU
      double t1 = timer<double>();
      hipMemcpy(_data, info->orig_data, this->_m * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
      hipMalloc(&_de, (m + n) * sizeof(T));
      thrust::device_ptr<T> dev_ptr = thrust::device_pointer_cast(static_cast<T*>(&_de[0]));
      T fill_value=0.0;
      thrust::fill(dev_ptr, dev_ptr + (m + n), fill_value);

      if(sharedA>0){
        Init(); // does nothing right now
        Equil(1); // JONTODO: Hack for now.  Need to pass equil
      }
      double t2 = timer<double>();
#ifdef DEBUG
      printf("Time to allocate the data matrix on the GPU: %f\n", t1-t0);
      printf("Time to copy the data matrix to the GPU    : %f\n", t2-t1);
#endif
      POP_RANGE("MDsend",MDsend,1);
    }
  }
}


  // like original MatrixDense, but also feed in CPU data for trainY, validX, and validY
  // Used by elastic_net_ptr.cpp to pass CPU data and put on GPU
template <typename T>
MatrixDense<T>::MatrixDense(int sharedA, int me, int wDev, char ord, size_t m, size_t n, size_t mValid, const T *data, const T *datay, const T *vdata, const T *vdatay, const T *weight)
  : Matrix<T>(m, n, mValid), _sharedA(sharedA), _me(me), _wDev(wDev), _datatype(0), _dopredict(0), _data(0), _datay(0), _vdata(0), _vdatay(0), _weight(0), _de(0) {
  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));

  ASSERT(ord == 'r' || ord == 'R' || ord == 'c' || ord == 'C');
  _ord = (ord == 'r' || ord == 'R') ? ROW : COL;

  
  DEBUG_FPRINTF(stderr,"MatrixDense3: ord=%c m=%d n=%d mValid=%d\n",ord,(int)m,(int)n,int(mValid));fflush(stderr);
  
#ifdef DEBUG
  //    CUDACHECK(hipSetDeviceFlags(hipDeviceMapHost)); // TODO: MapHostMemory
  hipDeviceProp_t props;
  CUDACHECK(hipGetDeviceProperties(&props, _wDev));
  fprintf(stderr,"Using: Compute %d.%d CUDA device: [%s] with id=%2d\n", props.major, props.minor, props.name,wDev); fflush(stderr);
#endif


  // source pointer is on CPU
  // Set GPU specific _info.
  PUSH_RANGE("MDnew",MDnew,1);
  GpuData<T> *info = new GpuData<T>(data); // new structure (holds pointer to data and GPU handle)
  GpuData<T> *infoy = new GpuData<T>(datay); // new structure (holds pointer to data and GPU handle)
  GpuData<T> *vinfo = new GpuData<T>(vdata); // new structure (holds pointer to data and GPU handle)
  GpuData<T> *vinfoy = new GpuData<T>(vdatay); // new structure (holds pointer to data and GPU handle)
  GpuData<T> *weightinfo = new GpuData<T>(weight); // new structure (holds pointer to data and GPU handle)
  this->_info = reinterpret_cast<void*>(info);
  this->_infoy = reinterpret_cast<void*>(infoy);
  this->_vinfo = reinterpret_cast<void*>(vinfo);
  this->_vinfoy = reinterpret_cast<void*>(vinfoy);
  this->_weightinfo = reinterpret_cast<void*>(weightinfo);
  POP_RANGE("MDnew",MDnew,1);


  if(!this->_done_alloc){
    this->_done_alloc = true;

    // Unlike CPU case, can't pointer copy even if sharedA!=0
    
    // Copy Matrix to GPU.
    PUSH_RANGE("MDsend",MDsend,1);
    //  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info); // cast void -> GpuData
    double t0 = timer<double>();
    hipMalloc(&_data, this->_m * this->_n * sizeof(T)); // allocate on GPU
    hipMalloc(&_datay, this->_m * sizeof(T)); // allocate on GPU
    hipMalloc(&_vdata, this->_mvalid * this->_n * sizeof(T)); // allocate on GPU
    hipMalloc(&_vdatay, this->_mvalid * sizeof(T)); // allocate on GPU
    hipMalloc(&_weight, this->_m * sizeof(T)); // allocate on GPU
    double t1 = timer<double>();

    hipMemcpy(_data, info->orig_data, this->_m * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
    if(infoy->orig_data){
      hipMemcpy(_datay, infoy->orig_data, this->_m * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
      _dopredict=0;
    }
    else{
      _dopredict=1;
    }
    if(vinfo->orig_data){
      hipMemcpy(_vdata, vinfo->orig_data, this->_mvalid * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
    }
    else{
      if(this->_mvalid>0){ fprintf(stderr,"vinfo->orig_data NULL but this->_mvalid>0\n"); fflush(stderr); exit(1); }
    }
    if(vinfoy->orig_data){
      hipMemcpy(_vdatay, vinfoy->orig_data, this->_mvalid * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
    }
    else{
      if(this->_mvalid>0){ fprintf(stderr,"vinfoy->orig_data NULL but this->_mvalid>0\n"); fflush(stderr); exit(1); }
    }
    if(weightinfo->orig_data){
      hipMemcpy(_weight, weightinfo->orig_data, this->_m * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
    }
    else{// if no weights, set as unity weights
      thrust::device_ptr<T> dev_ptr = thrust::device_pointer_cast(static_cast<T*>(&_weight[0]));
      T fill_value=1.0;
      thrust::fill(dev_ptr, dev_ptr + m, fill_value);
    }
    hipMalloc(&_de, (m + n) * sizeof(T));
    thrust::device_ptr<T> dev_ptr = thrust::device_pointer_cast(static_cast<T*>(&_de[0]));
    T fill_value=0.0;
    thrust::fill(dev_ptr, dev_ptr + (m + n), fill_value);

    if(sharedA>0){
      Init(); // does nothing right now
      Equil(1); // JONTODO: Hack for now.  Need to pass equil
    }
    double t2 = timer<double>();
#ifdef DEBUG
    printf("Time to allocate the data matrix on the GPU: %f\n", t1-t0);
    printf("Time to copy the data matrix to the GPU    : %f\n", t2-t1);
#endif
    POP_RANGE("MDsend",MDsend,1);
  }
}

  template <typename T>
  MatrixDense<T>::MatrixDense(int wDev, char ord, size_t m, size_t n, size_t mValid, const T *data, const T *datay, const T *vdata, const T *vdatay, const T *weight)
    : MatrixDense<T>(0,wDev,wDev,ord,m,n,mValid,data,datay,vdata,vdatay,weight){} // assume sharedA=0 and source thread=wDev if not given

  // like original MatrixDense, but also feed in CPU data for trainY, validX, and validY
  // Used by elastic_net_ptr.cpp to pass CPU data and put on GPU
  // datatype=0: CPU pointer to data
  // datatype=1: GPU pointer to data
template <typename T>
MatrixDense<T>::MatrixDense(int sharedA, int me, int wDev, int datatype, char ord, size_t m, size_t n, size_t mValid, T *data, T *datay, T *vdata, T *vdatay, T *weight)
  : Matrix<T>(m, n, mValid), _sharedA(sharedA), _me(me), _wDev(wDev), _datatype(datatype), _dopredict(0), _data(0), _datay(0), _vdata(0), _vdatay(0), _weight(0), _de(0) {
  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));

  DEBUG_FPRINTF(stderr,"%d\n", ord == 'r');
  DEBUG_FPRINTF(stderr,"%d\n", ord == 'c');
  DEBUG_FPRINTF(stderr,"ord=%c m=%d n=%d mValid=%d\n",ord,(int)m,(int)n,int(mValid));
  DEBUG_FPRINTF(stderr,"MatrixDense4: ord=%c m=%d n=%d mValid=%d\n",ord,(int)m,(int)n,int(mValid));

  ASSERT(ord == 'r' || ord == 'R' || ord == 'c' || ord == 'C');
  _ord = (ord == 'r' || ord == 'R') ? ROW : COL;

#ifdef DEBUG
  //    CUDACHECK(hipSetDeviceFlags(hipDeviceMapHost)); // TODO: MapHostMemory
  hipDeviceProp_t props;
  CUDACHECK(hipGetDeviceProperties(&props, _wDev));
  DEBUG_FPRINTF(stderr,"Using: Compute %d.%d CUDA device: [%s] with id=%2d\n", props.major, props.minor, props.name,wDev);
#endif


  if(datatype==1){
    // source pointer is on GPU already
    // Set GPU specific _info.
    PUSH_RANGE("MDnew",MDnew,1);
    GpuData<T> *info = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *infoy = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *vinfo = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *vinfoy = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *weightinfo = new GpuData<T>(0); // new structure (holds pointer to data and GPU handle)
    this->_info = reinterpret_cast<void*>(info);
    this->_infoy = reinterpret_cast<void*>(infoy);
    this->_vinfo = reinterpret_cast<void*>(vinfo);
    this->_vinfoy = reinterpret_cast<void*>(vinfoy);
    this->_weightinfo = reinterpret_cast<void*>(weightinfo);
    POP_RANGE("MDnew",MDnew,1);


    // Just copy GPU pointer
    _data = data;
    _datay = datay;
    _vdata = vdata;
    _vdatay = vdatay;
    _weight = weight;

    if(_datay) _dopredict=0;
    else _dopredict=1;

    
    if(_weight==NULL){
      DEBUG_FPRINTF(stderr,"datatype=1: making up unity weights: %d %p\n",m,&_weight);
      CUDACHECK(hipMalloc(&_weight, m * sizeof(T))); // allocate on GPU
      thrust::device_ptr<T> dev_ptr = thrust::device_pointer_cast(static_cast<T*>(&_weight[0]));
      T fill_value=1.0;
      thrust::fill(dev_ptr, dev_ptr + m, fill_value);
    }
    
      
    if(!this->_done_alloc){
      this->_done_alloc = true;
      hipMalloc(&_de, (m + n) * sizeof(T));
      thrust::device_ptr<T> dev_ptr = thrust::device_pointer_cast(static_cast<T*>(&_de[0]));
      T fill_value=0.0;
      thrust::fill(dev_ptr, dev_ptr + (m + n), fill_value);
      if(sharedA>0){
        Init(); // does nothing right now
        Equil(1); // JONTODO: Hack for now.  Need to pass equil
      }
    }
  }
  else{
    // source pointer is on CPU
    // Set GPU specific _info.
    PUSH_RANGE("MDnew",MDnew,1);
    GpuData<T> *info = new GpuData<T>(data); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *infoy = new GpuData<T>(datay); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *vinfo = new GpuData<T>(vdata); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *vinfoy = new GpuData<T>(vdatay); // new structure (holds pointer to data and GPU handle)
    GpuData<T> *weightinfo = new GpuData<T>(weight); // new structure (holds pointer to data and GPU handle)
    this->_info = reinterpret_cast<void*>(info);
    this->_infoy = reinterpret_cast<void*>(infoy);
    this->_vinfo = reinterpret_cast<void*>(vinfo);
    this->_vinfoy = reinterpret_cast<void*>(vinfoy);
    this->_weightinfo = reinterpret_cast<void*>(weightinfo);
    POP_RANGE("MDnew",MDnew,1);


    if(!this->_done_alloc){
      this->_done_alloc = true;
      // Copy CPU Matrix to GPU.
      PUSH_RANGE("MDsend",MDsend,1);
      //  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info); // cast void -> GpuData
      double t0 = timer<double>();
      hipMalloc(&_data, this->_m * this->_n * sizeof(T)); // allocate on GPU
      hipMalloc(&_datay, this->_m * sizeof(T)); // allocate on GPU
      hipMalloc(&_vdata, this->_mvalid * this->_n * sizeof(T)); // allocate on GPU
      hipMalloc(&_vdatay, this->_mvalid * sizeof(T)); // allocate on GPU
      hipMalloc(&_weight, this->_m * sizeof(T)); // allocate on GPU
      double t1 = timer<double>();

      hipMemcpy(_data, info->orig_data, this->_m * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
      if(infoy->orig_data){
        hipMemcpy(_datay, infoy->orig_data, this->_m * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
        _dopredict=0;
      }
      else{
        _dopredict=1;
      }
      hipMemcpy(_vdata, vinfo->orig_data, this->_mvalid * this->_n * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
      hipMemcpy(_vdatay, vinfoy->orig_data, this->_mvalid * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
      if(weightinfo->orig_data){
        hipMemcpy(_weight, weightinfo->orig_data, this->_m * sizeof(T),hipMemcpyHostToDevice); // copy from orig CPU data to GPU
      }
      else{
        DEBUG_FPRINTF(stderr,"datatype=0: making up unity weights: %d\n",m);
        CUDACHECK(hipMalloc(&_weight, this->_m * sizeof(T))); // allocate on GPU
        thrust::device_ptr<T> dev_ptr=thrust::device_pointer_cast(static_cast<T*>(_weight));
        T fill_value=1.0;
        thrust::fill(dev_ptr, dev_ptr + this->_m, fill_value);
      }
      hipMalloc(&_de, (m + n) * sizeof(T));
      thrust::device_ptr<T> dev_ptr = thrust::device_pointer_cast(static_cast<T*>(&_de[0]));
      T fill_value=0.0;
      thrust::fill(dev_ptr, dev_ptr + (m + n), fill_value);
      if(sharedA>0){
        Init(); // does nothing right now
        Equil(1); // JONTODO: Hack for now.  Need to pass equil
      }
      double t2 = timer<double>();
#ifdef DEBUG
      printf("Time to allocate the data matrix on the GPU: %f\n", t1-t0);
      printf("Time to copy the data matrix to the GPU    : %f\n", t2-t1);
#endif
      POP_RANGE("MDsend",MDsend,1);
    }
  }
}

template <typename T>
MatrixDense<T>::MatrixDense(int wDev, int datatype, char ord, size_t m, size_t n, size_t mValid, T *data, T *datay, T *vdata, T *vdatay, T *weight)
  : MatrixDense<T>(0,wDev,wDev,datatype,ord,m,n,mValid,data,datay,vdata,vdatay,weight){} // assume sharedA=0 and thread=wDev if not given

  // MatrixDense where input actual A object that contains all CPU information, but need to go from 1 GPU to multiple GPU
  // Used by elastic_net_ptr.cpp inside openmp loop for each core


template <typename T>
MatrixDense<T>::MatrixDense(int sharedA, int me, int wDev, const MatrixDense<T>& A)
  : Matrix<T>(A._m, A._n, A._mvalid), _sharedA(sharedA), _me(me), _wDev(wDev), _data(0),_de(0), _ord(A._ord) {

  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));

  DEBUG_FPRINTF(stderr,"MatrixDense5: ord=%c m=%d n=%d mValid=%d\n",A._ord,A._m,A._n,A._mvalid);

  PUSH_RANGE("MDnew",MDnew,2);
  GpuData<T> *info_A   = reinterpret_cast<GpuData<T>*>(A._info); // cast from void to GpuData
  GpuData<T> *infoy_A  = reinterpret_cast<GpuData<T>*>(A._infoy); // cast from void to GpuData
  GpuData<T> *vinfo_A  = reinterpret_cast<GpuData<T>*>(A._vinfo); // cast from void to GpuData
  GpuData<T> *vinfoy_A = reinterpret_cast<GpuData<T>*>(A._vinfoy); // cast from void to GpuData
  GpuData<T> *weightinfo_A = reinterpret_cast<GpuData<T>*>(A._weightinfo); // cast from void to GpuData

  GpuData<T> *info;
  GpuData<T> *infoy;
  GpuData<T> *vinfo;
  GpuData<T> *vinfoy;
  GpuData<T> *weightinfo;
  if(info_A->orig_data) info = new GpuData<T>(info_A->orig_data); // create new GpuData structure with point to CPU data
  else info = new GpuData<T>(0); // create new GpuData structure with point to CPU data
  if(infoy_A->orig_data) infoy  = new GpuData<T>(infoy_A->orig_data); // create new GpuData structure with point to CPU data
  else infoy = new GpuData<T>(0); // create new GpuData structure with point to CPU data
  if(vinfo_A->orig_data) vinfo  = new GpuData<T>(vinfo_A->orig_data); // create new GpuData structure with point to CPU data
  else vinfo = new GpuData<T>(0); // create new GpuData structure with point to CPU data
  if(vinfoy_A->orig_data) vinfoy = new GpuData<T>(vinfoy_A->orig_data); // create new GpuData structure with point to CPU data
  else vinfoy = new GpuData<T>(0); // create new GpuData structure with point to CPU data
  if(weightinfo_A->orig_data) weightinfo = new GpuData<T>(weightinfo_A->orig_data); // create new GpuData structure with point to CPU data
  else weightinfo = new GpuData<T>(0); // create new GpuData structure with point to CPU data


  this->_info = reinterpret_cast<void*>(info); // back to cast as void
  this->_infoy = reinterpret_cast<void*>(infoy); // back to cast as void
  this->_vinfo = reinterpret_cast<void*>(vinfo); // back to cast as void
  this->_vinfoy = reinterpret_cast<void*>(vinfoy); // back to cast as void
  this->_weightinfo = reinterpret_cast<void*>(weightinfo); // back to cast as void
  POP_RANGE("MDnew",MDnew,2);


  if(!this->_done_alloc){
    this->_done_alloc = true;
    if(A._wDev == _wDev && A._me == _me && (A._sharedA==0 || _sharedA==0)){ // if on same device and same thread, just copy pointer
      DEBUG_FPRINTF(stderr,"ATYPE%d\n",0);
      _data   = A._data;
      _datay  = A._datay;
      _vdata  = A._vdata;
      _vdatay = A._vdatay;
      _weight = A._weight;
      _de = A._de;
      _dopredict = A._dopredict;
      //      Init();
      //      this->_done_equil=1;
    }
    else if(A._wDev == _wDev && A._sharedA!=0 && _sharedA!=0){ // if on same device and sharing memory, then just copy pointer
      DEBUG_FPRINTF(stderr,"ATYPE%d\n",1);
      _data   = A._data;
      _datay  = A._datay;
      _vdata  = A._vdata;
      _vdatay = A._vdatay;
      _weight = A._weight;
      _de = A._de;
      _dopredict = A._dopredict;
      Init();
      this->_done_equil=1;
    }
    else{
      DEBUG_FPRINTF(stderr,"ATYPE%d\n",2);
      // Copy Matrix to from source GPU to this GPU
      PUSH_RANGE("MDcopy",MDcopy,1);
      //GpuData<T> *info = reinterpret_cast<GpuData<T>*>(_info); // cast void -> GpuData
      double t0 = timer<double>();
      if(A._data) hipMalloc(&_data, A._m * A._n * sizeof(T)); // allocate on GPU
      if(A._datay) hipMalloc(&_datay, A._m * sizeof(T)); // allocate on GPU
      if(A._vdata) hipMalloc(&_vdata, A._mvalid * A._n * sizeof(T)); // allocate on GPU
      if(A._vdatay) hipMalloc(&_vdatay, A._mvalid * sizeof(T)); // allocate on GPU
      if(A._weight) hipMalloc(&_weight, A._m * sizeof(T)); // allocate on GPU
      double t1 = timer<double>();
      if(A._data) hipMemcpyPeer(_data, _wDev, A._data, A._wDev, A._m * A._n * sizeof(T)); // dest: _data destid: _wDev  source: A._data sourceid: A._wDev
      if(A._datay){
        hipMemcpyPeer(_datay, _wDev, A._datay, A._wDev, A._m * sizeof(T)); // dest: _data destid: _wDev  source: A._data sourceid: A._wDev
        _dopredict=0;
      }
      else{
        _dopredict=1;
      }
      if(A._vdata) hipMemcpyPeer(_vdata, _wDev, A._vdata, A._wDev, A._mvalid * A._n * sizeof(T)); // dest: _data destid: _wDev  source: A._data sourceid: A._wDev
      if(A._vdatay) hipMemcpyPeer(_vdatay, _wDev, A._vdatay, A._wDev, A._mvalid * sizeof(T)); // dest: _data destid: _wDev  source: A._data sourceid: A._wDev
      if(A._weight) hipMemcpyPeer(_weight, _wDev, A._weight, A._wDev, A._m * sizeof(T)); // dest: _data destid: _wDev  source: A._data sourceid: A._wDev
      if(A._de) hipMalloc(&_de, (A._m + A._n) * sizeof(T)); hipMemcpyPeer(_de, _wDev, A._de, A._wDev, (A._m + A._n) * sizeof(T));
      if(sharedA>0){
        Init();
        Equil(1);
      }
      double t2 = timer<double>();
#ifdef DEBUG
      printf("Time to allocate the data matrix on the GPU: %f\n", t1-t0);
      printf("Time to copy the data matrix to the GPU    : %f\n", t2-t1);
#endif
      POP_RANGE("MDcopy",MDcopy,1);
    }
  }

}

template <typename T>
MatrixDense<T>::MatrixDense(int me, int wDev, const MatrixDense<T>& A)
  : MatrixDense<T>(0, me, wDev, A){} // then assume not sharing memory

template <typename T>
MatrixDense<T>::MatrixDense(int wDev, const MatrixDense<T>& A)
  : MatrixDense<T>(wDev, wDev, A){} // then assume thread=wDev for the new matrix (i.e. not input A)


template <typename T>
MatrixDense<T>::MatrixDense(const MatrixDense<T>& A)
  : MatrixDense<T>(A._wDev, A){} // then assume same device as input A

template <typename T>
MatrixDense<T>::~MatrixDense() {

  
  // return;//TODO: Some deconstructor issue FIXME.  Segfaults after adding weights.  Can't find issue.
  
  checkwDev(_wDev);
  CUDACHECK(hipSetDevice(_wDev));

  if(0){
    GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
    GpuData<T> *infoy = reinterpret_cast<GpuData<T>*>(this->_infoy);
    GpuData<T> *vinfo = reinterpret_cast<GpuData<T>*>(this->_vinfo);
    GpuData<T> *vinfoy = reinterpret_cast<GpuData<T>*>(this->_vinfoy);
    GpuData<T> *weightinfo = reinterpret_cast<GpuData<T>*>(this->_weightinfo);

    if(info) delete info; this->_info = 0;
    if(infoy) delete infoy; this->_infoy = 0;
    if(vinfo) delete vinfo; this->_vinfo = 0;
    if(vinfoy) delete vinfoy; this->_vinfoy = 0;
    if(weightinfo) delete weightinfo; this->_weightinfo = 0;
  }

  //  fprintf(stderr,"HERE1\n"); fflush(stderr);

  if(0){ // Note that this frees these pointers as soon as MatrixDense constructor goes out of scope, and might want more fine-grained control over GPU memory if inside (say) high-level python API
    // If 0 is used, then need to ensure user calls a finish() or something to free memory.  If 0, also allows user to call (say) fit() or fitptr() multiple times
    
    if (this->_done_init && _data) {
      //      fprintf(stderr,"Freeing _data: %p\n",(void*)_data); fflush(stderr);
      hipFree(_data);
      this->_data = 0;
      DEBUG_CUDA_CHECK_ERR();
    }
    //  fprintf(stderr,"HERE2\n"); fflush(stderr);
    if (this->_done_init && _datay) {
      //      fprintf(stderr,"Freeing _datay: %p\n",(void*)_datay); fflush(stderr);
      hipFree(_datay);
      this->_datay = 0;
      DEBUG_CUDA_CHECK_ERR();
    }
    //  fprintf(stderr,"HERE3\n"); fflush(stderr);
    if (this->_done_init && _vdata) {
      //      fprintf(stderr,"Freeing _vdata: %p\n",(void*)_vdata); fflush(stderr);
      hipFree(_vdata);
      this->_vdata = 0;
      DEBUG_CUDA_CHECK_ERR();
    }
    //  fprintf(stderr,"HERE4\n"); fflush(stderr);
    if (this->_done_init && _vdatay) {
      //      fprintf(stderr,"Freeing _vdatay: %p\n",(void*)_vdatay); fflush(stderr);
      hipFree(_vdatay);
      this->_vdatay = 0;
      DEBUG_CUDA_CHECK_ERR();
    }
    //  fprintf(stderr,"HERE5\n"); fflush(stderr);

    if (this->_done_init && _weight) {
      //      fprintf(stderr,"Freeing _weight: %p\n",(void*)_weight); fflush(stderr);
      hipFree(_weight);
      this->_weight = 0;
      DEBUG_CUDA_CHECK_ERR();
    }
    //  fprintf(stderr,"HERE6\n"); fflush(stderr);

    if(this->_done_init && _de && !_sharedA){ // JONTODO: When sharedA=1, only free on sourceme thread and sourcewDev device (can store sourcethread for-- sourceme -- data and only free if on source thread)
      //      fprintf(stderr,"Freeing _de: %p\n",(void*)_weight); fflush(stderr);
      hipFree(_de);
      this->_de=0;
      DEBUG_CUDA_CHECK_ERR();
    }
  }
  
}

      
template <typename T>
int MatrixDense<T>::Init() {
  DEBUG_EXPECT(!this->_done_init);
  if (this->_done_init)
    return 1;
  this->_done_init = true;
  CUDACHECK(hipSetDevice(_wDev));

  PUSH_RANGE("MDinit",MDinit,1);
  POP_RANGE("MDinit",MDinit,1);

  DEBUG_CUDA_CHECK_ERR();

  return 0;
}

template <typename T>
int MatrixDense<T>::GetTrainX(int datatype, size_t size, T**data) const {

  CUDACHECK(hipSetDevice(_wDev));

  if(_data){
    if(datatype==1){
      hipMemcpy(*data, _data, size* sizeof(T),hipMemcpyDeviceToHost);
      CUDA_CHECK_ERR();
    }
    else{
      std::memcpy(*data, _data, size * sizeof(T));
    }
    return(0);
  }
  else return(1);
}
template <typename T>
int MatrixDense<T>::GetTrainY(int datatype, size_t size, T**data) const {

  CUDACHECK(hipSetDevice(_wDev));

  if(_datay){
    if(datatype==1){
      hipMemcpy(*data, _datay, size* sizeof(T),hipMemcpyDeviceToHost);
      CUDA_CHECK_ERR();
    }
    else{
      std::memcpy(*data, _datay, size * sizeof(T));
    }
    return(0);
  }
  else return(1);
}

template <typename T>
int MatrixDense<T>::GetValidX(int datatype, size_t size, T**data) const {

  CUDACHECK(hipSetDevice(_wDev));

  if(_vdata){
    if(datatype==1){
      hipMemcpy(*data, _vdata, size* sizeof(T),hipMemcpyDeviceToHost);
      CUDA_CHECK_ERR();
    }
    else{
      std::memcpy(*data, _vdata, size * sizeof(T));
    }
    return(0);
  }
  else return(1);
}
template <typename T>
int MatrixDense<T>::GetValidY(int datatype, size_t size, T**data) const {

  CUDACHECK(hipSetDevice(_wDev));

  if(_vdatay){
    if(datatype==1){
      hipMemcpy(*data, _vdatay, size* sizeof(T),hipMemcpyDeviceToHost);
      CUDA_CHECK_ERR();
    }
    else{
      std::memcpy(*data, _vdatay, size * sizeof(T));
    }
    return(0);
  }
  else return(1);
}
template <typename T>
int MatrixDense<T>::GetWeight(int datatype, size_t size, T**data) const {

  CUDACHECK(hipSetDevice(_wDev));

  if(_weight){
    if(datatype==1){
      hipMemcpy(*data, _weight, size* sizeof(T),hipMemcpyDeviceToHost);
      CUDA_CHECK_ERR();
    }
    else{
      std::memcpy(*data, _weight, size * sizeof(T));
    }
    return(0);
  }
  else return(1);
}


template <typename T>
int MatrixDense<T>::Mul(char trans, T alpha, const T *x, T beta, T *y) const {

  DEBUG_EXPECT(this->_done_init);
  if (!this->_done_init)
    return 1;
  CUDACHECK(hipSetDevice(_wDev));

  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  hipblasHandle_t hdl = info->handle;

  const cml::vector<T> x_vec = cml::vector_view_array<T>(x, this->_n);
  cml::vector<T> y_vec = cml::vector_view_array<T>(y, this->_m);

  //  Performs the matrix-vector operations y := alpha*A*x + beta*y or y := alpha*A'*x + beta*y where alpha and beta are scalars, x and y are vectors and A is an m by n matrix
  // _data is A on GPU
  //https://docs.oracle.com/cd/B19306_01/appdev.102/b14258/u_nla.htm#CIAFEAFG
  if (_ord == ROW) {
    cml::matrix<T, CblasRowMajor> A =
        cml::matrix_view_array<T, CblasRowMajor>(_data, this->_m, this->_n);
    cml::blas_gemv(hdl, OpToCublasOp(trans), alpha, &A, &x_vec, beta,
        &y_vec);
  } else {
    cml::matrix<T, CblasColMajor> A =
        cml::matrix_view_array<T, CblasColMajor>(_data, this->_m, this->_n);
    cml::blas_gemv(hdl, OpToCublasOp(trans), alpha, &A, &x_vec, beta, &y_vec);
  }
  CUDA_CHECK_ERR();

  return 0;
}

  template <typename T>
int MatrixDense<T>::Mulvalid(char trans, T alpha, const T *x, T beta, T *y) const {

  DEBUG_EXPECT(this->_done_init);
  if (!this->_done_init)
    return 1;
  CUDACHECK(hipSetDevice(_wDev));

  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  hipblasHandle_t hdl = info->handle;

  const cml::vector<T> x_vec = cml::vector_view_array<T>(x, this->_n);
  cml::vector<T> y_vec = cml::vector_view_array<T>(y, this->_mvalid);

  //  Performs the matrix-vector operations y := alpha*A*x + beta*y or y := alpha*A'*x + beta*y where alpha and beta are scalars, x and y are vectors and A is an m by n matrix
  // _vdata is A on GPU
  //https://docs.oracle.com/cd/B19306_01/appdev.102/b14258/u_nla.htm#CIAFEAFG
  if (_ord == ROW) {
    cml::matrix<T, CblasRowMajor> A =
        cml::matrix_view_array<T, CblasRowMajor>(_vdata, this->_mvalid, this->_n);
    cml::blas_gemv(hdl, OpToCublasOp(trans), alpha, &A, &x_vec, beta,
        &y_vec);
  } else {
    cml::matrix<T, CblasColMajor> A =
        cml::matrix_view_array<T, CblasColMajor>(_vdata, this->_mvalid, this->_n);
    cml::blas_gemv(hdl, OpToCublasOp(trans), alpha, &A, &x_vec, beta, &y_vec);
  }
  CUDA_CHECK_ERR();

  return 0;
}

  // col-major order (fortran) A, but still print as row major
template <typename T>
void printMatrix(int m, int n, const T*A, int lda, const char* name)
{
  printf("rows=%d cols=%d lda=%d\n",m,n,lda);
  for(int row = 0 ; row < m ; row++){
    for(int col = 0 ; col < n ; col++){
      T Areg = A[row + col*lda];
      printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
    }
  }
}

  // row-major order (c) A printed as row major
template <typename T>
void printMatrix2(int m, int n, const T*A, int lda, const char* name)
{
  printf("rows=%d cols=%d lda=%d\n",m,n,lda);
  for(int row = 0 ; row < m ; row++){
    for(int col = 0 ; col < n ; col++){
      T Areg = A[col + row*n];
      printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
    }
  }
}

/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include svd_example.cpp 
 *   g++ -fopenmp -o a.out svd_example.o -L/usr/local/cuda/lib64 -lcudart -lcublas -lcusolver
 *
 */

  inline hipsolverStatus_t cusolverDngesvd (    hipsolverHandle_t handle,    signed char jobu,    signed char jobvt,    int m,    int n,    float *A,    int lda,    float *S,    float *U,    int ldu,    float *VT,    int ldvt,    float *work,    int lwork,    float *rwork,    int *devInfo){
  return(hipsolverDnSgesvd(handle,    jobu,    jobvt,    m,    n,    A,    lda,    S,    U,    ldu,    VT,    ldvt,    work,    lwork,    rwork,    devInfo));
}
  
inline hipsolverStatus_t cusolverDngesvd (    hipsolverHandle_t handle,    signed char jobu,    signed char jobvt,    int m,    int n,    double *A,    int lda,    double *S,    double *U,    int ldu,    double *VT,    int ldvt,    double *work,    int lwork,    double *rwork,    int *devInfo){
  return(hipsolverDnDgesvd(handle,    jobu,    jobvt,    m,    n,    A,    lda,    S,    U,    ldu,    VT,    ldvt,    work,    lwork,    rwork,    devInfo));
}

inline hipblasStatus_t cublasgemm(hipblasHandle_t handle,                           hipblasOperation_t transa, hipblasOperation_t transb,                           int m, int n, int k,                           const float           *alpha,                           const float           *A, int lda,                           const float           *B, int ldb,                           const float           *beta,                           float           *C, int ldc){
  return(hipblasSgemm(handle,                           transa, transb,                           m, n, k,                   alpha,                                A, lda,                           B, ldb,                      beta,                       C, ldc));
  

}

  
inline hipblasStatus_t cublasgemm(hipblasHandle_t handle,                           hipblasOperation_t transa, hipblasOperation_t transb,                           int m, int n, int k,                           const double          *alpha,                           const double          *A, int lda,                           const double          *B, int ldb,                           const double          *beta,                           double          *C, int ldc){
  return(hipblasDgemm(handle,                           transa, transb,                           m, n, k,                   alpha,                                A, lda,                           B, ldb,                      beta,                       C, ldc));

}

inline hipblasStatus_t cublasdgmm(hipblasHandle_t handle,
                                 hipblasSideMode_t mode, 
                                 int m, 
                                 int n,
                                 const float *A, 
                                 int lda,
                                 const float *x, 
                                 int incx,
                                 float *C, 
                                 int ldc){
  
  return(hipblasSdgmm(handle,
                     mode, 
                     m, 
                     n,
                     A, 
                     lda,
                     x, 
                     incx,
                     C, 
                     ldc));

}
inline hipblasStatus_t cublasdgmm(hipblasHandle_t handle,
                                 hipblasSideMode_t mode, 
                                 int m, 
                                 int n,
                                 const double *A, 
                                 int lda,
                                 const double *x, 
                                 int incx,
                                 double *C, 
                                 int ldc){
  
  return(hipblasDdgmm(handle,
                     mode, 
                     m, 
                     n,
                     A, 
                     lda,
                     x, 
                     incx,
                     C, 
                     ldc));

}

inline hipblasStatus_t cublasnrm2(hipblasHandle_t handle, 
                                 int n, 
                                 const double *x, 
                                 int incx, 
                                 double *result){
    return(hipblasDnrm2(handle, 
                      n, 
                      x, 
                      incx, 
                      result));
    
  }
    
inline hipblasStatus_t cublasnrm2(hipblasHandle_t handle, 
                                 int n, 
                                 const float *x, 
                                 int incx, 
                                 float *result){
    return(hipblasSnrm2(handle, 
                      n, 
                      x, 
                      incx, 
                      result));
    
}
    
// // Each block transposes/copies a tile of TILE_DIM x TILE_DIM elements
// // using TILE_DIM x BLOCK_ROWS threads, so that each thread transposes
// // TILE_DIM/BLOCK_ROWS elements.  TILE_DIM must be an integral multiple of BLOCK_ROWS

#define TILE_DIM    16
#define BLOCK_ROWS  16

//   __global__ void transposeNaive(float *odata, float* idata,
//                                  int width, int height)
//   {
//     int xIndex = blockIdx.x*TILE_DIM + threadIdx.x;
//     int yIndex = blockIdx.y*TILE_DIM + threadIdx.y;
//     int index_in = xIndex + width * yIndex;
//     int index_out = yIndex + height * xIndex;
//     for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
//       odata[index_out+i] = idata[index_in+i*width];
//     }
//   }
//   __global__ void transposeNaive(double *odata, double* idata,
//                                  int width, int height)
//   {
//     int xIndex = blockIdx.x*TILE_DIM + threadIdx.x;
//     int yIndex = blockIdx.y*TILE_DIM + threadIdx.y;
//     int index_in = xIndex + width * yIndex;
//     int index_out = yIndex + height * xIndex;
//     for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
//       odata[index_out+i] = idata[index_in+i*width];
//     }
//   } 
//   __global__ void transposeCoalesced(float *odata,
//                                      float *idata, int width, int height)
//   {
//     __shared__ float tile[TILE_DIM][TILE_DIM];
//     int xIndex = blockIdx.x*TILE_DIM + threadIdx.x;
//     int yIndex = blockIdx.y*TILE_DIM + threadIdx.y;
//     int index_in = xIndex + (yIndex)*width;
//     xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
//     yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
//     int index_out = xIndex + (yIndex)*height;
//     for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
//       tile[threadIdx.y+i][threadIdx.x] =
//         idata[index_in+i*width];
//     }

//     __syncthreads();

//     for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
//       odata[index_out+i*height] =
//         tile[threadIdx.x][threadIdx.y+i];
//     }
//   } 
//   __global__ void transposeCoalesced(double *odata,
//                                      double *idata, int width, int height)
//   {
//     __shared__ double tile[TILE_DIM][TILE_DIM];
//     int xIndex = blockIdx.x*TILE_DIM + threadIdx.x;
//     int yIndex = blockIdx.y*TILE_DIM + threadIdx.y;
//     int index_in = xIndex + (yIndex)*width;
//     xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
//     yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
//     int index_out = xIndex + (yIndex)*height;
//     for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
//       tile[threadIdx.y+i][threadIdx.x] =
//         idata[index_in+i*width];
//     }

//     __syncthreads();

//     for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
//       odata[index_out+i*height] =
//         tile[threadIdx.x][threadIdx.y+i];
//     }
//   }

// in-place transpose for row-major matrix on device of A[m][n]
void cudaintranspose(float *odata, float *idata, int m, int n){

  hipError_t cudaStat1 = hipSuccess;
  cudaStat1 = hipMemcpy(odata, idata, sizeof(float)*m*n, hipMemcpyDeviceToDevice);
  assert(hipSuccess == cudaStat1);
  
  float const alpha(1.0);
  float const beta(0.0);
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSgeam( handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &alpha, idata, n, &beta, idata, m, odata, m );
  hipblasDestroy(handle);
}
void cudaintranspose(double *odata, double *idata, int m, int n){

  hipError_t cudaStat1 = hipSuccess;
  cudaStat1 = hipMemcpy(odata, idata, sizeof(double)*m*n, hipMemcpyDeviceToDevice);
  assert(hipSuccess == cudaStat1);
  
  double const alpha(1.0);
  double const beta(0.0);
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDgeam( handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &alpha, idata, n, &beta, idata, m, odata, m );
  hipblasDestroy(handle);
}
#define MIN(a,b) ((a)<(b) ? (a) : (b))

template <typename T>
int MatrixDense<T>::svd1(void) {
  fprintf(stderr,"begin svd inside0\n"); fflush(stderr); fflush(stdout);
  DEBUG_ASSERT(this->_done_init);
  if (!this->_done_init)
    Init();

  fprintf(stderr,"begin svd inside\n"); fflush(stderr); fflush(stdout);
  hipsolverHandle_t cusolverH = NULL;
  hipblasHandle_t cublasH = NULL;
  hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
  hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
  hipError_t cudaStat1 = hipSuccess;
  hipError_t cudaStat2 = hipSuccess;
  hipError_t cudaStat3 = hipSuccess;
  hipError_t cudaStat4 = hipSuccess;
  hipError_t cudaStat5 = hipSuccess;
  hipError_t cudaStat6 = hipSuccess;
  int m = this->_m;
  int n = this->_n;
  //    const int m = this->_m;
  //    const int n = this->_n;
  int lda = m;
  /*       | 1 2  |
   *   A = | 4 5  |
   *       | 2 1  |
   */


    
  unsigned char ord='r'; // TODO; should be inputted
      
  // original device vector
  T *d_A0;
  d_A0 = this->_data;

  // device vectors
  T *d_A = NULL;
  T *d_S = NULL;
  T *d_U = NULL;
  T *d_VT = NULL;
  int *devInfo = NULL;
  T *d_work = NULL;
  T *d_rwork = NULL;
  T *d_W = NULL;  // W = S*VT

  int lwork = 0;
  int info_gpu = 0;
  const T h_one = 1;
  const T h_minus_one = -1;

  double t0 = timer<double>();
  // step 1: create cusolverDn/cublas handle
  cusolver_status = hipsolverDnCreate(&cusolverH);
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

  cublas_status = hipblasCreate(&cublasH);
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);


  fprintf(stderr,"HERE1\n"); fflush(stderr); fflush(stdout);

  // step 2: copy A to device
  //    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(T)*lda*n);

  // svd destroys d_A, so make copy for testing error // OPTMARK
  cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(T)*lda*n);
  assert(hipSuccess == cudaStat1);
  cudaStat1 = hipMemcpy(d_A, d_A0, sizeof(T)*lda*n, hipMemcpyDeviceToDevice);
  assert(hipSuccess == cudaStat1);
  cudaStat1 = hipDeviceSynchronize();
  assert(hipSuccess == cudaStat1);


  int ldu=m; //lda;
  int ldureal=n; // actual storage
  int ldvt=n;
  if(ord=='r'){
    // transpose
    // execution configuration parameters
    //dim3 grid(n/TILE_DIM, lda/TILE_DIM), threads(TILE_DIM,BLOCK_ROWS);
    //      transposeCoalesced<<<grid, threads>>>(d_A, d_A0, n, lda);
    //    transposeNaive<<<grid, threads>>>(d_A, d_A0, n, lda);
    cudaintranspose(d_A,d_A0,m,n); // OPTMARK
    cudaStat1 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);

    // below debug only for printMatrix2 to view, shouldn't actually swap for use.
    if(0){
      int temp=m;
      m=n;
      n=temp;
      
      lda=m;
      ldu=m; //lda;
      ldureal=n; // actual storage
      ldvt=n;
    }
  }
  else{
    d_A = d_A0;
  }



  
  fprintf(stderr,"HERE PRE\n"); fflush(stderr); fflush(stdout);
  // old host side vectors
  //    T A[lda*n] = { 1.0, 4.0, 2.0, 2.0, 5.0, 1.0};
  //    GpuData<T> *info_A   = reinterpret_cast<GpuData<T>*>(this->_info); // cast from void to GpuData
  //    T *A = const_cast<T*>(info_A->orig_data);
#if(0)
  T A[lda*n]; // for debug
  T U[ldureal*m]; // m-by-m unitary matrix 
  T VT[ldvt*n];  // n-by-n unitary matrix
  T S[MIN(n,m)]; // singular value
#endif
  //    T S_exact[n] = {7.065283497082729, 1.040081297712078};
  fprintf(stderr,"HERE POST\n"); fflush(stderr); fflush(stdout);
  
  // now d_A has column-major order matrix
  fprintf(stderr,"HERE2\n"); fflush(stderr); fflush(stdout);
    
#if(0) // debug
    cudaStat1 = hipMemcpy(A, d_A, sizeof(T)*lda*n, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    printf("A = (matlab base-1)\n");
    printMatrix(m, n, A, lda, "A");
    printf("=====\n");
    printf("A = (matlab base-1)\n");
    printMatrix2(m, n, A, lda, "A");
    printf("=====\n");
#endif

  fprintf(stderr,"HERE3\n"); fflush(stderr); fflush(stdout);

  cudaStat2 = hipMalloc ((void**)&d_S  , sizeof(T)*MIN(n,m));
  cudaStat3 = hipMalloc ((void**)&d_U  , sizeof(T)*ldureal*m);
  cudaStat4 = hipMalloc ((void**)&d_VT , sizeof(T)*ldvt*n);
  cudaStat5 = hipMalloc ((void**)&devInfo, sizeof(int));
  cudaStat6 = hipMalloc ((void**)&d_W  , sizeof(T)*lda*n);
  //    assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);
  assert(hipSuccess == cudaStat3);
  assert(hipSuccess == cudaStat4);
  assert(hipSuccess == cudaStat5);
  assert(hipSuccess == cudaStat6);

  // host->device
  //    cudaStat1 = hipMemcpy(d_A, A, sizeof(T)*lda*n, hipMemcpyHostToDevice);
  //    assert(hipSuccess == cudaStat1);

  // step 3: query working space of SVD
  //The dense matrices are assumed to be stored in column-major order in memory.
  cusolver_status = hipsolverDnDgesvd_bufferSize(
                                                cusolverH,
                                                m,
                                                n,
                                                &lwork );
  assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

  cudaStat1 = hipMalloc((void**)&d_work , sizeof(T)*lwork);
  assert(hipSuccess == cudaStat1);
  double t1 = timer<double>();
  fprintf(stderr,"SVD init: %g\n",t1-t0); fflush(stderr); fflush(stdout);

  // step 4: compute SVD
  double t0c = timer<double>();
  signed char jobu = 'A'; // all m columns of U
  signed char jobvt = 'A'; // all n columns of VT
  cusolver_status = cusolverDngesvd(
                                    cusolverH,
                                    jobu,
                                    jobvt,
                                    m,
                                    n,
                                    d_A,
                                    lda,
                                    d_S,
                                    d_U,
                                    ldu,
                                    d_VT,
                                    ldvt,
                                    d_work,
                                    lwork,
                                    d_rwork,
                                    devInfo);


  cudaStat4 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
  printf("after gesvd: info_gpu = %d\n", info_gpu); fflush(stdout);
  assert(0 == info_gpu);
  printf("=====\n"); fflush(stdout);

  cudaStat1 = hipDeviceSynchronize();
  assert(hipSuccess == cudaStat1);
  fprintf(stderr,"BAD: %d\n",cusolver_status); fflush(stderr);
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
  double t1c = timer<double>();
  fprintf(stderr,"SVD compute: %g\n",t1-t0); fflush(stderr); fflush(stdout);



#if(0)
    /////////////////////////
    // Copy solution device->host
    double t0h = timer<double>();
    cudaStat1 = hipMemcpy(U , d_U , sizeof(T)*ldureal*m, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(VT, d_VT, sizeof(T)*ldvt*n, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(S , d_S , sizeof(T)*MIN(n,m), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
  

  if(0){ // debug
    printf("S = (matlab base-1)\n");
    printMatrix(n, 1, S, lda, "S");
    printf("=====\n");
      
    printf("U = (matlab base-1)\n");
    printMatrix(m, m, U, ldureal, "U");
    printf("=====\n");
      
    printf("VT = (matlab base-1)\n");
    printMatrix(n, n, VT, ldvt, "VT");
    printf("=====\n");

    /////////////////////////
    // measure error of singular value
    //      T ds_sup = 0;
    //      for(int j = 0; j < n; j++){
    //        T err = fabs( S[j] - S_exact[j] );
    //        ds_sup = (ds_sup > err)? ds_sup : err;
    //      }
    //      printf("|S - S_exact| = %E \n", ds_sup);
  }
  double t1h = timer<double>();
  fprintf(stderr,"SVD back to host: %g\n",t1h-t0h); fflush(stderr); fflush(stdout);
#endif

  /////////////////////////
  // now check
  double t0c1 = timer<double>();
  // step 5: |A - U*S*VT|
  // W = S*VT
  cublas_status = cublasdgmm(
                             cublasH,
                             HIPBLAS_SIDE_LEFT,
                             n,
                             n,
                             d_VT,
                             ldvt,
                             d_S,
                             1,
                             d_W,
                             lda);
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
  double t1c1 = timer<double>();
  fprintf(stderr,"SVD check1: %g\n",t1c1-t0c1); fflush(stderr); fflush(stdout);


 
  // A := -U*W + A
  double t0c2 = timer<double>();
  cudaStat1 = hipMemcpy(d_A, d_A0, sizeof(T)*lda*n, hipMemcpyDeviceToDevice); // copy because original d_A was destroyed
  assert(hipSuccess == cudaStat1);
  cublas_status = cublasgemm(
                             cublasH,
                             HIPBLAS_OP_N, // U
                             HIPBLAS_OP_N, // W
                             m, // number of rows of A
                             n, // number of columns of A
                             n, // number of columns of U 
                             &h_minus_one, /* host pointer */
                             d_U, // U
                             ldu,
                             d_W, // W
                             lda,
                             &h_one, /* hostpointer */
                             d_A,
                             lda);
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
  double t1c2 = timer<double>();
  fprintf(stderr,"SVD check2: %g\n",t1c2-t0c2); fflush(stderr); fflush(stdout);

  double t0c3 = timer<double>();
  T dR_fro = 0.0;
  cublas_status = cublasnrm2(
                             cublasH, lda*n, d_A, 1, &dR_fro);
  assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

  printf("|A - U*S*VT| = %E \n", dR_fro); fflush(stdout);
  double t1c3 = timer<double>();
  fprintf(stderr,"SVD check3: %g\n",t1c3-t0c3); fflush(stderr); fflush(stdout);

    
  // free resources
  double t0f = timer<double>();
  //if (d_A    ) hipFree(d_A);
  if (d_S    ) hipFree(d_S);
  if (d_U    ) hipFree(d_U);
  if (d_VT   ) hipFree(d_VT);
  if (devInfo) hipFree(devInfo);
  if (d_work ) hipFree(d_work);
  if (d_rwork) hipFree(d_rwork);
  if (d_W    ) hipFree(d_W);

  if (cublasH ) hipblasDestroy(cublasH);
  if (cusolverH) hipsolverDnDestroy(cusolverH);
  //    hipDeviceReset();
  double t1f = timer<double>();
  fprintf(stderr,"SVD free: %g\n",t1f-t0f); fflush(stderr); fflush(stdout);

  fprintf(stderr,"end svd inside\n"); fflush(stderr); fflush(stdout);

  return 0;
}

// Equilibration (precondition) matrix using Sinkhorn Knopp method wrapped to allow any norm
// See https://arxiv.org/pdf/1610.03871.pdf for more information
template <typename T>
int MatrixDense<T>::Equil(bool equillocal) {
  DEBUG_ASSERT(this->_done_init);
  if (!this->_done_init)
    return 1;

  if (this->_done_equil) return 0;
  else this->_done_equil=1;
  
  CUDACHECK(hipSetDevice(_wDev));

  // Extract cublas handle from _info.
  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  hipblasHandle_t hdl = info->handle;

  T *d = _de;
  T *e = d + this->_m;

  
  // Number of elements in matrix.
  size_t num_el = this->_m * this->_n;

  // Create bit-vector with signs of entries in A and then let A = f(A),
  // where f = |A| or f = |A|.^2.
  unsigned char *sign;
  size_t num_sign_bytes = (num_el + 7) / 8;
  hipMalloc(&sign, num_sign_bytes);
  CUDA_CHECK_ERR();

  size_t num_chars = num_el / 8;
  size_t grid_size = cml::calc_grid_dim(num_chars, cml::kBlockSize);
  if(equillocal){
    // Fill sign bits, assigning each thread a multiple of 8 elements.
    if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
      __SetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
                                                SquareF<T>());
    } else {
      __SetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
                                                AbsF<T>());
    }
    wrapcudaDeviceSynchronize();
    CUDA_CHECK_ERR();

    // If numel(A) is not a multiple of 8, then we need to set the last couple
    // of sign bits too. 
    if (num_el > num_chars * 8) {
      if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
        __SetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
                                  num_el - num_chars * 8, SquareF<T>());
      } else {
        __SetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
                                  num_el - num_chars * 8, AbsF<T>());
      }
      wrapcudaDeviceSynchronize();
      CUDA_CHECK_ERR();
    }
  }
  
  // Perform Sinkhorn-Knopp equilibration to obtain a doubly stochastic matrix.
  SinkhornKnopp(this, d, e, equillocal);
  wrapcudaDeviceSynchronize();

  if(equillocal){
    // Transform A = sign(A) .* sqrt(A) if 2-norm equilibration was performed,
    // or A = sign(A) .* A if the 1-norm was equilibrated.
    if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
      __UnSetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
                                                  SqrtF<T>());
    } else {
      __UnSetSign<<<grid_size, cml::kBlockSize>>>(_data, sign, num_chars,
                                                  IdentityF<T>());
    }
    wrapcudaDeviceSynchronize();
    CUDA_CHECK_ERR();

    // Deal with last few entries if num_el is not a multiple of 8.
    if (num_el > num_chars * 8) {
      if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
        __UnSetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
                                    num_el - num_chars * 8, SqrtF<T>());
      } else {
        __UnSetSignSingle<<<1, 1>>>(_data + num_chars * 8, sign + num_chars, 
                                    num_el - num_chars * 8, IdentityF<T>());
      }
      wrapcudaDeviceSynchronize();
      CUDA_CHECK_ERR();
    }
  }
  
  // Compute D := sqrt(D), E := sqrt(E), if 2-norm was equilibrated.
  if (kNormEquilibrate == kNorm2 || kNormEquilibrate == kNormFro) {
    thrust::transform(thrust::device_pointer_cast(d),
        thrust::device_pointer_cast(d + this->_m),
        thrust::device_pointer_cast(d), SqrtF<T>());
    thrust::transform(thrust::device_pointer_cast(e),
        thrust::device_pointer_cast(e + this->_n),
        thrust::device_pointer_cast(e), SqrtF<T>());
    wrapcudaDeviceSynchronize();
    CUDA_CHECK_ERR();
  }

  // Compute A := D * A * E.
  MultDiag(d, e, this->_m, this->_n, _ord, _data);
  wrapcudaDeviceSynchronize();
  CUDA_CHECK_ERR();

  // Scale A to have norm of 1 (in the kNormNormalize norm).
  T normA = NormEst(hdl, kNormNormalize, *this);
  CUDA_CHECK_ERR();
  wrapcudaDeviceSynchronize();
  cml::vector<T> a_vec = cml::vector_view_array(_data, num_el);
  cml::vector_scale(&a_vec, 1 / normA);
  wrapcudaDeviceSynchronize();

  // Scale d and e to account for normalization of A.
  cml::vector<T> d_vec = cml::vector_view_array<T>(d, this->_m);
  cml::vector<T> e_vec = cml::vector_view_array<T>(e, this->_n);
  cml::vector_scale(&d_vec, 1 / sqrt(normA));
  cml::vector_scale(&e_vec, 1 / sqrt(normA));
  wrapcudaDeviceSynchronize();

  DEBUG_PRINTF("norm A = %e, normd = %e, norme = %e\n", normA,
      cml::blas_nrm2(hdl, &d_vec), cml::blas_nrm2(hdl, &e_vec));

  hipFree(sign);
  CUDA_CHECK_ERR();

  return 0;
}

// This example computes several statistical properties of a data
// series in a single reduction.  The algorithm is described in detail here:
// http://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Parallel_algorithm
//
// Thanks to Joseph Rhoads for contributing this example
// structure used to accumulate the moments and other
// statistical properties encountered so far.
template <typename T>
struct summary_stats_data
{
  T n;
  T min;
  T max;
  T mean;
  T M2;
  T M3;
  T M4;

  // initialize to the identity element
  void initialize()
  {
    n = mean = M2 = M3 = M4 = 0;
    min = std::numeric_limits<T>::max();
    max = std::numeric_limits<T>::min();
  }

  T variance()   { return M2 / (n - 1); }
  T variance_n() { return M2 / n; }
  T skewness()   { return std::sqrt(n) * M3 / std::pow(M2, (T) 1.5); }
  T kurtosis()   { return n * M4 / (M2 * M2); }
};

  // stats_unary_op is a functor that takes in a value x and
  // returns a variace_data whose mean value is initialized to x.
template <typename T>
struct summary_stats_unary_op
{
  __host__ __device__
  summary_stats_data<T> operator()(const T& x) const
  {
    summary_stats_data<T> result;
    result.n    = 1;
    result.min  = x;
    result.max  = x;
    result.mean = x;
    result.M2   = 0;
    result.M3   = 0;
    result.M4   = 0;

    return result;
  }
};

  // summary_stats_binary_op is a functor that accepts two summary_stats_data
  // structs and returns a new summary_stats_data which are an
  // approximation to the summary_stats for
  // all values that have been agregated so far
template <typename T>
struct summary_stats_binary_op
  : public thrust::binary_function<const summary_stats_data<T>&,
                                   const summary_stats_data<T>&,
                                   summary_stats_data<T> >
{
  __host__ __device__
  summary_stats_data<T> operator()(const summary_stats_data<T>& x, const summary_stats_data <T>& y) const
  {
    summary_stats_data<T> result;

    // precompute some common subexpressions
    T n  = x.n + y.n;
    T n2 = n  * n;
    T n3 = n2 * n;

    T delta  = y.mean - x.mean;
    T delta2 = delta  * delta;
    T delta3 = delta2 * delta;
    T delta4 = delta3 * delta;

    //Basic number of samples (n), min, and max
    result.n   = n;
    result.min = thrust::min(x.min, y.min);
    result.max = thrust::max(x.max, y.max);

    result.mean = x.mean + delta * y.n / n;

    result.M2  = x.M2 + y.M2;
    result.M2 += delta2 * x.n * y.n / n;

    result.M3  = x.M3 + y.M3;
    result.M3 += delta3 * x.n * y.n * (x.n - y.n) / n2;
    result.M3 += (T) 3.0 * delta * (x.n * y.M2 - y.n * x.M2) / n;

    result.M4  = x.M4 + y.M4;
    result.M4 += delta4 * x.n * y.n * (x.n * x.n - x.n * y.n + y.n * y.n) / n3;
    result.M4 += (T) 6.0 * delta2 * (x.n * x.n * y.M2 + y.n * y.n * x.M2) / n2;
    result.M4 += (T) 4.0 * delta * (x.n * y.M3 - y.n * x.M3) / n;

    return result;
  }
};

template <typename Iterator>
void print_range(const std::string& name, Iterator first, Iterator last)
{
  typedef typename std::iterator_traits<Iterator>::value_type T;

  std::cout << name << ": ";
  thrust::copy(first, last, std::ostream_iterator<T>(std::cout, " "));
  std::cout << "\n";
}

template<typename T>
struct absolute_value : public thrust::unary_function<T,T>
{
  __host__ __device__ T operator()(const T &x) const
  {
    return x < T(0) ? -x : x;
  }
};
// --- Operator for testing nan values
template<typename T>
struct isnan_test {
    __host__ __device__ bool operator()(const T a) const {
        return isnan(a) || isinf(a);
    }
};

// check properties of input data
template <typename T>
int MatrixDense<T>::Stats(int intercept, T *min, T *max, T *mean, T *var, T *sd, T *skew, T *kurt, T &lambda_max0)
{
  CUDACHECK(hipSetDevice(_wDev));

  if(_data!=NULL) {// check for nan or inf in data
      thrust::device_ptr<T> begin = thrust::device_pointer_cast(_data);
	  thrust::device_ptr<T> end = thrust::device_pointer_cast(_data+this->_m*this->_n);
	  bool h_result = thrust::transform_reduce(begin, end, isnan_test<T>(), 0, thrust::plus<bool>());
    if(h_result==true){
		  fprintf(stderr,"Data matrix (trainX) has nan/inf or missing was not encoded\n");
	  	  fflush(stderr);
	  	  exit(1);
	  }
  }
  if(_datay!=NULL) {// check for nan or inf in data
      thrust::device_ptr<T> begin = thrust::device_pointer_cast(_datay);
	  thrust::device_ptr<T> end = thrust::device_pointer_cast(_datay+this->_m);
	  bool h_result = thrust::transform_reduce(begin, end, isnan_test<T>(), 0, thrust::plus<bool>());
      if(h_result==true){
		  fprintf(stderr,"Data training predictions/labels (trainY) has nan/inf or missing was not encoded\n");
	  	  fflush(stderr);
	  	  exit(1);
	  }
  }
  if(_vdata!=NULL) {// check for nan or inf in data
	  thrust::device_ptr<T> begin = thrust::device_pointer_cast(_vdata);
	  thrust::device_ptr<T> end = thrust::device_pointer_cast(_vdata+this->_mvalid*this->_n);
	  bool h_result = thrust::transform_reduce(begin, end, isnan_test<T>(), 0, thrust::plus<bool>());
	  if(h_result==true){
		  fprintf(stderr,"Validation Data matrix (validX) has nan/inf or missing was not encoded\n");
	  	  fflush(stderr);
	  	  exit(1);
	  }
  }
  if(_vdatay!=NULL) {// check for nan or inf in data
	  thrust::device_ptr<T> begin = thrust::device_pointer_cast(_vdatay);
	  thrust::device_ptr<T> end = thrust::device_pointer_cast(_vdatay+this->_mvalid);
	  bool h_result = thrust::transform_reduce(begin, end, isnan_test<T>(), 0, thrust::plus<bool>());
	  if(h_result==true){
		  fprintf(stderr,"Validation Data training predictions/labels (validY) has nan/inf or missing was not encoded\n");
	  	  fflush(stderr);
	  	  exit(1);
	  }
  }
  if(_weight!=NULL) {// check for nan or inf in data
      thrust::device_ptr<T> begin = thrust::device_pointer_cast(_weight);
	  thrust::device_ptr<T> end = thrust::device_pointer_cast(_weight+this->_m);
	  bool h_result = thrust::transform_reduce(begin, end, isnan_test<T>(), 0, thrust::plus<bool>());
      if(h_result==true){
		  fprintf(stderr,"Weight Training Data has nan/inf or missing was not encoded\n");
	  	  fflush(stderr);
	  	  exit(1);
	  }
  }

  // nothing else to do if _datay==NULL
  if(_datay==NULL) return(0);

  // setup arguments
  summary_stats_unary_op<T>  unary_op;
  summary_stats_binary_op<T> binary_op;
  summary_stats_data<T>      init;
  
  init.initialize();
  int len=0;
  
  // cast GPU pointer as thrust pointer
  thrust::device_ptr<T> dataybegin=thrust::device_pointer_cast(_datay);
  len=this->_m;
  thrust::device_ptr<T> datayend=thrust::device_pointer_cast(_datay+len);
  

  // compute summary statistics
  summary_stats_data<T> resulty = thrust::transform_reduce(dataybegin, datayend, unary_op, init, binary_op);

  min[0]=resulty.min;
  max[0]=resulty.max;
  mean[0]=resulty.mean;
  var[0]=resulty.variance();
  sd[0]=std::sqrt(resulty.variance_n());
  skew[0]=resulty.skewness();
  kurt[0]=resulty.kurtosis();

#ifdef DEBUG
  std::cout <<"******Summary Statistics of Response Train*****"<<std::endl;
//  print_range("The data", dataybegin, datayend);
  std::cout <<"Count              : "<< resulty.n << std::endl;
  std::cout <<"Minimum            : "<< min[0]<<std::endl;
  std::cout <<"Maximum            : "<< max[0]<<std::endl;
  std::cout <<"Mean               : "<< mean[0]<< std::endl;
  std::cout <<"Variance           : "<< var[0]<< std::endl;
  std::cout <<"Standard Deviation : "<< sd[0]<< std::endl;
  std::cout <<"Skewness           : "<< skew[0]<< std::endl;
  std::cout <<"Kurtosis           : "<< kurt[0]<< std::endl;
#endif

  // cast GPU pointer as thrust pointer
  thrust::device_ptr<T> vdataybegin=thrust::device_pointer_cast(_vdatay);
  len=this->_mvalid;
  thrust::device_ptr<T> vdatayend=thrust::device_pointer_cast(_vdatay+len);

  // compute summary statistics
  summary_stats_data<T> vresulty = thrust::transform_reduce(vdataybegin, vdatayend, unary_op, init, binary_op);

  
  min[1]=vresulty.min;
  max[1]=vresulty.max;
  mean[1]=vresulty.mean;
  var[1]=vresulty.variance();
  sd[1]=std::sqrt(vresulty.variance_n());
  skew[1]=vresulty.skewness();
  kurt[1]=vresulty.kurtosis();

#ifdef DEBUG
  std::cout <<"******Summary Statistics of Response Valid*****"<<std::endl;
  //  print_range("The data", vdataybegin, vdatayend);
  std::cout <<"Count              : "<< vresulty.n << std::endl;
  std::cout <<"Minimum            : "<< min[1]<<std::endl;
  std::cout <<"Maximum            : "<< max[1]<<std::endl;
  std::cout <<"Mean               : "<< mean[1]<< std::endl;
  std::cout <<"Variance           : "<< var[1]<< std::endl;
  std::cout <<"Standard Deviation : "<< sd[1]<< std::endl;
  std::cout <<"Skewness           : "<< skew[1]<< std::endl;
  std::cout <<"Kurtosis           : "<< kurt[1]<< std::endl;
#endif

  if(1){ // normal usage
    // Get Cublas handle
    GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
    hipblasHandle_t hdl = info->handle;

    // Set up views for raw vectors.
    cml::vector<T> y_vec = cml::vector_view_array(_datay, this->_m); // b
    cml::vector<T> weight_vec;
    if(_weight) weight_vec = cml::vector_view_array(_weight, this->_m); // weight
    else{
      weight_vec = cml::vector_calloc<T>(this->_m); // weight make up
      cml::vector_add_constant(&weight_vec, static_cast<T>(1.0)); // make unity weights
    }
    cml::vector<T> ytemp = cml::vector_calloc<T>(this->_m); // b
    cml::vector<T> xtemp = cml::vector_calloc<T>(this->_n); // x
    cml::vector_memcpy(&ytemp, &y_vec); // y_vec->ytemp
    cml::vector_add_constant(&ytemp, -static_cast<T>(intercept)*mean[0]); // ytemp -> ytemp - intercept*mean[0]
    cml::vector_mul(&ytemp,&weight_vec); // ytemp*weight -> ytemp

    // Compute A^T . b
    if (_ord == MatrixDense<T>::ROW) {
      const cml::matrix<T, CblasRowMajor> A = cml::matrix_view_array<T, CblasRowMajor>(_data, this->_m, this->_n); // just view
      cml::blas_gemv(hdl, HIPBLAS_OP_T, static_cast<T>(1.), &A, &ytemp, static_cast<T>(0.), &xtemp); // A.ytemp -> xtemp
    }
    else{
      const cml::matrix<T, CblasColMajor> A = cml::matrix_view_array<T, CblasColMajor>(_data, this->_m, this->_n); // just view
      cml::blas_gemv(hdl, HIPBLAS_OP_T, static_cast<T>(1.), &A, &ytemp, static_cast<T>(0.), &xtemp); // A.ytemp -> xtemp
    }

    thrust::device_ptr<T> dev_ptr = thrust::device_pointer_cast(&xtemp.data[0]);

    lambda_max0 = thrust::transform_reduce(thrust::device,
                                           dev_ptr, dev_ptr + this->_n-intercept,
                                           absolute_value<T>(),
                                           static_cast<T>(0.0),
                                           thrust::maximum<T>());
  }
  else{
    lambda_max0 = 7000; // test
  }
  CUDA_CHECK_ERR();

  return 0;
}

  

////////////////////////////////////////////////////////////////////////////////
/////////////////////// Equilibration Helpers //////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
namespace {

// Estimates norm of A. norm_type should either be kNorm2 or kNormFro.
template <typename T>
T NormEst(hipblasHandle_t hdl, NormTypes norm_type, const MatrixDense<T>& A) {
  switch (norm_type) {
    case kNorm2: {
      return Norm2Est(hdl, &A);
    }
    case kNormFro: {
      const cml::vector<T> a = cml::vector_view_array(A.Data(),
          A.Rows() * A.Cols());
      return cml::blas_nrm2(hdl, &a) / std::sqrt(std::min(A.Rows(), A.Cols()));
    }
    case kNorm1:
      // 1-norm normalization doens't make make sense since it treats rows and
      // columns differently.
    default:
      ASSERT(false);
      return static_cast<T>(0.);
  }
}


  

// Performs A := D * A * E for A in row major
template <typename T>
void __global__ __MultRow(size_t m, size_t n, const T *d, const T *e, T *data) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t t = tid; t < m * n; t += gridDim.x * blockDim.x)
    data[t] *= d[t / n] * e[t % n];
}

// Performs A := D * A * E for A in col major
template <typename T>
void __global__ __MultCol(size_t m, size_t n, const T *d, const T *e, T *data) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t t = tid; t < m * n; t += gridDim.x * blockDim.x)
    data[t] *= d[t % m] * e[t / m];
}

template <typename T>
void MultDiag(const T *d, const T *e, size_t m, size_t n,
              typename MatrixDense<T>::Ord ord, T *data) {
  if (ord == MatrixDense<T>::ROW) {
    size_t grid_dim_row = cml::calc_grid_dim(m * n, cml::kBlockSize);
    __MultRow<<<grid_dim_row, cml::kBlockSize>>>(m, n, d, e, data);
  } else {
    size_t grid_dim_row = cml::calc_grid_dim(m * n, cml::kBlockSize);
    __MultCol<<<grid_dim_row, cml::kBlockSize>>>(m, n, d, e, data);
  }
}

}  // namespace

// Explicit template instantiation.
#if !defined(H2O4GPU_DOUBLE) || H2O4GPU_DOUBLE==1
template class MatrixDense<double>;
#endif

#if !defined(H2O4GPU_SINGLE) || H2O4GPU_SINGLE==1
template class MatrixDense<float>;
#endif




  // upload data function.  Uploads to a single GPU.
  // mimics otherwise similar MatrixDense constructor, but has no destruction of uploaded data pointers
template <typename T>
int makePtr_dense(int sharedA, int me, int wDev, size_t m, size_t n, size_t mValid, const char ord, const T *data, const T *datay, const T *vdata, const T *vdatay, const T *weight, T **_data, T **_datay, T **_vdata, T **_vdatay, T **_weight){
    checkwDev(wDev);
    CUDACHECK(hipSetDevice(wDev));

    DEBUG_FPRINTF(stderr,"makePtr_dense: %d\n",0);

#ifdef DEBUG
    //    CUDACHECK(hipSetDeviceFlags(hipDeviceMapHost)); // TODO: MapHostMemory
    hipDeviceProp_t props;
    CUDACHECK(hipGetDeviceProperties(&props, wDev));
    DEBUG_FPRINTF(stderr,"Using: Compute %d.%d CUDA device: [%s] with id=%2d\n", props.major, props.minor, props.name,wDev);
#endif

    // Copy Matrix to GPU (unlike CPU case, cannot copy just pointer because always assume input is CPU and output is GPU)
    double t0 = timer<double>();
    PUSH_RANGE("MDsendsource",MDsendsource,1);

    if(data){
      CUDACHECK(hipMalloc(_data, m * n * sizeof(T))); // allocate on GPU
      CUDACHECK(hipMemcpy(*_data, data, m * n * sizeof(T),hipMemcpyHostToDevice)); // copy from orig CPU data to GPU
      //      fprintf(stderr,"_data: %p\n",(void*)*_data); fflush(stderr);
    }
    else *_data=NULL;

    if(datay){
      CUDACHECK(hipMalloc(_datay, m * sizeof(T))); // allocate on GPU
      CUDACHECK(hipMemcpy(*_datay, datay, m * sizeof(T),hipMemcpyHostToDevice)); // copy from orig CPU data to GPU
      //      fprintf(stderr,"_datay: %p\n",(void*)*_datay); fflush(stderr);
    }
    else *_datay=NULL;

    if(vdata){
      CUDACHECK(hipMalloc(_vdata, mValid * n * sizeof(T))); // allocate on GPU
      CUDACHECK(hipMemcpy(*_vdata, vdata, mValid * n * sizeof(T),hipMemcpyHostToDevice)); // copy from orig CPU data to GPU
      //      fprintf(stderr,"_vdata: %p\n",(void*)*_vdata); fflush(stderr);
    }
    else *_vdata=NULL;

    if(vdatay){
      CUDACHECK(hipMalloc(_vdatay, mValid * sizeof(T))); // allocate on GPU
      CUDACHECK(hipMemcpy(*_vdatay, vdatay, mValid * sizeof(T),hipMemcpyHostToDevice)); // copy from orig CPU data to GPU
      //      fprintf(stderr,"_vdatay: %p\n",(void*)*_vdatay); fflush(stderr);
    }
    else *_vdatay=NULL;

    //    fprintf(stderr,"weight=%p\n",weight); fflush(stderr);
    if(weight){
      CUDACHECK(hipMalloc(_weight, m * sizeof(T))); // allocate on GPU
      CUDACHECK(hipMemcpy(*_weight, weight, m * sizeof(T),hipMemcpyHostToDevice)); // copy from orig CPU data to GPU
    }
    else{
      DEBUG_FPRINTF(stderr,"making up unity weights: %d\n",m);
      CUDACHECK(hipMalloc(_weight, m * sizeof(T))); // allocate on GPU
      thrust::device_ptr<T> dev_ptr=thrust::device_pointer_cast(static_cast<T*>(*_weight));
      T fill_value=1.0;
      thrust::fill(dev_ptr, dev_ptr + m, fill_value);
      //      fprintf(stderr,"_weight: %p\n",(void*)*_weight); fflush(stderr);
    }
    
    POP_RANGE("MDsendsource",MDsendsource,1);
    double t2 = timer<double>();
    DEBUG_FPRINTF(stdout,"Time to allocate and copy the data matrix on the GPU: %f\n", t2-t0);
    hipDeviceSynchronize();
    
    DEBUG_FPRINTF(stderr,"pointer data   %p\n",(void*)*_data);
    DEBUG_FPRINTF(stderr,"pointer datay  %p\n",(void*)*_datay);
    DEBUG_FPRINTF(stderr,"pointer vdata  %p\n",(void*)*_vdata);
    DEBUG_FPRINTF(stderr,"pointer vdaty  %p\n",(void*)*_vdatay);
    DEBUG_FPRINTF(stderr,"pointer weight %p\n",(void*)*_weight);


    return(0);
}

  

  template int makePtr_dense<double>(int sharedA, int me, int wDev, size_t m, size_t n, size_t mValid, const char ord,
                                     const double *data, const double *datay, const double *vdata, const double *vdatay, const double *weight,
                                     double **_data, double **_datay, double **_vdata, double **_vdatay, double **_weight);
  template int makePtr_dense<float>(int sharedA, int me, int wDev, size_t m, size_t n, size_t mValid, const char ord,
                                    const float *data, const float *datay, const float *vdata, const float *vdatay, const float *weight,
                                    float **_data, float **_datay, float **_vdata, float **_vdatay, float **_weight);



template <typename T>
int modelFree1(T *aptr){

  if(aptr!=NULL){
    // for now, freed during ~
    //hipFree(aptr);
    //CUDA_CHECK_ERR();
  }
  return(0);
}


  template int modelFree1<float>(float *aptr);
  template int modelFree1<double>(double *aptr);

}  // namespace h2o4gpu

int modelfree1_double(double *aptr){
  return h2o4gpu::modelFree1<double>(aptr);
}
int modelfree1_float(float *aptr){
  return h2o4gpu::modelFree1<float>(aptr);
}


    int make_ptr_double(int sharedA, int sourceme, int sourceDev, size_t mTrain, size_t n, size_t mValid, const char ord,
                        const double* trainX, const double* trainY, const double* validX, const double* validY, const double *weight,
                        double**a, double**b, double**c, double**d, double **e) {
      return h2o4gpu::makePtr_dense<double>(sharedA, sourceme, sourceDev, mTrain, n, mValid, ord, trainX, trainY, validX, validY, weight, a, b, c, d, e);
    }
    int make_ptr_float(int sharedA, int sourceme, int sourceDev, size_t mTrain, size_t n, size_t mValid, const char ord,
                       const float* trainX, const float* trainY, const float* validX, const float* validY, const float *weight,
                       float**a, float**b, float**c, float**d, float **e) {
      return h2o4gpu::makePtr_dense<float>(sharedA, sourceme, sourceDev, mTrain, n, mValid, ord, trainX, trainY, validX, validY, weight, a, b, c, d, e);
    }




