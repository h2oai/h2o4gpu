#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 H2O.ai, Inc.
 * License   Apache License Version 2.0 (see LICENSE for details)
 */

#include <thrust/device_vector.h>

#include <cub/device/device_histogram.cuh>

#include <random>
#include <limits>
#include <string>

#include <stdio.h>

#include "kmeans_init.cuh"

#include "../matrix/KmMatrix/KmMatrix.hpp"
#include "../matrix/KmMatrix/Arith.hpp"
#include "../matrix/KmMatrix/blas.cuh"
#include "../utils/utils.cuh"
#include "../utils/GpuInfo.cuh"

namespace h2o4gpu {
namespace kMeans {

using namespace Matrix;

namespace kernel {
// X^2 + Y^2, here only calculates the + operation.
template <typename T>
__global__ void construct_distance_pairs_kernel(
    kParam<T> _distance_pairs,
    kParam<T> _data_dots, kParam<T> _centroids_dots) {

  size_t idx = global_thread_idx();  // indexing data
  size_t idy = global_thread_idy();  // indexing centroids

  // FIXME: Is using shared memory necessary?

  size_t stride_x = grid_stride_x ();
  // strides only for data.
  for (size_t i = idx; i < _data_dots.rows; i += stride_x) {
    if (idy < _centroids_dots.rows ) {
      // i + idy: x^2 + y^2 between i^th data (a.k.a x) and idy^th
      // centroid (a.k.a y)
      _distance_pairs.ptr[i*_centroids_dots.rows + idy] =
          _data_dots.ptr[idx] + _centroids_dots.ptr[idy];
    }
  }
}

// See SelfArgMinOp
template <typename T>
__global__ void self_row_argmin_sequential(kParam<int> _res, kParam<T> _val) {

  size_t idx = global_thread_idx();
  if (idx < _val.rows) {
    T min = std::numeric_limits<T>::max();
    int min_idx = -1;
    for (size_t i = 0; i < _val.cols; ++i) {
      T value = _val.ptr[idx * _val.cols + i];
      if (value < min && value != 0) {
        min = value;
        min_idx = i;
      }
    }
    _res.ptr[idx] = min_idx;
  }
}

}  // namespace kernel

namespace detail {

template <typename T>
void PairWiseDistanceOp<T>::initialize(KmMatrix<T>& _data_dot,
                                       KmMatrix<T>& _centroids_dot,
                                       KmMatrix<T>& _distance_pairs) {
  data_dot_ = _data_dot;
  centroids_dot_ = _centroids_dot;
  distance_pairs_ = _distance_pairs;
  initialized_ = true;
}

template <typename T>
PairWiseDistanceOp<T>::PairWiseDistanceOp(KmMatrix<T>& _data_dot,
                                          KmMatrix<T>& _centroids_dot,
                                          KmMatrix<T>& _distance_pairs) :
    data_dot_(_data_dot), centroids_dot_(_centroids_dot),
    distance_pairs_(_distance_pairs), initialized_(true) {}

template <typename T>
KmMatrix<T> PairWiseDistanceOp<T>::operator()(KmMatrix<T>& _data,
                                              KmMatrix<T>& _centroids) {

  kernel::construct_distance_pairs_kernel<<<
      dim3(GpuInfo::ins().blocks(32), div_roundup(_centroids.rows(), 16)),
      dim3(32, 16)>>>(  // FIXME: Tune this.
          distance_pairs_.k_param(),
          data_dot_.k_param(),
          centroids_dot_.k_param());

  safe_cuda(hipGetLastError());

  hipblasHandle_t handle = GpuInfo::ins().cublas_handle();

  T alpha = -2.0;
  T beta = 1.0;

  Blas::gemm(
      handle,
      HIPBLAS_OP_T, HIPBLAS_OP_N,
      // n, d, d/k
      _centroids.rows(), _data.rows(), _data.cols(),
      &alpha,
      _centroids.dev_ptr(), _centroids.cols(),
      _data.dev_ptr(), _data.cols(),
      &beta,
      distance_pairs_.dev_ptr(), _centroids.rows());

  return distance_pairs_;
}

// ArgMin operation that excludes 0. Used when dealing with distance_pairs
// in recluster where distance between points with itself is calculated,
// hence the name.
// FIXME: Maybe generalize it to selection algorithm.
template <typename T>
struct SelfArgMinOp {
  KmMatrix<int> argmin(KmMatrix<T>& _val) {
    KmMatrix<int> _res(_val.rows(), 1);
    kernel::self_row_argmin_sequential<<<
        div_roundup(_val.rows(), 256), 256>>>(_res.k_param(),
                                              _val.k_param());
    return _res;
  }
};

// We use counting to construct the weight as described in the paper. Counting
// is performed by histogram algorithm.
// For re-cluster, the paper suggests using K-Means++, but that will require
// copying data back to host. So we simply use those selected centroids with
// highest probability.
template <typename T>
KmMatrix<T> GreedyRecluster<T>::recluster(KmMatrix<T>& _centroids, size_t _k) {

  // Get the distance pairs for centroids
  KmMatrix<T> centroids_dot (_centroids.rows(), 1);
  VecBatchDotOp<T>().dot(centroids_dot, _centroids);
  KmMatrix<T> distance_pairs (_centroids.rows(), _centroids.rows());
  PairWiseDistanceOp<T> centroids_distance_op(
      centroids_dot, centroids_dot, distance_pairs);

  distance_pairs = centroids_distance_op(_centroids, _centroids);

  // get the closest x_j for each x_i in centroids.
  KmMatrix<int> min_indices = SelfArgMinOp<T>().argmin(distance_pairs);

  // use historgram to get counting for weights
  KmMatrix<int> weights (1, _centroids.rows());

  size_t temp_storage_bytes = 0;
  void *d_temp_storage = NULL;

  // determine the temp_storage_bytes
  safe_cuda(hipcub::DeviceHistogram::HistogramEven(
      d_temp_storage, temp_storage_bytes,
      min_indices.dev_ptr(),
      weights.dev_ptr(),
      min_indices.rows() + 1,
      (T)0.0,
      (T)min_indices.rows(),
      (int)_centroids.rows()));

  // cub has level bound, which deals with -1 returned by SelfArgMinOp
  safe_cuda(hipMalloc((void**)&d_temp_storage, temp_storage_bytes));
  safe_cuda(hipcub::DeviceHistogram::HistogramEven(
      d_temp_storage, temp_storage_bytes,
      min_indices.dev_ptr(),    // d_samples
      weights.dev_ptr(),        // d_histogram
      min_indices.rows() + 1,   // num_levels
      (T)0.0,                   // lower_level
      (T)min_indices.rows(),    // upper_level
      (int)_centroids.rows())); // num_samples
  safe_cuda(hipFree(d_temp_storage));

  // Sort the indices by weights in ascending order, then use those at front
  // as result.
  thrust::sort_by_key(thrust::device,
                      weights.dev_ptr(),
                      weights.dev_ptr() + weights.size(),
                      min_indices.dev_ptr(),
                      thrust::greater<int>());

  int * min_indices_ptr = min_indices.dev_ptr();

  KmMatrix<T> new_centroids (_k, _centroids.cols());
  T * new_centroids_ptr = new_centroids.dev_ptr();
  int cols = _centroids.cols();

  T * old_centroids_ptr = _centroids.dev_ptr();

  auto k_iter = thrust::make_counting_iterator(0);
  thrust::for_each(
      thrust::device,
      k_iter, k_iter + _k,
      [=] __device__ (int idx) {
        size_t index = min_indices_ptr[idx];

        size_t in_begin = index * cols;
        size_t in_end = (index + 1) * cols;

        size_t res_begin = idx * cols;

        for (size_t i = in_begin, j = res_begin; i < in_end; ++i, ++j) {
          new_centroids_ptr[j] = old_centroids_ptr[i];
        }
      });

  return new_centroids;
}

}  // namespace detail


/* ============ KmeansRandomInit Class member functions ============ */

template <typename T>
KmMatrix<T> KmeansRandomInit<T>::operator()(KmMatrix<T>& _data, size_t _k) {

  KmMatrix<T> dist = generator_impl_->generate(_k);
  MulOp<T>().mul(dist, dist, (T)_data.rows() - 1);

  dist = dist.reshape(dist.cols(), dist.rows());

  KmMatrix<T> centroids = _data.rows(dist);

  return centroids;
}


/* ============== KmeansLlInit Class member functions ============== */

// Although the paper suggested calculating the probability independently,
// but due to zero distance between a point and itself, the already selected
// points will have very low probability.
template <typename T, template <class> class ReclusterPolicy >
KmMatrix<T> KmeansLlInit<T, ReclusterPolicy>::probability(
    KmMatrix<T>& _data, KmMatrix<T>& _centroids) {

  KmMatrix<T> centroids_dot (_centroids.rows(), 1);
  VecBatchDotOp<T>().dot(centroids_dot, _centroids);

  // FIXME: Time this
  distance_pairs_ = KmMatrix<T>(_data.rows(), _centroids.rows());
  detail::PairWiseDistanceOp<T> distance_op (
      data_dot_, centroids_dot, distance_pairs_);
  distance_pairs_ = distance_op(_data, _centroids);

  KmMatrix<T> min_distances = MinOp<T>().min(distance_pairs_,
                                             KmMatrixDim::ROW);

  T cost = SumOp<T>().sum(min_distances);
  cost += ESP;

  KmMatrix<T> prob (min_distances.rows(), 1);
  MulOp<T>().mul(prob, min_distances, over_sample_ * k_ / cost);

  return prob;
}


template <typename T, template <class> class ReclusterPolicy >
KmMatrix<T> KmeansLlInit<T, ReclusterPolicy>::sample_centroids(
    KmMatrix<T>& _data, KmMatrix<T>& _prob) {

  KmMatrix<T> thresholds = generator_->generate(_data.rows());

  T * thresholds_ptr = thresholds.dev_ptr();

  // If use kParam, nvcc complains:
  // identifier "H2O4GPU::KMeans::kParam<double> ::kParam" is undefined in
  // device code.
  T* prob_ptr = _prob.dev_ptr();

  auto prob_iter = thrust::make_counting_iterator(0);
  size_t n_new_centroids = thrust::count_if(
      thrust::device, prob_iter,
      prob_iter + _prob.size(),
      [=] __device__ (int idx) {
        float thresh = thresholds_ptr[idx];
        T prob_x = prob_ptr[idx];
        return prob_x > thresh;
      });

  KmMatrix<T> new_centroids(n_new_centroids, _data.cols());
  thrust::device_ptr<T> new_centroids_ptr (new_centroids.dev_ptr());

  thrust::device_ptr<T> data_ptr (_data.dev_ptr());

  size_t cols = _data.cols();
  // renew iterator
  prob_iter = thrust::make_counting_iterator(0);
  thrust::copy_if(
      thrust::device,
      data_ptr, data_ptr + _data.size(), prob_iter,
      new_centroids_ptr,
      [=] __device__(int idx) {
        size_t row = idx / cols;
        T thresh = thresholds_ptr[row];
        T prob_x = prob_ptr[row];
        return prob_x > thresh;
      });

  return new_centroids;
}

template <typename T, template <class> class ReclusterPolicy>
KmMatrix<T>
KmeansLlInit<T, ReclusterPolicy>::operator()(KmMatrix<T>& _data, size_t _k) {

  if (_k > _data.size()) {
    char err_msg[128];
    sprintf(
        err_msg,
        "k must be less than or equal to the number of data points"
        ", k: %lu, data points: %lu",
        _k, _data.rows());
    h2o4gpu_error(err_msg);
  }

  if (seed_ < 0) {
    std::random_device rd;
    seed_ = rd();
  }
  k_ = _k;

  std::mt19937 generator(0);

  std::uniform_int_distribution<> distribution(0, _data.rows());
  size_t idx = distribution(generator);

  // Calculate X^2 (point-wise)
  data_dot_ = KmMatrix<T>(_data.rows(), 1);
  VecBatchDotOp<T>().dot(data_dot_, _data);

  // First centroid
  KmMatrix<T> centroids = _data.row(idx);

  KmMatrix<T> prob = probability(_data, centroids);

  T cost = SumOp<T>().sum(prob);

  size_t max_iter = std::max((size_t)(MAX_ITER),
                             (size_t)std::ceil(std::log(cost)));
  for (size_t i = 0; i < max_iter; ++i) {
    prob = probability(_data, centroids);
    KmMatrix<T> new_centroids = sample_centroids(_data, prob);
    centroids = stack(centroids, new_centroids, KmMatrixDim::ROW);
  }

  if (centroids.rows() < _k) {
    KmMatrix<T> new_centroids = KmeansRandomInit<T>(generator_)(_data,
        _k - centroids.rows());
    centroids = stack(centroids, new_centroids, KmMatrixDim::ROW);
  }

  centroids = ReclusterPolicy<T>::recluster(centroids, k_);

  return centroids;
}

#define INSTANTIATE(T)                                                  \
  template KmMatrix<T> KmeansLlInit<T>::operator()(                     \
      KmMatrix<T>& _data, size_t _k);                                   \
  template KmMatrix<T> KmeansLlInit<T>::probability(                    \
      KmMatrix<T>& data, KmMatrix<T>& centroids);                       \
  template KmMatrix<T> KmeansLlInit<T>::sample_centroids(               \
      KmMatrix<T>& data, KmMatrix<T>& centroids);                       \
  template KmMatrix<T> KmeansRandomInit<T>::operator()(                 \
      KmMatrix<T>& _data, size_t _k);

INSTANTIATE(float)
INSTANTIATE(double)
INSTANTIATE(int)

#undef INSTANTIATE

namespace detail {

#define INSTANTIATE(T)                                          \
  template PairWiseDistanceOp<T>::PairWiseDistanceOp (          \
      KmMatrix<T>& _data_dot,                                   \
      KmMatrix<T>& _centroids_dot,                              \
      KmMatrix<T>& _distance_pairs);                            \
  template void PairWiseDistanceOp<T>::initialize(              \
      KmMatrix<T>& _data_dot,                                   \
      KmMatrix<T>& _centroids_dot,                              \
      KmMatrix<T>& _distance_pairs);                            \
  template KmMatrix<T> PairWiseDistanceOp<T>::operator()(       \
      KmMatrix<T>& _data, KmMatrix<T>& _centroids);             \

INSTANTIATE(float)
INSTANTIATE(double)
INSTANTIATE(int)

#undef INSTANTIATE
}

}  // namespace kMeans
}  // namespace h2o4gpu
