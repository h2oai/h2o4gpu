#include "hip/hip_runtime.h"
#include <cstdio>
#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "../device/device_context.cuh"
#include "../../include/solver/tsvd.h"
#include <ctime>
#include <thrust/iterator/counting_iterator.h>
#include<algorithm>
#include <thrust/sequence.h>
#include <thrust/inner_product.h>
#include <thrust/transform_reduce.h>
#include "../data/matrix.cuh"

namespace tsvd
{

	/**
	 * Division utility to get explained variance ratio
	 *
	 * @param XVar
	 * @param XVarSum
	 * @param ExplainedVarRatio
	 * @param context
	 */
	template<typename T>
	void divide(const Matrix<T> &XVar, const Matrix<T> &XVarSum, Matrix<T> &ExplainedVarRatio, DeviceContext &context){
		auto d_x_var = XVar.data();
		auto d_x_var_sum = XVarSum.data();
		auto d_expl_var_ratio = ExplainedVarRatio.data();
		auto counting = thrust::make_counting_iterator <int>(0);
		thrust::for_each(counting, counting+ExplainedVarRatio.size(), [=]__device__(int idx){
			float div_val = 0.0;
			//XVarSum can possibly be zero
			if(d_x_var_sum[0] != 0.0){
				div_val = d_x_var[idx] / d_x_var_sum[0];
			}
			d_expl_var_ratio[idx] = div_val;
		} );
	}

	/**
	 * Square each value in a matrix
	 *
	 * @param UmultSigma
	 * @param UmultSigmaSquare
	 * @param context
	 */
	template<typename T>
	void square_val(const Matrix<T> &UmultSigma, Matrix<T> &UmultSigmaSquare, DeviceContext &context){
		auto n = UmultSigma.columns();
		auto m = UmultSigma.rows();
		auto k = UmultSigmaSquare.rows();
		auto d_u_mult_sigma = UmultSigma.data();
		auto d_u_mult_sigma_square = UmultSigmaSquare.data();
		auto counting = thrust::make_counting_iterator <int>(0);
		thrust::for_each(counting, counting+UmultSigmaSquare.size(), [=]__device__(int idx){
			float square_val = std::pow(d_u_mult_sigma[idx],2);
			d_u_mult_sigma_square[idx] = square_val;
		} );
	}

	/**
	 * Utility to calculate variance for each column of a matrix
	 *
	 * @param X
	 * @param UColMean
	 * @param UVar
	 * @param context
	 */
	template<typename T>
	void calc_var_numerator(Matrix<T> &X, const Matrix<T> &UColMean, Matrix<T> &UVar, DeviceContext &context){
		auto counting = thrust::make_counting_iterator(0);
		auto d_X = X.data();
		auto d_Mean = UColMean.data();
		auto rows = X.rows();
		thrust::for_each(counting, counting+X.size(),  [=] __device__ (size_t idx)
		{
			auto column = idx/rows;
			d_X[idx] = std::pow(d_X[idx] - d_Mean[column],2);
		});

		Matrix<T>Ones(X.rows(), 1);
		Ones.fill(1.0f);
		multiply(X, Ones, UVar, context, true, false, 1.0f);

	}

	/**
	 * Utility to reverse q to show most import k to least important k
	 *
	 * @param Q
	 * @param QReversed
	 * @param context
	 */
	template<typename T>
	void col_reverse_q(const Matrix<T> &Q, Matrix<T> &QReversed, DeviceContext &context){
		auto n = Q.columns();
		auto m = Q.rows();
		auto k = QReversed.rows();
		auto d_q = Q.data();
		auto d_q_reversed = QReversed.data();
		auto counting = thrust::make_counting_iterator <int>(0);
		thrust::for_each(counting, counting+QReversed.size(), [=]__device__(int idx){
			int dest_row = idx % m;
			int dest_col = idx/m;
			int src_row = dest_row;
			int src_col = (n - dest_col) - 1;
			d_q_reversed[idx] = d_q[src_col * m + src_row];
		} );
	}

	/**
	 * Truncate Q transpose to top k
	 *
	 * @param Qt
	 * @param QtTrunc
	 * @param context
	 */
	template<typename T>
	void row_reverse_trunc_q(const Matrix<T> &Qt, Matrix<T> &QtTrunc, DeviceContext &context){
		auto m = Qt.rows();
		auto k = QtTrunc.rows();
		auto d_q = Qt.data();
		auto d_q_trunc = QtTrunc.data();
		auto counting = thrust::make_counting_iterator <int>(0);
		thrust::for_each(counting, counting+QtTrunc.size(), [=]__device__(int idx){
			int dest_row = idx % k;
			int dest_col = idx / k;
			int src_row = (m - dest_row) - 1;
			int src_col = dest_col;
			float q = d_q[src_col * m + src_row];
			d_q_trunc[idx] = q;
		} );
	}

	/**
	 * Transform matrix into absolute values
	 *
	 * @param UmultSigma
	 * @param UmultSigmaSquare
	 * @param context
	 */
	template<typename T>
	void get_abs(const Matrix<T> &U, Matrix<T> &U_abs, DeviceContext &context){
		thrust::transform(U.dptr(), U.dptr() + U.size(), U_abs.dptr(), [=]__device__(T val){
            return abs(val);
        });
	}

	/**
	 * Calculate the U matrix, which is defined as:
	 * U = A*V/sigma where A is our X Matrix, V is Q, and sigma is 1/w_i
	 *
	 * @param X
	 * @param Q
	 * @param w
	 * @param U
	 * @param context
	 */
	template<typename T>
	void calculate_u(const Matrix<T> &X, const Matrix<T> &Q, const Matrix<T> &w, Matrix<T> &U, DeviceContext &context){
		multiply(X, Q, U, context, false, false, 1.0f); //A*V
		auto d_u = U.data();
		auto d_sigma = w.data();
		auto column_size = U.rows();
		auto counting = thrust::make_counting_iterator <int>(0);
		thrust::for_each(counting, counting+U.size(), [=]__device__(int idx){
			int column = idx/column_size;
			float sigma = d_sigma[column];
			float u = d_u[idx];
			if(sigma != 0.0){
				d_u[idx] = u * 1.0/sigma;
			} else{
				d_u[idx] = 0.0;
			}
		} );
	}

	/**
	 * Obtain SVD attributes, which are as follows:
	 * 1.Singular Values
	 * 2.U matrix
	 * 3.Explained Variance
	 * 4.Explained Variance Ratio
	 */
	template<typename T, typename S>
	void get_tsvd_attr(Matrix<T> &X, Matrix<T> &Q, S _Q, Matrix<T> &w, S _w, S _U, S _X_transformed, S _explained_variance, S _explained_variance_ratio, params _param, DeviceContext &context){

		//Obtain Q^T to obtain vector as row major order
		Matrix<T>Qt(Q.columns(), Q.rows());
		transpose(Q, Qt, context); //Needed for calculate_u()
		Matrix<T>QtTrunc(_param.k, Qt.columns());
		row_reverse_trunc_q(Qt, QtTrunc, context);

		if (_param.whiten) {
			auto d_q = QtTrunc.data();
			auto x_sqrt_row = std::sqrt(X.rows());
			auto d_sigma = w.data();
			auto column_size = QtTrunc.rows();
			auto counting = thrust::make_counting_iterator <int>(0);
			thrust::for_each(counting, counting+QtTrunc.size(), [=]__device__(int idx){
				int column = idx/column_size;
				T sigma = d_sigma[column];
				T q = d_q[idx];
				d_q[idx] = (q * x_sqrt_row)/std::sqrt(sigma);
			} );
		}

		//Obtain square root of eigenvalues, which are singular values
		T generic_zero = 0.0;
		w.transform([=]__device__(T elem){
			if(elem > generic_zero){
				return std::sqrt(elem);
			}else{
				return generic_zero;
			}
		}
		);

		//Sort from biggest singular value to smallest
		std::vector<double> w_temp(w.size());
		w.copy_to_host(w_temp.data()); //Send to host
		std::reverse(w_temp.begin(), w_temp.end());
		std::copy(w_temp.begin(), w_temp.begin() + _param.k, _w);
		Matrix<T>sigma(_param.k, 1);
		sigma.copy(w_temp.data());

		//Get U matrix
		Matrix<T>U(X.rows(), _param.k);
		Matrix<T>QReversed(Q.rows(), Q.columns());
		col_reverse_q(Q, QReversed, context);
		calculate_u(X, QReversed, sigma, U, context);

		/**
		 * SVD sign correction (same as svd_flip() in sklearn) -> https://github.com/scikit-learn/scikit-learn/blob/master/sklearn/utils/extmath.py#L499
		   Sign correction to ensure deterministic output from SVD.
    	   Adjusts the columns of u and the rows of v such that the loadings in the
    	   columns in u that are largest in absolute value are always positive.
		 */
		std::vector<int> result_array(U.columns());
		max_index_per_column(U, result_array, context);
		Matrix<T>Signs(1, _param.k);
		thrust::device_vector<int> d_results = result_array;
		auto d_U = U.data();
		auto d_Signs = Signs.data();
		auto counting = thrust::make_counting_iterator <int>(0);
		auto ptr = d_results.data();
		thrust::for_each(counting, counting+Signs.size(), [=]__device__(int idx){
			int u_idx = ptr[idx];
			T val = 1.0;
			if (d_U[u_idx] < 0.0) {
				val = -1.0;
			} else if (d_U[u_idx] == 0.0) {
				val = 0.0;
			} else {
				val = 1.0;
			}
			d_Signs[idx] = val;
		} );
		multiply_diag(U, Signs, U, context, false);
		multiply_diag(QtTrunc, Signs, QtTrunc, context, true);
		U.copy_to_host(_U); //Send to host
		QtTrunc.copy_to_host(_Q); //Send to host

		//U * Sigma (_X_transformed)
		Matrix<T>UmultSigma(U.rows(), U.columns());
		multiply_diag(U, sigma, UmultSigma, context, false);
		UmultSigma.copy_to_host(_X_transformed); //Send to host

		//Explained Variance
		Matrix<T>UOnesSigma(UmultSigma.rows(), 1);
		UOnesSigma.fill(1.0f);
		Matrix<T>USigmaVar(_param.k, 1);
		Matrix<T>USigmaColMean(_param.k, 1);
		multiply(UmultSigma, UOnesSigma, USigmaColMean, context, true, false, 1.0f);
		float m_usigma = UmultSigma.rows();
		multiply(USigmaColMean, 1/m_usigma, context);
		calc_var_numerator(UmultSigma, USigmaColMean, USigmaVar, context);
		multiply(USigmaVar, 1/m_usigma, context);
		USigmaVar.copy_to_host(_explained_variance); //Send to host

		//Explained Variance Ratio
		//Set aside matrix of 1's for getting sum of columnar variances
		Matrix<T>XmultOnes(X.rows(), 1);
		XmultOnes.fill(1.0f);
		Matrix<T>XVar(X.columns(), 1);
		Matrix<T>XColMean(X.columns(), 1);
		multiply(X, XmultOnes, XColMean, context, true, false, 1.0f);
		float m = X.rows();
		multiply(XColMean, 1/m, context);
		calc_var_numerator(X, XColMean, XVar, context);
		multiply(XVar, 1/m, context);

		Matrix<T>XVarSum(1,1);
		multiply(XVar, XmultOnes, XVarSum, context, true, false, 1.0f);
		Matrix<T>ExplainedVarRatio(_param.k, 1);
		divide(USigmaVar, XVarSum, ExplainedVarRatio, context);
		ExplainedVarRatio.copy_to_host(_explained_variance_ratio); //Send to host
	}


	void outer_product(Matrix<float>& A, float eigen_value, const Matrix<float>& eigen_vector, const Matrix<float>& eigen_vector_transpose, DeviceContext& context)
	{
		safe_cublas(hipblasSger(context.cublas_handle, A.rows(), A.columns(), &eigen_value, eigen_vector.data(), 1, eigen_vector_transpose.data(), 1, A.data(), A.rows()));
	}

	void outer_product(Matrix<double>& A, double eigen_value, const Matrix<double>& eigen_vector, const Matrix<double>& eigen_vector_transpose, DeviceContext& context)
	{
		safe_cublas(hipblasDger(context.cublas_handle, A.rows(), A.columns(), &eigen_value, eigen_vector.data(), 1, eigen_vector_transpose.data(), 1, A.data(), A.rows()));
	}

	/**
	 * Conduct truncated svd using hipsolverDnSsyevd
	 *
	 * @param X
	 * @param _Q
	 * @param _w
	 * @param _U
	 * @param _explained_variance
	 * @param _explained_variance_ratio
	 * @param _param
	 */
	template<typename T, typename S>
	void cusolver_tsvd(Matrix<T> &X, S _Q, S _w, S _U, S _X_transformed, S _explained_variance, S _explained_variance_ratio, params _param){
		//Allocate matrix for X^TX
		Matrix<T>XtX(_param.X_n, _param.X_n);

		//Create context
		DeviceContext context;

		//Multiply X and Xt and output result to XtX
		multiply(X, X, XtX, context, true, false, 1.0f);

		//Set up Q (V^T) and w (singular value) matrices (w is a matrix of size Q.rows() by 1; really just a vector
		Matrix<T>Q(XtX.rows(), XtX.columns()); // n X n -> V^T
		Matrix<T>w(Q.rows(), 1);
		calculate_eigen_pairs_exact(XtX, Q, w, context);

		//Get tsvd attributes
		get_tsvd_attr(X, Q, _Q, w, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param, context);
	}

	/**
	 * Conduct truncated svd using the power method
	 *
	 * @param X
	 * @param _Q
	 * @param _w
	 * @param _U
	 * @param _explained_variance
	 * @param _explained_variance_ratio
	 * @param _param
	 */
	template<typename T, typename S>
	void power_tsvd(Matrix<T> &X, S _Q, S _w, S _U, S _X_transformed, S _explained_variance, S _explained_variance_ratio, params _param){
		//Allocate matrix for X^TX
		Matrix<T>M(_param.X_n, _param.X_n);

		//Allocate matrix to be used in outer product calculation
		Matrix<T>A(M.rows(), M.columns());

		//Create context
		DeviceContext context;

		//Multiply X and Xt and output result to XtX
		multiply(X, X, M, context, true, false, 1.0f);

		//Set up Q (V^T) and w (singular value) matrices (w is a matrix of size Q.rows() by 1; really just a vector
		Matrix<T>Q(M.rows(), _param.k);
		Matrix<T>w(_param.k, 1);
		std::vector<T> w_temp(w.size());

		/* Power Method for finding all eigenvectors/eigen values
		 * Logic:
		 *
		 * Since the matrix is symmetric, there exists an orthogonal basis of eigenvectors. Once you have found an eigenvector, extend it to an orthogonal basis,
		 * rewrite the matrix in terms of this basis and restrict to the orthogonal space of the known eigenvector.
		 * This is comprised in the method of deflation:
		 *     You first find the first eigenvector v1 (with a maximal λ1), by iterating xn+1=Axn/|Axn|with a "random" initial x0. Once you have found a good approximation
		 *     for v1, you consider B=A−λ1|v1|2 * v1vT (this simple step replaces the "rewrite in terms of this basis" above). This is a matrix that behaves like A for anything
		 *     orthogonal to v1 and zeroes out v1. Use the power method on B again, which will reveal v2, an eigenvector of a largest eigenvalue of B.
		 *     Then switch to C=B−λ2|v2|2 * v2vT so on.
		 */
		Matrix<T>b_k(_param.X_n, 1);
		Matrix<T>b_k1(_param.X_n, 1);

		for(int i = 0; i < _param.k; i ++){
			//Set aside vector of randoms (n x 1)
			b_k.random(_param.random_state + i);
			T previous_eigenvalue_estimate = FLT_MAX;
			T eigen_value_estimate = FLT_MAX;
			for(int iter=0; iter<_param.n_iter;iter++){
				//fprintf(stderr,"k=%d/%d iter=%d/%d\n",i,_param.k-1,iter,_param.n_iter); fflush(stderr);
				multiply(M, b_k, b_k1, context);
				dot_product(b_k1, b_k, &eigen_value_estimate, context);
				if(std::abs(eigen_value_estimate - previous_eigenvalue_estimate) <= (_param.tol * std::abs(previous_eigenvalue_estimate))) {
					break;
				}
				normalize_vector_cublas(b_k1, context);
				b_k.copy(b_k1);
				previous_eigenvalue_estimate = eigen_value_estimate;
			}
			//Obtain eigen value
			w_temp[i] = eigen_value_estimate;

			//Put eigen vector into (starting at last column of Q)
			thrust::copy(b_k.dptr(), b_k.dptr()+b_k.size(), Q.dptr()+Q.rows()*(Q.columns()-i-1));

			//Get rid of eigen effect from original matrix (deflation)
			//multiply(A, 0.0, context);
			A.zero();
			outer_product(A, eigen_value_estimate, b_k, b_k, context);
			subtract(M, A, M, context);
		}
		//Fill in w from vector w_temp
		std::reverse(w_temp.begin(), w_temp.end());
		w.copy(w_temp.data());

		//Get tsvd attributes
		get_tsvd_attr(X, Q, _Q, w, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param, context);

	}

	/**
	 * Conduct truncated SVD on a matrix with float type
	 *
	 * @param _X
	 * @param _Q
	 * @param _w
	 * @param _U
	 * @param _explained_variance
	 * @param _explained_variance_ratio
	 * @param _param
	 */
	void truncated_svd_float(const float *_X, float *_Q, float *_w, float *_U, float* _X_transformed, float *_explained_variance, float *_explained_variance_ratio, params _param)
	{
		safe_cuda(hipSetDevice(_param.gpu_id));
		Matrix<float>X(_param.X_m, _param.X_n);
		X.copy(_X);
		truncated_svd_matrix(X, _Q, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param);
	}

	/**
	 * Conduct truncated SVD on a matrix with double type
	 *
	 * @param _X
	 * @param _Q
	 * @param _w
	 * @param _U
	 * @param _explained_variance
	 * @param _explained_variance_ratio
	 * @param _param
	 */
	void truncated_svd_double(const double *_X, double *_Q, double *_w, double *_U, double* _X_transformed, double *_explained_variance, double *_explained_variance_ratio, params _param)
	{
		safe_cuda(hipSetDevice(_param.gpu_id));
		Matrix<double>X(_param.X_m, _param.X_n);
		X.copy(_X);
		truncated_svd_matrix(X, _Q, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param);
	}

	template<typename T, typename S>
	tsvd_export void truncated_svd_matrix(Matrix<T> &X, S _Q, S _w, S _U, S _X_transformed, S _explained_variance, S _explained_variance_ratio, params _param)
	{
		std::string algorithm(_param.algorithm);
		try
		{
			if(algorithm == "cusolver"){
				if(_param.verbose==1){
				 fprintf(stderr,"Algorithm is cusolver with k = %d\n",_param.k); fflush(stderr);
				}
				tsvd::cusolver_tsvd(X, _Q, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param);
			}
			else {
				if(_param.verbose==1){
				 fprintf(stderr,"Algorithm is power with k = %d and number of iterations = %d\n",_param.k,_param.n_iter); fflush(stderr);
				}
				tsvd::power_tsvd(X, _Q, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param);
			}
		}
		catch (const std::exception &e)
		  {
			std::cerr << "tsvd error: " << e.what() << "\n";
		  }
		catch (std::string e)
		  {
			std::cerr << "tsvd error: " << e << "\n";
		  }
		catch (...)
		  {
			std::cerr << "tsvd error\n";
		  }
	}
}

//Impl for floats and doubles
template void tsvd::truncated_svd_matrix<float>(Matrix<float> &X, float* _Q, float* _w, float* _U, float* _X_transformed, float* _explained_variance, float* _explained_variance_ratio, params _param);
template void tsvd::truncated_svd_matrix<double>(Matrix<double> &X, double* _Q, double* _w, double* _U, double* _X_transformed, double* _explained_variance, double* _explained_variance_ratio, params _param);
template void tsvd::cusolver_tsvd<double>(Matrix<double> &X, double* _Q, double* _w, double* _U, double* _X_transformed, double* _explained_variance, double* _explained_variance_ratio, params _param);
template void tsvd::power_tsvd<double>(Matrix<double> &X, double* _Q, double* _w, double* _U, double* _X_transformed, double* _explained_variance, double* _explained_variance_ratio, params _param);
template void tsvd::cusolver_tsvd<float>(Matrix<float> &X, float* _Q, float* _w, float* _U, float* _X_transformed, float* _explained_variance, float* _explained_variance_ratio, params _param);
template void tsvd::power_tsvd<float>(Matrix<float> &X, float* _Q, float* _w, float* _U, float* _X_transformed, float* _explained_variance, float* _explained_variance_ratio, params _param);


