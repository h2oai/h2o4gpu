#include "hip/hip_runtime.h"
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../utils/utils.cuh"
#include "../device/device_context.cuh"
#include "../../include/solver/tsvd.h"
#include <ctime>
#include <thrust/iterator/counting_iterator.h>
#include<algorithm>
#include <thrust/sequence.h>
#include <thrust/inner_product.h>
#include <thrust/transform_reduce.h>
#include "../data/matrix.cuh"

namespace tsvd
{

	using namespace h2o4gpu;

	/**
	 * Division utility to get explained variance ratio
	 *
	 * @param XVar
	 * @param XVarSum
	 * @param ExplainedVarRatio
	 * @param context
	 */
	template<typename T>
	void divide(const matrix::Matrix<T> &XVar, const matrix::Matrix<T> &XVarSum, matrix::Matrix<T> &ExplainedVarRatio, device::DeviceContext &context){
		auto d_x_var = XVar.data();
		auto d_x_var_sum = XVarSum.data();
		auto d_expl_var_ratio = ExplainedVarRatio.data();
		auto counting = thrust::make_counting_iterator <int>(0);
		thrust::for_each(counting, counting+ExplainedVarRatio.size(), [=]__device__(int idx){
			float div_val = 0.0;
			//XVarSum can possibly be zero
			if(d_x_var_sum[0] != 0.0){
				div_val = d_x_var[idx] / d_x_var_sum[0];
			}
			d_expl_var_ratio[idx] = div_val;
		} );
	}

	/**
	 * Square each value in a matrix::Matrix
	 *
	 * @param UmultSigma
	 * @param UmultSigmaSquare
	 * @param context
	 */
	template<typename T>
	void square_val(const matrix::Matrix<T> &UmultSigma, matrix::Matrix<T> &UmultSigmaSquare, device::DeviceContext &context){
		auto n = UmultSigma.columns();
		auto m = UmultSigma.rows();
		auto k = UmultSigmaSquare.rows();
		auto d_u_mult_sigma = UmultSigma.data();
		auto d_u_mult_sigma_square = UmultSigmaSquare.data();
		auto counting = thrust::make_counting_iterator <int>(0);
		thrust::for_each(counting, counting+UmultSigmaSquare.size(), [=]__device__(int idx){
			float square_val = std::pow(d_u_mult_sigma[idx],2);
			d_u_mult_sigma_square[idx] = square_val;
		} );
	}

	/**
	 * Utility to calculate variance for each column of a matrix::Matrix
	 *
	 * @param X
	 * @param UColMean
	 * @param UVar
	 * @param context
	 */
	template<typename T>
	void calc_var_numerator(matrix::Matrix<T> &X, const matrix::Matrix<T> &UColMean, matrix::Matrix<T> &UVar, device::DeviceContext &context){
		auto counting = thrust::make_counting_iterator(0);
		auto d_X = X.data();
		auto d_Mean = UColMean.data();
		auto rows = X.rows();
		thrust::for_each(counting, counting+X.size(),  [=] __device__ (size_t idx)
		{
			auto column = idx/rows;
			d_X[idx] = std::pow(d_X[idx] - d_Mean[column],2);
		});

		matrix::Matrix<T>Ones(X.rows(), 1);
		Ones.fill(1.0f);
		multiply(X, Ones, UVar, context, true, false, 1.0f);

	}

	/**
	 * Utility to reverse q to show most import k to least important k
	 *
	 * @param Q
	 * @param QReversed
	 * @param context
	 */
	template<typename T>
	void col_reverse_q(const matrix::Matrix<T> &Q, matrix::Matrix<T> &QReversed, device::DeviceContext &context){
		auto n = Q.columns();
		auto m = Q.rows();
		auto k = QReversed.rows();
		auto d_q = Q.data();
		auto d_q_reversed = QReversed.data();
		auto counting = thrust::make_counting_iterator <int>(0);
		thrust::for_each(counting, counting+QReversed.size(), [=]__device__(int idx){
			int dest_row = idx % m;
			int dest_col = idx/m;
			int src_row = dest_row;
			int src_col = (n - dest_col) - 1;
			d_q_reversed[idx] = d_q[src_col * m + src_row];
		} );
	}

	/**
	 * Truncate Q transpose to top k
	 *
	 * @param Qt
	 * @param QtTrunc
	 * @param context
	 */
	template<typename T>
	void row_reverse_trunc_q(const matrix::Matrix<T> &Qt, matrix::Matrix<T> &QtTrunc, device::DeviceContext &context){
		auto m = Qt.rows();
		auto k = QtTrunc.rows();
		auto d_q = Qt.data();
		auto d_q_trunc = QtTrunc.data();
		auto counting = thrust::make_counting_iterator <int>(0);
		thrust::for_each(counting, counting+QtTrunc.size(), [=]__device__(int idx){
			int dest_row = idx % k;
			int dest_col = idx / k;
			int src_row = (m - dest_row) - 1;
			int src_col = dest_col;
			float q = d_q[src_col * m + src_row];
			d_q_trunc[idx] = q;
		} );
	}

	/**
	 * Transform matrix::Matrix into absolute values
	 *
	 * @param UmultSigma
	 * @param UmultSigmaSquare
	 * @param context
	 */
	template<typename T>
	void get_abs(const matrix::Matrix<T> &U, matrix::Matrix<T> &U_abs, device::DeviceContext &context){
		thrust::transform(U.dptr(), U.dptr() + U.size(), U_abs.dptr(), [=]__device__(T val){
            return abs(val);
        });
	}

	/**
	 * Calculate the U matrix::Matrix, which is defined as:
	 * U = A*V/sigma where A is our X matrix::Matrix, V is Q, and sigma is 1/w_i
	 *
	 * @param X
	 * @param Q
	 * @param w
	 * @param U
	 * @param context
	 */
	template<typename T>
	void calculate_u(const matrix::Matrix<T> &X, const matrix::Matrix<T> &Q, const matrix::Matrix<T> &w, matrix::Matrix<T> &U, device::DeviceContext &context){
		multiply(X, Q, U, context, false, false, 1.0f); //A*V
		auto d_u = U.data();
		auto d_sigma = w.data();
		auto column_size = U.rows();
		auto counting = thrust::make_counting_iterator <int>(0);
		thrust::for_each(counting, counting+U.size(), [=]__device__(int idx){
			int column = idx/column_size;
			float sigma = d_sigma[column];
			float u = d_u[idx];
			if(sigma != 0.0){
				d_u[idx] = u * 1.0/sigma;
			} else{
				d_u[idx] = 0.0;
			}
		} );
	}

	/**
	 * Obtain SVD attributes, which are as follows:
	 * 1.Singular Values
	 * 2.U matrix::Matrix
	 * 3.Explained Variance
	 * 4.Explained Variance Ratio
	 */
	template<typename T, typename S>
	void get_tsvd_attr(matrix::Matrix<T> &X, matrix::Matrix<T> &Q, S _Q, matrix::Matrix<T> &w, S _w, S _U, S _X_transformed, S _explained_variance, S _explained_variance_ratio, params _param, device::DeviceContext &context){

		//Obtain Q^T to obtain vector as row major order
		matrix::Matrix<T>Qt(Q.columns(), Q.rows());
		transpose(Q, Qt, context); //Needed for calculate_u()
		matrix::Matrix<T>QtTrunc(_param.k, Qt.columns());
		row_reverse_trunc_q(Qt, QtTrunc, context);

		if (_param.whiten) {
			auto d_q = QtTrunc.data();
			auto x_sqrt_row = std::sqrt(X.rows());
			auto d_sigma = w.data();
			auto column_size = QtTrunc.rows();
			auto counting = thrust::make_counting_iterator <int>(0);
			thrust::for_each(counting, counting+QtTrunc.size(), [=]__device__(int idx){
				int column = idx/column_size;
				T sigma = d_sigma[column];
				T q = d_q[idx];
				d_q[idx] = (q * x_sqrt_row)/std::sqrt(sigma);
			} );
		}

		//Obtain square root of eigenvalues, which are singular values
		T generic_zero = 0.0;
		w.transform([=]__device__(T elem){
			if(elem > generic_zero){
				return std::sqrt(elem);
			}else{
				return generic_zero;
			}
		}
		);

		//Sort from biggest singular value to smallest
		std::vector<double> w_temp(w.size());
		w.copy_to_host(w_temp.data()); //Send to host
		std::reverse(w_temp.begin(), w_temp.end());
		std::copy(w_temp.begin(), w_temp.begin() + _param.k, _w);
		matrix::Matrix<T>sigma(_param.k, 1);
		sigma.copy(w_temp.data());

		//Get U matrix::Matrix
		matrix::Matrix<T>U(X.rows(), _param.k);
		matrix::Matrix<T>QReversed(Q.rows(), Q.columns());
		col_reverse_q(Q, QReversed, context);
		calculate_u(X, QReversed, sigma, U, context);

		/**
		 * SVD sign correction (same as svd_flip() in sklearn) -> https://github.com/scikit-learn/scikit-learn/blob/master/sklearn/utils/extmath.py#L499
		   Sign correction to ensure deterministic output from SVD.
    	   Adjusts the columns of u and the rows of v such that the loadings in the
    	   columns in u that are largest in absolute value are always positive.
		 */
		std::vector<int> result_array(U.columns());
		max_index_per_column(U, result_array, context);
		matrix::Matrix<T>Signs(1, _param.k);
		thrust::device_vector<int> d_results = result_array;
		auto d_U = U.data();
		auto d_Signs = Signs.data();
		auto counting = thrust::make_counting_iterator <int>(0);
		auto ptr = d_results.data();
		thrust::for_each(counting, counting+Signs.size(), [=]__device__(int idx){
			int u_idx = ptr[idx];
			d_Signs[idx] = (T(0) < d_U[u_idx]) - (d_U[u_idx] < T(0));
		} );
		multiply_diag(U, Signs, U, context, false);
		multiply_diag(QtTrunc, Signs, QtTrunc, context, true);
		U.copy_to_host(_U); //Send to host
		QtTrunc.copy_to_host(_Q); //Send to host

		//U * Sigma (_X_transformed)
		matrix::Matrix<T>UmultSigma(U.rows(), U.columns());
		multiply_diag(U, sigma, UmultSigma, context, false);
		UmultSigma.copy_to_host(_X_transformed); //Send to host

		//Explained Variance
		matrix::Matrix<T>UOnesSigma(UmultSigma.rows(), 1);
		UOnesSigma.fill(1.0f);
		matrix::Matrix<T>USigmaVar(_param.k, 1);
		matrix::Matrix<T>USigmaColMean(_param.k, 1);
		multiply(UmultSigma, UOnesSigma, USigmaColMean, context, true, false, 1.0f);
		float m_usigma = UmultSigma.rows();
		multiply(USigmaColMean, 1/m_usigma, context);
		calc_var_numerator(UmultSigma, USigmaColMean, USigmaVar, context);
		multiply(USigmaVar, 1/m_usigma, context);
		USigmaVar.copy_to_host(_explained_variance); //Send to host

		//Explained Variance Ratio
		//Set aside matrix::Matrix of 1's for getting sum of columnar variances
		matrix::Matrix<T>XmultOnes(X.rows(), 1);
		XmultOnes.fill(1.0f);
		matrix::Matrix<T>XVar(X.columns(), 1);
		matrix::Matrix<T>XColMean(X.columns(), 1);
		multiply(X, XmultOnes, XColMean, context, true, false, 1.0f);
		float m = X.rows();
		multiply(XColMean, 1/m, context);
		calc_var_numerator(X, XColMean, XVar, context);
		multiply(XVar, 1/m, context);

		matrix::Matrix<T>XVarSum(1,1);
		multiply(XVar, XmultOnes, XVarSum, context, true, false, 1.0f);
		matrix::Matrix<T>ExplainedVarRatio(_param.k, 1);
		divide(USigmaVar, XVarSum, ExplainedVarRatio, context);
		ExplainedVarRatio.copy_to_host(_explained_variance_ratio); //Send to host
	}


	void outer_product(matrix::Matrix<float>& A, float eigen_value, const matrix::Matrix<float>& eigen_vector, const matrix::Matrix<float>& eigen_vector_transpose, device::DeviceContext& context)
	{
		safe_cublas(hipblasSger(context.cublas_handle, A.rows(), A.columns(), &eigen_value, eigen_vector.data(), 1, eigen_vector_transpose.data(), 1, A.data(), A.rows()));
	}

	void outer_product(matrix::Matrix<double>& A, double eigen_value, const matrix::Matrix<double>& eigen_vector, const matrix::Matrix<double>& eigen_vector_transpose, device::DeviceContext& context)
	{
		safe_cublas(hipblasDger(context.cublas_handle, A.rows(), A.columns(), &eigen_value, eigen_vector.data(), 1, eigen_vector_transpose.data(), 1, A.data(), A.rows()));
	}

	/**
	 * Conduct truncated svd using hipsolverDnSsyevd
	 *
	 * @param X
	 * @param _Q
	 * @param _w
	 * @param _U
	 * @param _explained_variance
	 * @param _explained_variance_ratio
	 * @param _param
	 */
	template<typename T, typename S>
	void cusolver_tsvd(matrix::Matrix<T> &X, S _Q, S _w, S _U, S _X_transformed, S _explained_variance, S _explained_variance_ratio, params _param){
		//Allocate matrix::Matrix for X^TX
		matrix::Matrix<T>XtX(_param.X_n, _param.X_n);

		//Create context
		device::DeviceContext context;

		//Multiply X and Xt and output result to XtX
		multiply(X, X, XtX, context, true, false, 1.0f);

		//Set up Q (V^T) and w (singular value) matrices (w is a matrix::Matrix of size Q.rows() by 1; really just a vector
		matrix::Matrix<T>Q(XtX.rows(), XtX.columns()); // n X n -> V^T
		matrix::Matrix<T>w(Q.rows(), 1);
		calculate_eigen_pairs_exact(XtX, Q, w, context);

		//Get tsvd attributes
		get_tsvd_attr(X, Q, _Q, w, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param, context);
	}

	/**
	 * Conduct truncated svd using the power method
	 *
	 * @param X
	 * @param _Q
	 * @param _w
	 * @param _U
	 * @param _explained_variance
	 * @param _explained_variance_ratio
	 * @param _param
	 */
	template<typename T, typename S>
	void power_tsvd(matrix::Matrix<T> &X, S _Q, S _w, S _U, S _X_transformed, S _explained_variance, S _explained_variance_ratio, params _param){
		//Allocate matrix::Matrix for X^TX
		matrix::Matrix<T>M(_param.X_n, _param.X_n);

		//Allocate matrix::Matrix to be used in outer product calculation
		matrix::Matrix<T>A(M.rows(), M.columns());

		//Create context
		device::DeviceContext context;

		//Multiply X and Xt and output result to XtX
		multiply(X, X, M, context, true, false, 1.0f);

		//Set up Q (V^T) and w (singular value) matrices (w is a matrix::Matrix of size Q.rows() by 1; really just a vector
		matrix::Matrix<T>Q(M.rows(), _param.k);
		matrix::Matrix<T>w(_param.k, 1);
		std::vector<T> w_temp(w.size());

		/* Power Method for finding all eigenvectors/eigen values
		 * Logic:
		 *
		 * Since the matrix::Matrix is symmetric, there exists an orthogonal basis of eigenvectors. Once you have found an eigenvector, extend it to an orthogonal basis,
		 * rewrite the matrix::Matrix in terms of this basis and restrict to the orthogonal space of the known eigenvector.
		 * This is comprised in the method of deflation:
		 *     You first find the first eigenvector v1 (with a maximal λ1), by iterating xn+1=Axn/|Axn|with a "random" initial x0. Once you have found a good approximation
		 *     for v1, you consider B=A−λ1|v1|2 * v1vT (this simple step replaces the "rewrite in terms of this basis" above). This is a matrix::Matrix that behaves like A for anything
		 *     orthogonal to v1 and zeroes out v1. Use the power method on B again, which will reveal v2, an eigenvector of a largest eigenvalue of B.
		 *     Then switch to C=B−λ2|v2|2 * v2vT so on.
		 */
		matrix::Matrix<T>b_k(_param.X_n, 1);
		matrix::Matrix<T>b_k1(_param.X_n, 1);

		for(int i = 0; i < _param.k; i ++){
			//Set aside vector of randoms (n x 1)
			b_k.random(_param.random_state + i);
			T previous_eigenvalue_estimate = FLT_MAX;
			T eigen_value_estimate = FLT_MAX;
			for(int iter=0; iter<_param.n_iter;iter++){
				//fprintf(stderr,"k=%d/%d iter=%d/%d\n",i,_param.k-1,iter,_param.n_iter); fflush(stderr);
				multiply(M, b_k, b_k1, context);
				dot_product(b_k1, b_k, &eigen_value_estimate, context);
				if(std::abs(eigen_value_estimate - previous_eigenvalue_estimate) <= (_param.tol * std::abs(previous_eigenvalue_estimate))) {
					break;
				}
				normalize_vector_cublas(b_k1, context);
				b_k.copy(b_k1);
				previous_eigenvalue_estimate = eigen_value_estimate;
			}
			//Obtain eigen value
			w_temp[i] = eigen_value_estimate;

			//Put eigen vector into (starting at last column of Q)
			thrust::copy(b_k.dptr(), b_k.dptr()+b_k.size(), Q.dptr()+Q.rows()*(Q.columns()-i-1));

			//Get rid of eigen effect from original matrix::Matrix (deflation)
			//multiply(A, 0.0, context);
			A.zero();
			outer_product(A, eigen_value_estimate, b_k, b_k, context);
			subtract(M, A, M, context);
		}
		//Fill in w from vector w_temp
		std::reverse(w_temp.begin(), w_temp.end());
		w.copy(w_temp.data());

		//Get tsvd attributes
		get_tsvd_attr(X, Q, _Q, w, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param, context);

	}

	/**
	 * Conduct truncated SVD on a matrix::Matrix with float type
	 *
	 * @param _X
	 * @param _Q
	 * @param _w
	 * @param _U
	 * @param _explained_variance
	 * @param _explained_variance_ratio
	 * @param _param
	 */
	void truncated_svd_float(const float *_X, float *_Q, float *_w, float *_U, float* _X_transformed, float *_explained_variance, float *_explained_variance_ratio, params _param)
	{
		safe_cuda(hipSetDevice(_param.gpu_id));
		matrix::Matrix<float>X(_param.X_m, _param.X_n);
		X.copy(_X);
		truncated_svd_matrix(X, _Q, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param);
	}

	/**
	 * Conduct truncated SVD on a matrix::Matrix with double type
	 *
	 * @param _X
	 * @param _Q
	 * @param _w
	 * @param _U
	 * @param _explained_variance
	 * @param _explained_variance_ratio
	 * @param _param
	 */
	void truncated_svd_double(const double *_X, double *_Q, double *_w, double *_U, double* _X_transformed, double *_explained_variance, double *_explained_variance_ratio, params _param)
	{
		safe_cuda(hipSetDevice(_param.gpu_id));
		matrix::Matrix<double>X(_param.X_m, _param.X_n);
		X.copy(_X);
		truncated_svd_matrix(X, _Q, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param);
	}

	template<typename T, typename S>
	tsvd_export void truncated_svd_matrix(matrix::Matrix<T> &X, S _Q, S _w, S _U, S _X_transformed, S _explained_variance, S _explained_variance_ratio, params _param)
	{
		std::string algorithm(_param.algorithm);
		try
		{
			if(algorithm == "cusolver"){
				if(_param.verbose==1){
				 fprintf(stderr,"Algorithm is cusolver with k = %d\n",_param.k); fflush(stderr);
				}
				tsvd::cusolver_tsvd(X, _Q, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param);
			}
			else {
				if(_param.verbose==1){
				 fprintf(stderr,"Algorithm is power with k = %d and number of iterations = %d\n",_param.k,_param.n_iter); fflush(stderr);
				}
				tsvd::power_tsvd(X, _Q, _w, _U, _X_transformed, _explained_variance, _explained_variance_ratio, _param);
			}
		}
		catch (const std::exception &e)
		  {
			std::cerr << "tsvd error: " << e.what() << "\n";
		  }
		catch (std::string e)
		  {
			std::cerr << "tsvd error: " << e << "\n";
		  }
		catch (...)
		  {
			std::cerr << "tsvd error\n";
		  }
	}
}

//Impl for floats and doubles
template void tsvd::truncated_svd_matrix<float>(matrix::Matrix<float> &X, float* _Q, float* _w, float* _U, float* _X_transformed, float* _explained_variance, float* _explained_variance_ratio, params _param);
template void tsvd::truncated_svd_matrix<double>(matrix::Matrix<double> &X, double* _Q, double* _w, double* _U, double* _X_transformed, double* _explained_variance, double* _explained_variance_ratio, params _param);
template void tsvd::cusolver_tsvd<double>(matrix::Matrix<double> &X, double* _Q, double* _w, double* _U, double* _X_transformed, double* _explained_variance, double* _explained_variance_ratio, params _param);
template void tsvd::power_tsvd<double>(matrix::Matrix<double> &X, double* _Q, double* _w, double* _U, double* _X_transformed, double* _explained_variance, double* _explained_variance_ratio, params _param);
template void tsvd::cusolver_tsvd<float>(matrix::Matrix<float> &X, float* _Q, float* _w, float* _U, float* _X_transformed, float* _explained_variance, float* _explained_variance_ratio, params _param);
template void tsvd::power_tsvd<float>(matrix::Matrix<float> &X, float* _Q, float* _w, float* _U, float* _X_transformed, float* _explained_variance, float* _explained_variance_ratio, params _param);


