#include "hip/hip_runtime.h"
#include "device_utilities.h"

__global__ void fp32Array2fp16Array(const float *fp32Array, half *fp16Array,
                                    const int size) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < size) {
    fp16Array[i] = __float2half(fp32Array[i]);
  }
}

__global__ void fp16Array2fp32Array(float *fp32Array, const half *fp16Array,
                                    const int size) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < size) {
    fp32Array[i] = __half2float(fp16Array[i]);
  }
}