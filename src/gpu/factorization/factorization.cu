#include "als.h"
#include "cuda_utils.h"
#include "solver/factorization.h"

template <class T> void free_data(T **ptr) {
  if (ptr)
    CUDACHECK(hipFree(*ptr));
}

void free_data_float(float **ptr) { free_data<float>(ptr); }

void free_data_double(double **ptr) { free_data<double>(ptr); }

void free_data_int(int **ptr) { free_data<int>(ptr); }

template <class T>
void copy_fecatorization_result(T *dst, const T **src, const int size) {
  CUDACHECK(hipMemcpy(dst, *src, sizeof(T) * size, hipMemcpyDeviceToHost));
}

void copy_fecatorization_result_float(float *dst, const float **src,
                                      const int size) {
  copy_fecatorization_result(dst, src, size);
}

void copy_fecatorization_result_double(double *dst, const double **src,
                                       const int size) {
  copy_fecatorization_result(dst, src, size);
}

template <class T>
int make_factorization_data(
    const int m, const int n, const int f, const long nnz, const long nnz_test,
    const int *csrRowIndexHostPtr, const int *csrColIndexHostPtr,
    const T *csrValHostPtr, const int *cscRowIndexHostPtr,
    const int *cscColIndexHostPtr, const T *cscValHostPtr,
    const int *cooRowIndexHostPtr, const int *cooColIndexHostPtr,
    const T *cooValHostPtr, T *thetaTHost, T *XTHost,
    const int *cooRowIndexTestHostPtr, const int *cooColIndexTestHostPtr,
    const T *cooValTestHostPtr,

    int **csrRowIndexDevicePtr, int **csrColIndexDevicePtr, T **csrValDevicePtr,
    int **cscRowIndexDevicePtr, int **cscColIndexDevicePtr, T **cscValDevicePtr,
    int **cooRowIndexDevicePtr, int **cooColIndexDevicePtr, T **cooValDevicePtr,
    T **thetaTDevice, T **XTDevice, int **cooRowIndexTestDevicePtr,
    int **cooColIndexTestDevicePtr, T **cooValDeviceTestPtr) {

  CUDACHECK(hipMalloc((void **)cooRowIndexDevicePtr,
                       nnz * sizeof(**cooRowIndexDevicePtr)));
  CUDACHECK(hipMemcpy(*cooRowIndexDevicePtr, cooRowIndexHostPtr,
                       (size_t)(nnz * sizeof(**cooRowIndexDevicePtr)),
                       hipMemcpyHostToDevice));
  CUDACHECK(hipMalloc((void **)cooColIndexDevicePtr,
                       nnz * sizeof(**cooColIndexDevicePtr)));
  CUDACHECK(hipMemcpy(*cooColIndexDevicePtr, cooColIndexHostPtr,
                       (size_t)(nnz * sizeof(**cooColIndexDevicePtr)),
                       hipMemcpyHostToDevice));
  CUDACHECK(
      hipMalloc((void **)cooValDevicePtr, nnz * sizeof(**cooValDevicePtr)));
  CUDACHECK(hipMemcpy(*cooValDevicePtr, cooValHostPtr,
                       (size_t)(nnz * sizeof(**cooValDevicePtr)),
                       hipMemcpyHostToDevice));

  CUDACHECK(hipMalloc((void **)cscRowIndexDevicePtr,
                       nnz * sizeof(**cscRowIndexDevicePtr)));
  CUDACHECK(hipMalloc((void **)cscColIndexDevicePtr,
                       (n + 1) * sizeof(**cscColIndexDevicePtr)));
  CUDACHECK(
      hipMalloc((void **)cscValDevicePtr, nnz * sizeof(**cscValDevicePtr)));
  // dimension: F*N
  CUDACHECK(hipMalloc((void **)thetaTDevice, f * n * sizeof(**thetaTDevice)));
  // dimension: M*F
  CUDACHECK(hipMalloc((void **)XTDevice, f * m * sizeof(**XTDevice)));
  CUDACHECK(hipMemcpy(*cscRowIndexDevicePtr, cscRowIndexHostPtr,
                       (size_t)nnz * sizeof(**cscRowIndexDevicePtr),
                       hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(*cscColIndexDevicePtr, cscColIndexHostPtr,
                       (size_t)(n + 1) * sizeof(**cscColIndexDevicePtr),
                       hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(*cscValDevicePtr, cscValHostPtr,
                       (size_t)(nnz * sizeof(**cscValDevicePtr)),
                       hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(*thetaTDevice, thetaTHost,
                       (size_t)(n * f * sizeof(**thetaTDevice)),
                       hipMemcpyHostToDevice));
  // CG needs XT
  CUDACHECK(hipMemcpy(*XTDevice, XTHost, (size_t)(m * f * sizeof(**XTDevice)),
                       hipMemcpyHostToDevice));

  CUDACHECK(hipMalloc((void **)csrRowIndexDevicePtr,
                       (m + 1) * sizeof(**csrRowIndexDevicePtr)));
  CUDACHECK(hipMalloc((void **)csrColIndexDevicePtr,
                       nnz * sizeof(**csrColIndexDevicePtr)));
  CUDACHECK(
      hipMalloc((void **)csrValDevicePtr, nnz * sizeof(**csrValDevicePtr)));
  CUDACHECK(hipMemcpy(*csrRowIndexDevicePtr, csrRowIndexHostPtr,
                       (size_t)((m + 1) * sizeof(**csrRowIndexDevicePtr)),
                       hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(*csrColIndexDevicePtr, csrColIndexHostPtr,
                       (size_t)(nnz * sizeof(**csrColIndexDevicePtr)),
                       hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(*csrValDevicePtr, csrValHostPtr,
                       (size_t)(nnz * sizeof(**csrValDevicePtr)),
                       hipMemcpyHostToDevice));

  if (cooColIndexTestHostPtr && cooRowIndexTestHostPtr && cooValTestHostPtr &&
      nnz_test > 0) {
    CUDACHECK(hipMalloc((void **)cooRowIndexTestDevicePtr,
                         nnz_test * sizeof(**cooRowIndexTestDevicePtr)));
    CUDACHECK(hipMalloc((void **)cooColIndexTestDevicePtr,
                         nnz_test * sizeof(**cooColIndexTestDevicePtr)));
    CUDACHECK(hipMalloc((void **)cooValDeviceTestPtr,
                         nnz_test * sizeof(**cooValDeviceTestPtr)));
    CUDACHECK(
        hipMemcpy(*cooRowIndexTestDevicePtr, cooRowIndexTestHostPtr,
                   (size_t)(nnz_test * sizeof(**cooRowIndexTestDevicePtr)),
                   hipMemcpyHostToDevice));
    CUDACHECK(
        hipMemcpy(*cooColIndexTestDevicePtr, cooColIndexTestHostPtr,
                   (size_t)(nnz_test * sizeof(**cooColIndexTestDevicePtr)),
                   hipMemcpyHostToDevice));
    CUDACHECK(hipMemcpy(*cooValDeviceTestPtr, cooValTestHostPtr,
                         (size_t)(nnz_test * sizeof(**cooValDeviceTestPtr)),
                         hipMemcpyHostToDevice));
  }
  return 0;
}

template <typename T>
T factorization_score(const int m, const int n, const int f, const long nnz,
                      const T lambda, T **thetaTDevice, T **XTDevice,
                      int **cooRowIndexDevicePtr, int **cooColIndexDevicePtr,
                      T **cooValDevicePtr) {
  ALSFactorization<T> factorization(m, n, f, lambda, *thetaTDevice, *XTDevice);
  return factorization.Score(*cooRowIndexDevicePtr, *cooColIndexDevicePtr,
                             *cooValDevicePtr, nnz);
}

template <typename T>
int run_factorization_step(
    const int m, const int n, const int f, const long nnz, const T lambda,
    int **csrRowIndexDevicePtr, int **csrColIndexDevicePtr, T **csrValDevicePtr,
    int **cscRowIndexDevicePtr, int **cscColIndexDevicePtr, T **cscValDevicePtr,
    T **thetaTDevice, T **XTDevice, const int X_BATCH, const int THETA_BATCH) {
  ALSFactorization<T> factorization(m, n, f, lambda, *thetaTDevice, *XTDevice);
  factorization.Iter(*csrRowIndexDevicePtr, *csrColIndexDevicePtr,
                     *csrValDevicePtr, *cscRowIndexDevicePtr,
                     *cscColIndexDevicePtr, *cscValDevicePtr, nnz, X_BATCH,
                     THETA_BATCH);
  return 0;
}

float factorization_score_float(const int m, const int n, const int f,
                                const long nnz, const float lambda,
                                float **thetaTDevice, float **XTDevice,
                                int **cooRowIndexDevicePtr,
                                int **cooColIndexDevicePtr,
                                float **cooValDevicePtr) {
  return factorization_score<float>(m, n, f, nnz, lambda, thetaTDevice,
                                    XTDevice, cooRowIndexDevicePtr,
                                    cooColIndexDevicePtr, cooValDevicePtr);
}

double factorization_score_double(const int m, const int n, const int f,
                                  const long nnz, const float lambda,
                                  double **thetaTDevice, double **XTDevice,
                                  int **cooRowIndexDevicePtr,
                                  int **cooColIndexDevicePtr,
                                  double **cooValDevicePtr) {
  return 0.0;
}

int run_factorization_step_double(
    const int m, const int n, const int f, const long nnz, const double lambda,
    int **csrRowIndexDevicePtr, int **csrColIndexDevicePtr,
    double **csrValDevicePtr, int **cscRowIndexDevicePtr,
    int **cscColIndexDevicePtr, double **cscValDevicePtr, double **thetaTDevice,
    double **XTDevice, const int X_BATCH, const int THETA_BATCH) {
  return 1;
  // TODO: implement
  // return run_factorization_step<double>(m, n, f, nnz, nnz_test,
  //     csrRowIndexDevicePtr, csrColIndexDevicePtr, csrValDevicePtr,
  //     cscRowIndexDevicePtr, cscColIndexDevicePtr, cscValDevicePtr,
  //     cooRowIndexDevicePtr, thetaTDevice, XTDevice, cooRowIndexTestDevicePtr,
  //     cooColIndexTestDevicePtr, cooValTestDevicePtr);
}

int run_factorization_step_float(
    const int m, const int n, const int f, const long nnz, const float lambda,
    int **csrRowIndexDevicePtr, int **csrColIndexDevicePtr,
    float **csrValDevicePtr, int **cscRowIndexDevicePtr,
    int **cscColIndexDevicePtr, float **cscValDevicePtr, float **thetaTDevice,
    float **XTDevice, const int X_BATCH, const int THETA_BATCH) {
  return run_factorization_step<float>(
      m, n, f, nnz, lambda, csrRowIndexDevicePtr, csrColIndexDevicePtr,
      csrValDevicePtr, cscRowIndexDevicePtr, cscColIndexDevicePtr,
      cscValDevicePtr, thetaTDevice, XTDevice, X_BATCH, THETA_BATCH);
}

int make_factorization_data_double(
    const int m, const int n, const int f, const long nnz, const long nnz_test,
    const int *csrRowIndexHostPtr, const int *csrColIndexHostPtr,
    const double *csrValHostPtr, const int *cscRowIndexHostPtr,
    const int *cscColIndexHostPtr, const double *cscValHostPtr,
    const int *cooRowIndexHostPtr, const int *cooColIndexHostPtr,
    const double *cooValHostPtr, double *thetaTHost, double *XTHost,
    const int *cooRowIndexTestHostPtr, const int *cooColIndexTestHostPtr,
    const double *cooValTestHostPtr,

    int **csrRowIndexDevicePtr, int **csrColIndexDevicePtr,
    double **csrValDevicePtr, int **cscRowIndexDevicePtr,
    int **cscColIndexDevicePtr, double **cscValDevicePtr,
    int **cooRowIndexDevicePtr, int **cooColIndexDevicePtr,
    double **cooValDevicePtr, double **thetaTDevice, double **XTDevice,
    int **cooRowIndexTestDevicePtr, int **cooColIndexTestDevicePtr,
    double **cooValTestDevicePtr) {
  return make_factorization_data<double>(
      m, n, f, nnz, nnz_test, csrRowIndexHostPtr, csrColIndexHostPtr,
      csrValHostPtr, cscRowIndexHostPtr, cscColIndexHostPtr, cscValHostPtr,
      cooRowIndexHostPtr, cooColIndexHostPtr, cooValHostPtr, thetaTHost, XTHost,
      cooRowIndexTestHostPtr, cooColIndexTestHostPtr, cooValTestHostPtr,

      csrRowIndexDevicePtr, csrColIndexDevicePtr, csrValDevicePtr,
      cscRowIndexDevicePtr, cscColIndexDevicePtr, cscValDevicePtr,
      cooRowIndexDevicePtr, cooColIndexDevicePtr, cooValDevicePtr, thetaTDevice,
      XTDevice, cooRowIndexTestDevicePtr, cooColIndexTestDevicePtr,
      cooValTestDevicePtr);
}

int make_factorization_data_float(
    const int m, const int n, const int f, const long nnz, const long nnz_test,
    const int *csrRowIndexHostPtr, const int *csrColIndexHostPtr,
    const float *csrValHostPtr, const int *cscRowIndexHostPtr,
    const int *cscColIndexHostPtr, const float *cscValHostPtr,
    const int *cooRowIndexHostPtr, const int *cooColIndexHostPtr,
    const float *cooValHostPtr, float *thetaTHost, float *XTHost,
    const int *cooRowIndexTestHostPtr, const int *cooColIndexTestHostPtr,
    const float *cooValTestHostPtr,

    int **csrRowIndexDevicePtr, int **csrColIndexDevicePtr,
    float **csrValDevicePtr, int **cscRowIndexDevicePtr,
    int **cscColIndexDevicePtr, float **cscValDevicePtr,
    int **cooRowIndexDevicePtr, int **cooColIndexDevicePtr,
    float **cooValDevicePtr, float **thetaTDevice, float **XTDevice,
    int **cooRowIndexTestDevicePtr, int **cooColIndexTestDevicePtr,
    float **cooValTestDevicePtr) {
  return make_factorization_data<float>(
      m, n, f, nnz, nnz_test, csrRowIndexHostPtr, csrColIndexHostPtr,
      csrValHostPtr, cscRowIndexHostPtr, cscColIndexHostPtr, cscValHostPtr,
      cooRowIndexHostPtr, cooColIndexHostPtr, cooValHostPtr, thetaTHost, XTHost,
      cooRowIndexTestHostPtr, cooColIndexTestHostPtr, cooValTestHostPtr,

      csrRowIndexDevicePtr, csrColIndexDevicePtr, csrValDevicePtr,
      cscRowIndexDevicePtr, cscColIndexDevicePtr, cscValDevicePtr,
      cooRowIndexDevicePtr, cooColIndexDevicePtr, cooValDevicePtr, thetaTDevice,
      XTDevice, cooRowIndexTestDevicePtr, cooColIndexTestDevicePtr,
      cooValTestDevicePtr);
}